#include "hip/hip_runtime.h"

/*
	Copyright (C) 2020 David Cornu
	for the Convolutional Interactive Artificial 
	Neural Networks by/for Astrophysicists (CIANNA) Code
	(https://github.com/Deyht/CIANNA)

	Licensed under the Apache License, Version 2.0 (the "License");
	you may not use this file except in compliance with the License.
	You may obtain a copy of the License at

		http://www.apache.org/licenses/LICENSE-2.0

	Unless required by applicable law or agreed to in writing, software
	distributed under the License is distributed on an "AS IS" BASIS,
	WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
	See the License for the specific language governing permissions and
	limitations under the License.
*/





#include "../prototypes.h"

static int cu_blocks;
static pool_param *p_param;

//public are in prototypes.h

//private
void cuda_forward_pool_layer(layer* current);
void cuda_backward_pool_layer(layer* current);

__global__ void pooling_kernel_FP32(float* input, float* output, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int w_size, int h_size, int d_size, int w_size_out, int h_size_out, int d_size_out, int length);
__global__ void pooling_kernel_FP16(half* input, half* output, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int w_size, int h_size, int d_size, int w_size_out, int h_size_out, int d_size_out, int length);
__global__ void avg_pooling_kernel_FP16(half* input, half* output, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int w_size, int h_size, int d_size, int w_size_out, int h_size_out, int d_size_out, int length);
/*__global__ void deltah_pool_FP32(float* delta_o, float* delta_o_unpool, int* pool_map, int pool_size, int len, int batch_size, int image_size, int map_size, int column_length);
__global__ void deltah_pool_FP16(half* delta_o, half* delta_o_unpool, int* pool_map, int pool_size, int len, int batch_size, int image_size, int map_size, int column_length);*/
__global__ void deltah_pool_cont_FP32(float* delta_o, float* delta_o_unpool, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int len, int batch_size, int image_size, int w_size, int h_size);
__global__ void deltah_pool_cont_FP16(half* delta_o, half* delta_o_unpool, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int len, int batch_size, int image_size, int w_size, int h_size);
__global__ void deltah_avg_pool_cont_FP16(half* delta_o, half* delta_o_unpool, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int len, int batch_size, int image_size, int w_size, int h_size);

__global__ void init_block_state_pool(unsigned int seed, hiprandState_t* states);
__global__ void cuda_dropout_select_pool(int* mask, int size, float drop_rate, hiprandState_t* states);
__global__ void cuda_dropout_apply_pool_FP32(float* table, int batch_size, int dim, int* mask, int size);
__global__ void cuda_dropout_apply_pool_FP16(half* table, int batch_size, int dim, int* mask, int size);

void cuda_pool_define(layer *current)
{
	current->forward = cuda_forward_pool_layer;
	current->backprop = cuda_backward_pool_layer;
}

void cuda_convert_pool_layer(layer *current)
{
	p_param = (pool_param*)current->param;

	cuda_convert_table_int(current->c_network, &(p_param->pool_map), p_param->nb_area[0] 
		* p_param->nb_area[1] * p_param->nb_area[2] * p_param->nb_maps * current->c_network->batch_size);
	cuda_convert_table(current->c_network, &(current->output), p_param->nb_area[0] 
		* p_param->nb_area[1] * p_param->nb_area[2] * p_param->nb_maps * current->c_network->batch_size);
	
	cuda_convert_table(current->c_network, &(current->delta_o), p_param->nb_area[0] 
		* p_param->nb_area[1] * p_param->nb_area[2] * p_param->nb_maps * current->c_network->batch_size);
	
	cuda_convert_table(current->c_network, &(p_param->temp_delta_o), p_param->prev_size[0] 
		* p_param->prev_size[1] * p_param->prev_size[2] * p_param->prev_depth * current->c_network->batch_size);
		
	if(p_param->dropout_rate > 0.01)
	{
		cuda_convert_table_int(current->c_network, &(p_param->dropout_mask), p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]));
		hipMalloc((void**) &p_param->block_state, (p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2])) * sizeof(hiprandState_t));
		cu_blocks = (p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]));
		init_block_state_pool<<< cu_blocks, 1>>>(time(NULL),(hiprandState_t*)p_param->block_state);
	}
}


void cuda_forward_pool_layer(layer* current)
{
	if(current->c_network->length == 0)
		return;
		
	p_param = (pool_param*) current->param;
	
	//late declaration of CUDA kernel sizes
	dim3 threadsPerBlock(8, 8);
	//create numBlocks regarding the layer dimensions
    dim3 numBlocks((p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2] + threadsPerBlock.x - 1) / threadsPerBlock.x,
    	(current->c_network->batch_size * p_param->nb_maps + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    switch(current->c_network->use_cuda_TC)
	{
		default:
		case 0:
			pooling_kernel_FP32<<< numBlocks , threadsPerBlock >>>((float*)current->input, 
				(float*)current->output, p_param->pool_map, p_param->p_size[0], p_param->p_size[1], p_param->p_size[2], p_param->prev_size[0],
				p_param->prev_size[1], p_param->prev_size[2], p_param->nb_area[0], p_param->nb_area[1],
				p_param->nb_area[2], p_param->nb_maps * current->c_network->batch_size);
			break;
		case 1:
			avg_pooling_kernel_FP16<<< numBlocks , threadsPerBlock >>>((half*)current->input, 
				(half*)current->output, p_param->pool_map, p_param->p_size[0], p_param->p_size[1], p_param->p_size[2], p_param->prev_size[0],
				p_param->prev_size[1], p_param->prev_size[2], p_param->nb_area[0], p_param->nb_area[1],
				p_param->nb_area[2], p_param->nb_maps * current->c_network->batch_size);
			break;
	}

	if(p_param->dropout_rate > 0.01 && (!current->c_network->is_inference || current->c_network->inference_drop_mode == MC_MODEL))
	{
		cu_blocks = (p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]));
		cuda_dropout_select_pool<<<cu_blocks, 1>>>(p_param->dropout_mask, p_param->nb_maps 
			* (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]), p_param->dropout_rate, (hiprandState_t*) p_param->block_state);	
		
		dim3 threadsPerBlock(32, 8);
		dim3 numBlocks((p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]) + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(current->c_network->batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);
		
		switch(current->c_network->use_cuda_TC)
		{
			default:
			case 0:
				cuda_dropout_apply_pool_FP32<<<numBlocks, threadsPerBlock>>>((float*)current->output, 
					current->c_network->batch_size, (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]),
					p_param->dropout_mask, p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]));
				break;
			case 1:
				cuda_dropout_apply_pool_FP16<<<numBlocks, threadsPerBlock>>>((half*)current->output, 
					current->c_network->batch_size, (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]), 
					p_param->dropout_mask, p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]));
				break;
		}
	}



}


void cuda_backward_pool_layer(layer* current)
{	
	p_param = (pool_param*) current->param;
	
	if(p_param->dropout_rate > 0.01)
	{
		dim3 threadsPerBlock(32, 8);
		dim3 numBlocks((p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]) + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(current->c_network->batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);
		
		switch(current->c_network->use_cuda_TC)
		{
			default:
			case 0:
				cuda_dropout_apply_pool_FP32<<<numBlocks, threadsPerBlock>>>((float*)current->delta_o, 
					current->c_network->batch_size, (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]), 
					p_param->dropout_mask, p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]));
				break;
			case 1:
				cuda_dropout_apply_pool_FP16<<<numBlocks, threadsPerBlock>>>((half*)current->delta_o, 
					current->c_network->batch_size, (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]), 
					p_param->dropout_mask, p_param->nb_maps * (p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2]));
				break;
		}
	}

	if(current->previous != NULL)
	{
	
		if(current->previous->type == CONV)
		{
			switch(current->c_network->use_cuda_TC)
			{
				default:
				case 0:
					//array must be set to 0 as deltah_pool do not erase previous values
					hipMemset(current->previous->delta_o, 0.0f, p_param->prev_depth 
						* p_param->prev_size[0] * p_param->prev_size[1] * p_param->prev_size[2]
						* current->c_network->batch_size*sizeof(float));
				
					cu_blocks = (current->c_network->batch_size*(p_param->nb_maps * p_param->nb_area[0] 
						* p_param->nb_area[1] * p_param->nb_area[2]) + cu_threads - 1) / cu_threads;

					deltah_pool_cont_FP32<<< cu_blocks, cu_threads >>>((float*)current->delta_o, 
						(float*)current->previous->delta_o, p_param->pool_map, p_param->p_size[0], p_param->p_size[1], p_param->p_size[2], 
						current->c_network->length, current->c_network->batch_size, p_param->nb_maps 
						* p_param->nb_area[0] * p_param->nb_area[1], p_param->nb_area[0], p_param->nb_area[1]);
					break;
				case 1:
					//array must be set to 0 as deltah_pool do not erase previous values
					hipMemset(current->previous->delta_o, 0.0f, p_param->prev_depth 
						* p_param->prev_size[0] * p_param->prev_size[1] *p_param->prev_size[2]
						* current->c_network->batch_size*sizeof(half));
				
					cu_blocks = (current->c_network->batch_size*(p_param->nb_maps * p_param->nb_area[0] 
						* p_param->nb_area[1] * p_param->nb_area[2]) + cu_threads - 1) / cu_threads;

					deltah_avg_pool_cont_FP16<<< cu_blocks, cu_threads >>>((half*)current->delta_o, 
						(half*)current->previous->delta_o, p_param->pool_map, p_param->p_size[0], p_param->p_size[1], p_param->p_size[2],
						current->c_network->length, current->c_network->batch_size, p_param->nb_maps 
						* p_param->nb_area[0] * p_param->nb_area[1] * p_param->nb_area[2], p_param->nb_area[0], p_param->nb_area[1]);
					break;
			}
		}
		
		current->previous->deriv_activation(current->previous);
	}
}


__global__ void pooling_kernel_FP32(float* input, float* output, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int w_size, int h_size, int d_size, int w_size_out, int h_size_out, int d_size_out, int length)
{
	/*
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int k = blockIdx.y*blockDim.y + threadIdx.y;
	int x, y, z, x_max, y_max, z_max, pos, pos_x, pos_y, pos_z, pos_out;
	
	pos_z = i / (w_size_out*h_size_out); 
	pos_y = (i % (w_size_out*h_size_out)) / w_size_out;
	pos_x = (i % (w_size_out*h_size_out)) % w_size_out;
	
	pos_out = k*(w_size_out*h_size_out*d_size_out) + pos_x + pos_y*w_size_out + pos_z*(w_size_out*h_size_out);
	
	pos = k*w_size*h_size*d_size + pos_x*pool_size + pos_y*pool_size*w_size + pos_z*pool_size*w_size*h_size;
	
	if(pos_x < w_size_out && pos_y < h_size_out && pos_z < d_size_out && k < length)
	{
		x_max = 0;
		y_max = 0;
		z_max = 0;
		for(x = 0; x < pool_size; x++)
			for(y = 0; y < pool_size; y++)
				for(z = 0; z < pool_size; z++)
					if(input[pos + x_max*w_size*h_size + y_max*w_size + z_max] < input[pos + x*w_size*h_size + y*w_size + z])
					{
						x_max = x;
						y_max = y;
						z_max = z;
					}
		pool_map[pos_out] = (x_max*pool_size*pool_size + y_max*pool_size + z_max);
		output[pos_out] = input[pos + x_max*w_size*h_size + y_max*w_size + z_max];
	}*/
}

__global__ void pooling_kernel_FP16(half* input, half* output, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int w_size, int h_size, int d_size, int w_size_out, int h_size_out, int d_size_out, int length)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int k = blockIdx.y*blockDim.y + threadIdx.y;
	int x, y, z, x_max, y_max, z_max, pos, pos_x, pos_y, pos_z, pos_out;
	
	pos_z = i / (w_size_out*h_size_out); 
	pos_y = (i % (w_size_out*h_size_out)) / w_size_out;
	pos_x = (i % (w_size_out*h_size_out)) % w_size_out;
	
	pos_out = k*(w_size_out*h_size_out*d_size_out) + pos_x + pos_y*w_size_out + pos_z*(w_size_out*h_size_out);
	
	pos = k*w_size*h_size*d_size + pos_x*pool_size_w + pos_y*pool_size_h*w_size + pos_z*pool_size_d*w_size*h_size;
	
	if(pos_x < w_size_out && pos_y < h_size_out && pos_z < d_size_out && k < length)
	{
		x_max = 0;
		y_max = 0;
		z_max = 0;
		for(x = 0; x < pool_size_d; x++)
			for(y = 0; y < pool_size_h; y++)
				for(z = 0; z < pool_size_w; z++)
					if(input[pos + x_max*w_size*h_size + y_max*w_size + z_max] < input[pos + x*w_size*h_size + y*w_size + z])
					{
						x_max = x;
						y_max = y;
						z_max = z;
					}
		pool_map[pos_out] = (x_max*pool_size_w*pool_size_h + y_max*pool_size_w + z_max);
		output[pos_out] = input[pos + x_max*w_size*h_size + y_max*w_size + z_max];
	}
}


__global__ void avg_pooling_kernel_FP16(half* input, half* output, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int w_size, int h_size, int d_size, int w_size_out, int h_size_out, int d_size_out, int length)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int k = blockIdx.y*blockDim.y + threadIdx.y;
	int x, y, z, pos, pos_x, pos_y, pos_z, pos_out;
	float r_avg = 0.0f;
	
	pos_z = i / (w_size_out*h_size_out); 
	pos_y = (i % (w_size_out*h_size_out)) / w_size_out;
	pos_x = (i % (w_size_out*h_size_out)) % w_size_out;
	
	pos_out = k*(w_size_out*h_size_out*d_size_out) + pos_x + pos_y*w_size_out + pos_z*(w_size_out*h_size_out);
	
	pos = k*w_size*h_size*d_size + pos_x*pool_size_w + pos_y*pool_size_h*w_size + pos_z*pool_size_d*w_size*h_size;
	
	if(pos_x < w_size_out && pos_y < h_size_out && pos_z < d_size_out && k < length)
	{
		for(x = 0; x < pool_size_d; x++)
			for(y = 0; y < pool_size_h; y++)
				for(z = 0; z < pool_size_w; z++)
					r_avg += (float) input[pos + x*w_size*h_size + y*w_size + z];
					
		output[pos_out] = (half) (r_avg/(pool_size_w*pool_size_h*pool_size_d));
	}
}

/*
// Do the same thing as the funciton below but slightly slower
__global__ void deltah_pool_FP32(float* delta_o, float* delta_o_unpool, int* pool_map, int pool_size, int len, int batch_size, int image_size, int map_size, int column_length)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int map_id, image_id, map_col, map_pos;

	if(i > batch_size*image_size)
		return;

	map_id = i / (map_size*batch_size);
	image_id = i % (map_size*batch_size) / map_size;
	map_col =  i % (map_size*batch_size) % map_size / column_length;
	map_pos = i % (map_size*batch_size) % map_size % column_length;
	
	delta_o_unpool += (map_id * (map_size*batch_size) + image_id * map_size) * pool_size * pool_size;
	delta_o_unpool += map_col * column_length * pool_size * pool_size + map_pos * pool_size;
	delta_o_unpool += + (int(pool_map[i])/pool_size) * column_length * pool_size 
		+ (int(pool_map[i])%pool_size);
	
	if(i < len*image_size)
		*delta_o_unpool = delta_o[i];
	else
		*delta_o_unpool = 0.0;
}

__global__ void deltah_pool_FP16(half* delta_o, half* delta_o_unpool, int* pool_map, int pool_size, int len, int batch_size, int image_size, int map_size, int column_length)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int map_id, image_id, map_col, map_pos;

	if(i > batch_size*image_size)
		return;

	map_id = i / (map_size*batch_size);
	image_id = i % (map_size*batch_size) / map_size;
	map_col =  i % (map_size*batch_size) % map_size / column_length;
	map_pos = i % (map_size*batch_size) % map_size % column_length;
	
	delta_o_unpool += (map_id * (map_size*batch_size) + image_id * map_size) * pool_size * pool_size;
	delta_o_unpool += map_col * column_length * pool_size * pool_size + map_pos * pool_size;
	delta_o_unpool += (int(pool_map[i])/pool_size) * column_length * pool_size 
		+ (int(pool_map[i])%pool_size);
	
	if(i < len*image_size)
		*delta_o_unpool = delta_o[i];
	else
		*delta_o_unpool = (half)0.0f;
}
*/

__global__ void deltah_pool_cont_FP32(float* delta_o, float* delta_o_unpool, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int len, int batch_size, int image_size, int w_size, int h_size)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int pos;

	pos = i;
	
	if(i < len*image_size)
	{
		//add mask of locations
		delta_o_unpool += (i/(w_size*h_size)) * (w_size*h_size) * pool_size_w * pool_size_h * pool_size_d
			+ ((i%(w_size*h_size))/w_size) * w_size * pool_size_w * pool_size_h
			+ ((i%(w_size*h_size))%w_size) * pool_size_w +
			+ (int(pool_map[i])/(pool_size_w*pool_size_h)) * w_size*h_size * pool_size_w*pool_size_h 
			+ ((int(pool_map[i])%(pool_size_w*pool_size_h))/pool_size_h) * w_size * pool_size_w
			+ ((int(pool_map[i])%(pool_size_w*pool_size_h))%pool_size_h);
		
		*delta_o_unpool = delta_o[pos];
	}
	
}


__global__ void deltah_pool_cont_FP16(half* delta_o, half* delta_o_unpool, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int len, int batch_size, int image_size, int w_size, int h_size)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int pos;

	pos = i;
	
	if(i < len*image_size)
	{
		//add mask of locations
		delta_o_unpool += (i/(w_size*h_size)) * (w_size*h_size) * pool_size_w * pool_size_h * pool_size_d
			+ ((i%(w_size*h_size))/w_size) * w_size * pool_size_w * pool_size_h
			+ ((i%(w_size*h_size))%w_size) * pool_size_w +
			+ (int(pool_map[i])/(pool_size_w*pool_size_h)) * w_size*h_size * pool_size_w*pool_size_h 
			+ ((int(pool_map[i])%(pool_size_w*pool_size_h))/pool_size_h) * w_size * pool_size_w
			+ ((int(pool_map[i])%(pool_size_w*pool_size_h))%pool_size_h);
		
		*delta_o_unpool = delta_o[pos];
	}
	
}

__global__ void deltah_avg_pool_cont_FP16(half* delta_o, half* delta_o_unpool, int* pool_map, int pool_size_w, int pool_size_h, int pool_size_d, int len, int batch_size, int image_size, int w_size, int h_size)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int pos, x, y, z;

	pos = i;
	
	if(i < len*image_size)
	{
		//add mask of locations
		delta_o_unpool += (i/(w_size*h_size)) * (w_size*h_size) * pool_size_w * pool_size_h * pool_size_d
						+ ((i%(w_size*h_size))/h_size) * h_size * pool_size_w * pool_size_h
						+ ((i%(w_size*h_size))%h_size) * pool_size_w;
		
		for(x = 0; x < pool_size_d; x++)
			for(y = 0; y < pool_size_h; y++)
				for(z = 0; z < pool_size_w; z++)
					 delta_o_unpool[(x) * w_size * h_size * pool_size_w * pool_size_h 
						+ (y) * w_size * pool_size_w + (z)] = (half)((float)delta_o[pos]/(pool_size_w*pool_size_h*pool_size_d));
	}
	
}


__global__ void init_block_state_pool(unsigned int seed,  hiprandState_t* states)
{
	hiprand_init((seed << 20) + blockIdx.x, /* the seed can be the same for each core, here we pass the time in from the CPU */
              0, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! 
			     Currently use an alternative definition with Id adjunct to seed*/
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}


__global__ void cuda_dropout_select_pool(int* mask, int size, float drop_rate, hiprandState_t* states)
{
	int i = blockIdx.x;
	
	float rand;
	if(i < size)
	{
		rand = hiprand_uniform(&states[i]);
		if(rand < drop_rate)
			mask[i] = 0;
		else
			mask[i] = 1;
	}
}

__global__ void cuda_dropout_apply_pool_FP32(float* table, int batch_size, int dim, int* mask, int size)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	
	int c_depth = j / dim;
	int current_id = j % dim;
	int offset = dim*batch_size;

	if(i < batch_size && j < size)
	{
		table[i*dim + c_depth*offset + current_id] *= mask[j];
	}
}

__global__ void cuda_dropout_apply_pool_FP16(half* table, int batch_size, int dim, int* mask, int size)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;

	int c_depth = j / dim;
        int current_id = j % dim;
        int offset = dim*batch_size;

        if(i < batch_size && j < size)
        {
                table[i*dim + c_depth*offset + current_id] *= mask[j];
        }
}









