#include "hip/hip_runtime.h"
	
/*
	Copyright (C) 2023 David Cornu
	for the Convolutional Interactive Artificial 
	Neural Networks by/for Astrophysicists (CIANNA) Code
	(https://github.com/Deyht/CIANNA)

	Licensed under the Apache License, Version 2.0 (the "License");
	you may not use this file except in compliance with the License.
	You may obtain a copy of the License at

		http://www.apache.org/licenses/LICENSE-2.0

	Unless required by applicable law or agreed to in writing, software
	distributed under the License is distributed on an "AS IS" BASIS,
	WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
	See the License for the specific language governing permissions and
	limitations under the License.
*/


#include "../prototypes.h"


static int cu_blocks;

//public are in "prototypes.h"

//#####################################################


//#####################################################
//		  ReLU activation related templates
//#####################################################

//Is in fact a leaky ReLU, to obtain true ReLU set leaking_factor to 0
#define ReLU_activation_kernel(name, type)																										\
__global__ void ReLU_activation_kernel_##name(void *i_tab, int dim, int biased_dim, int offset,													\
	float saturation, float leaking_factor, int length, size_t size)																			\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
																																				\
	type* tab = (type*) i_tab;																													\
																																				\
	if(i >= size)																																\
		return;																																	\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < (length*biased_dim) && (i+1)%(dim+1) != 0)																						\
		{																																		\
			if(tab[i] <= (type) 0.0f)																											\
				tab[i] *= (type) leaking_factor;																								\
			else if(tab[i] > (type) saturation)																									\
				tab[i] = (type) saturation + (tab[i] - (type) saturation)*((type)leaking_factor);												\
		}																																		\
		else																																	\
			tab[i] = (type) 0.0f;																												\
	}																																			\
	else																																		\
	{																																			\
		if((i / dim)%offset < length)																											\
		{																																		\
			if(tab[i] <= (type) 0.0f)																											\
				tab[i] *= (type) leaking_factor;																								\
			else if(tab[i] > (type) saturation)																									\
				tab[i] = (type) saturation + (tab[i] - (type) saturation)*((type)leaking_factor);												\
		}																																		\
		else																																	\
			tab[i] = (type) 0.0f;																												\
	}																																			\
}


#define ReLU_deriv_kernel(name, type)																											\
__global__ void ReLU_deriv_kernel_##name(void *i_deriv, void *i_value, int dim, int biased_dim,	int offset,										\
	 float saturation, float leaking_factor, int length, size_t size)																			\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
																																				\
	type* deriv = (type*) i_deriv;																												\
	type* value = (type*) i_value;																												\
																																				\
	if(i >= size)																																\
		return;																																	\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < (length*biased_dim) && (i+1)%(dim+1) != 0)																						\
		{																																		\
			if(value[i] <= (type) 0.0f)																											\
				deriv[i] *= leaking_factor;																										\
			else if(value[i] > (type) saturation)																								\
				deriv[i] *= leaking_factor;																										\
		}																																		\
		else																																	\
			deriv[i] = (type) 0.0f;																												\
	}																																			\
	else																																		\
	{																																			\
		if((i / dim)%offset < length)																											\
		{																																		\
			if(value[i] <= (type) 0.0f)																											\
				deriv[i] *= leaking_factor;																										\
			else if(value[i] > (type) saturation)																								\
				deriv[i] *= leaking_factor;																										\
		}																																		\
		else																																	\
			deriv[i] = (type) 0.0f;																												\
	}																																			\
}


#define quadratic_deriv_output_error_kernel(name, type)																							\
__global__ void quadratic_deriv_output_error_kernel_##name																						\
	(void *i_delta_o, void *i_output, void *i_target, int dim, int biased_dim, int offset, int length, size_t size, float TC_scale_factor)		\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int nb_filters, c_batch, c_filter, in_filter_pos, pos;																						\
																																				\
	type* delta_o = (type*) i_delta_o;																											\
	type* output  = (type*) i_output;																											\
	type* target  = (type*) i_target;																											\
																																				\
	if(i >= size)																																\
		return;																																	\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < (length*biased_dim) && (i+1)%(dim+1) != 0)																						\
		{																																		\
			pos = i - i/(dim+1);																												\
			delta_o[i] = (type)(((float)output[i] - (float)target[pos]) * TC_scale_factor);														\
		}																																		\
		else																																	\
			delta_o[i] = (type) 0.0f;																											\
	}																																			\
	else																																		\
	{																																			\
		if((i / dim)%offset < length)																											\
		{																																		\
			nb_filters = size / (dim*offset);																									\
			c_filter = i / (dim*offset);																										\
			c_batch = (i / dim)%offset;																											\
			in_filter_pos = i % dim;																											\
																																				\
			pos = in_filter_pos + (c_filter + c_batch*nb_filters)*dim;																			\
			delta_o[i] = (type)(((float)output[i] - (float)target[pos]) * TC_scale_factor);														\
		}																																		\
		else																																	\
			delta_o[i] = (type) 0.0f;																											\
	}																																			\
}


#define quadratic_output_error_kernel(name, type)																								\
__global__ void quadratic_output_error_kernel_##name																							\
	(float *output_error, void *i_output, void *i_target, int dim, int biased_dim, int offset, int length, size_t size)							\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int nb_filters, c_batch, c_filter, in_filter_pos, pos;																						\
																																				\
	type* output = (type*) i_output;																											\
	type* target = (type*) i_target;																											\
																																				\
	if(i >= size)																																\
		return;																																	\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < (length*biased_dim) && (i+1)%(dim+1) != 0)																						\
		{																																		\
			pos = i - i/(dim+1);																												\
			output_error[i] = (0.5f*((float)output[i] - (float)target[pos])*((float)output[i] - (float)target[pos]));							\
		}																																		\
		else																																	\
			output_error[i]	= 0.0f;																												\
	}																																			\
	else																																		\
	{																																			\
		if((i / dim)%offset < length)																											\
		{																																		\
			nb_filters = size / (dim*offset);																									\
			c_filter = i / (dim*offset);																										\
			c_batch = (i / dim)%offset;																											\
			in_filter_pos = i % dim;																											\
																																				\
			pos = in_filter_pos + (c_filter + c_batch*nb_filters)*dim;																			\
			output_error[i] = (0.5f*((float)output[i] - (float)target[pos])*((float)output[i] - (float)target[pos]));							\
		}																																		\
		else																																	\
			output_error[i]	= 0.0f;																												\
	}																																			\
}

//#####################################################


//#####################################################
//		  Logistic activation related templates
//#####################################################

#define logistic_activation_kernel(name, type, exp_fct)																							\
__global__ void logistic_activation_kernel_##name(void *i_tab, float beta, float saturation, int dim, 											\
	int biased_dim, int offset, int length, size_t size)																						\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
																																				\
	type* tab = (type*) i_tab;																													\
	float t_one = (type) 1.0f;																													\
	type t_beta = (type) beta;																													\
	type t_saturation = (type) saturation;																										\
																																				\
	if(i >= size)																																\
		return;																																	\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < (length*biased_dim) && (i+1)%(dim+1) != 0)																						\
		{																																		\
			tab[i] = -t_beta*tab[i];																											\
			if(tab[i] > t_saturation)																											\
				tab[i] = t_saturation;																											\
			tab[i] = t_one/(t_one + exp_fct((float)tab[i]));																					\
		}																																		\
		else																																	\
			tab[i] = (type)0.0f;																												\
	}																																			\
	else																																		\
	{																																			\
		if((i / dim)%offset < length)																											\
		{																																		\
			tab[i] = -t_beta*tab[i];																											\
			if(tab[i] > t_saturation)																											\
				tab[i] = t_saturation;																											\
			tab[i] = t_one/(t_one + exp_fct((float)tab[i]));																					\
		}																																		\
		else																																	\
			tab[i] = (type)0.0f;																												\
	}																																			\
}


#define logistic_deriv_kernel(name, type)																										\
__global__ void logistic_deriv_kernel_##name(void *i_deriv, void *i_value, float beta, int dim, 												\
	int biased_dim, int offset, int length, size_t size)																						\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
																																				\
	type* deriv = (type*) i_deriv;																												\
	type* value = (type*) i_value;																												\
																																				\
	if(i >= size)																																\
		return;																																	\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < (length*biased_dim) && (i+1)%(dim+1) != 0)																						\
			deriv[i] *= (type)beta*value[i]*((type)1.0f-value[i]);																				\
		else																																	\
			deriv[i] = (type) 0.0f;																												\
	}																																			\
	else																																		\
	{																																			\
		if((i / dim)%offset < length)																											\
			deriv[i] *= (type)beta*value[i]*((type)1.0f-value[i]);																				\
		else																																	\
			deriv[i] = (type) 0.0f;																												\
	}																																			\
}

//#####################################################


//#####################################################
//		  Soft-Max activation related templates
//#####################################################


#define softmax_activation_kernel(name, type, exp_fct)																							\
__global__ void softmax_activation_kernel_##name(void *i_tab, int dim, int biased_dim, 															\
	int offset, int length, int batch_size, size_t size)																						\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int j, k, l;																																\
	int nb_filters;																																\
	type *pos, *off_pos;																														\
	type vmax;																																	\
	float normal = 0.0f;																														\
	type* tab = (type*) i_tab;																													\
																																				\
	if(i >= batch_size)																															\
		return;																																	\
																																				\
	pos = tab + i*(biased_dim);																													\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < length)																															\
		{																																		\
			vmax = *pos;																														\
			for(j = 0; j < dim; j++)																											\
			{																																	\
				off_pos = pos + j;																												\
				if(*off_pos > vmax)																												\
					vmax = *off_pos;																											\
			}																																	\
																																				\
			for(j = 0; j < dim; j++)																											\
			{																																	\
				off_pos = pos + j;																												\
				*off_pos = exp_fct((float)(*off_pos-vmax));																						\
				normal += (float)*off_pos;																										\
			}																																	\
			pos[dim] = 0.0f;																													\
																																				\
			for(j = 0; j < dim; j++)																											\
			{																																	\
				off_pos = pos + j;																												\
				*off_pos = (type)((float)*off_pos/normal);																						\
			}																																	\
			pos[dim] = 0.0f;																													\
		}																																		\
		else																																	\
		{																																		\
			for(j = 0; j < dim; j++)																											\
			{																																	\
				off_pos = pos + j;																												\
				*off_pos = 0.0f;																												\
			}																																	\
			pos[dim] = 0.0f;																													\
		}																																		\
	}																																			\
	else																																		\
	{																																			\
		nb_filters = size / (dim*batch_size);																									\
		if(i < length)																															\
		{																																		\
			vmax = *pos;																														\
			for(k = 0; k < nb_filters ; k++)																									\
			{																																	\
				for(l = 0; l < dim; l++)																										\
				{																																\
					off_pos = pos + k*dim*batch_size + l;																						\
					if(*off_pos > vmax)																											\
						vmax = *off_pos;																										\
				}																																\
			}																																	\
																																				\
			for(k = 0; k < nb_filters ; k++)																									\
			{																																	\
				for(l = 0; l < dim; l++)																										\
				{																																\
					off_pos = pos + k*dim*batch_size + l;																						\
					*off_pos = exp_fct((float)(*off_pos-vmax));																					\
					normal += (float)*off_pos;																									\
				}																																\
			}																																	\
																																				\
			for(k = 0; k < nb_filters ; k++)																									\
			{																																	\
				for(l = 0; l < dim; l++)																										\
				{																																\
					off_pos = pos + k*dim*batch_size + l;																						\
					*off_pos = (type)((float)*off_pos/normal);																					\
				}																																\
			}																																	\
		}																																		\
		else																																	\
		{																																		\
			for(k = 0; k < nb_filters ; k++)																									\
			{																																	\
				for(l = 0; l < dim; l++)																										\
				{																																\
					off_pos = pos + k*dim*batch_size + l;																						\
					*off_pos = 0.0f;																											\
				}																																\
			}																																	\
		}																																		\
	}																																			\
}


#define cross_entropy_deriv_output_error_kernel(name, type)																						\
__global__ void cross_entropy_deriv_output_error_kernel_##name																					\
	(void *i_delta_o, void *i_output, void *i_target, int dim, int biased_dim, int offset, int length, size_t size, float TC_scale_factor)		\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int nb_filters, c_batch, c_filter, in_filter_pos, pos;																						\
																																				\
	type* delta_o = (type*)i_delta_o;																											\
	type* output  = (type*)i_output;																											\
	type* target  = (type*)i_target;																											\
																																				\
	if(i >= size)																																\
		return;																																	\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < (length*biased_dim) && (i+1)%(dim+1) != 0)																						\
		{																																		\
			pos = i - i/(dim+1);																												\
			delta_o[i] = (type)(((float)output[i] - (float)target[pos])* TC_scale_factor);														\
		}																																		\
		else																																	\
			delta_o[i] = (type) 0.0f;																											\
	}																																			\
	else																																		\
	{																																			\
		if((i / dim)%offset < length)																											\
		{																																		\
			nb_filters = size / (dim*offset);																									\
			c_filter = i / (dim*offset);																										\
			c_batch = (i / dim)%offset;																											\
			in_filter_pos = i % dim;																											\
																																				\
			pos = in_filter_pos + (c_filter + c_batch*nb_filters)*dim;																			\
			delta_o[i] = (type)(((float)output[i] - (float)target[pos])* TC_scale_factor);														\
		}																																		\
		else																																	\
			delta_o[i] = (type) 0.0f;																											\
	}																																			\
}


#define cross_entropy_output_error_kernel(name, type)																							\
__global__ void cross_entropy_output_error_kernel_##name																						\
	(float *output_error, void *i_output, void *i_target, int dim, int biased_dim, int offset, int length, size_t size)							\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int nb_filters, c_batch, c_filter, in_filter_pos, pos;																						\
																																				\
	type* output  = (type*)i_output;																											\
	type* target  = (type*)i_target;																											\
																																				\
	if(i >= size)																																\
		return;																																	\
																																				\
	if(biased_dim > dim)																														\
	{																																			\
		if(i < (length*biased_dim) && (i+1)%(dim+1) != 0)																						\
		{																																		\
			pos = i - i/(dim+1);																												\
			if((float)output[i] > 0.000001f)																									\
				output_error[i] = -(float)target[pos] * logf((float)output[i]);																	\
			else																																\
				output_error[i] = -(float)target[pos] * logf((float)0.000001f);																	\
		}																																		\
		else																																	\
			output_error[i] = 0.0f;																												\
	}																																			\
	else																																		\
	{																																			\
		if((i / dim)%offset < length)																											\
		{																																		\
			nb_filters = size / (dim*offset);																									\
			c_filter = i / (dim*offset);																										\
			c_batch = (i / dim)%offset;																											\
			in_filter_pos = i % dim;																											\
																																				\
			pos = in_filter_pos + (c_filter + c_batch*nb_filters)*dim;																			\
			if((float)output[i] > 0.000001f)																									\
				output_error[i] = -(float)target[pos] * logf((float)output[i]);																	\
			else																																\
				output_error[i] = -(float)target[pos] * logf((float)0.000001f);																	\
		}																																		\
		else																																	\
			output_error[i] = 0.0f;																												\
	}																																			\
}


//#####################################################
//Exp activation (GAN discriminator) related templates
//#####################################################

/*Only Dense layer and just a copy of softmax activation for know*/
#define exp_disc_activation_kernel(name, type, exp_fct)																							\
__global__ void exp_disc_activation_kernel_##name(void *i_tab, int dim, int biased_dim, 														\
	int offset, int length, int batch_size, size_t size, int halved, int revert)																\
{																																				\
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int j;																																		\
	type *pos, *off_pos;																														\
	type vmax;																																	\
	float normal = 0.0f;																														\
	type* tab = (type*) i_tab;																													\
																																				\
	if(i >= batch_size)																															\
		return;																																	\
																																				\
	pos = tab + i*(biased_dim);																													\
																																				\
	if(i < length)																																\
	{																																			\
		vmax = *pos;																															\
		for(j = 0; j < dim; j++)																												\
		{																																		\
			off_pos = pos + j;																													\
			if(*off_pos > vmax)																													\
				vmax = *off_pos;																												\
		}																																		\
																																				\
		for(j = 0; j < dim; j++)																												\
		{																																		\
			off_pos = pos + j;																													\
			*off_pos = exp_fct((float)(*off_pos-vmax));																							\
			normal += (float)*off_pos;																											\
		}																																		\
		pos[dim] = 0.0f;																														\
																																				\
		for(j = 0; j < dim; j++)																												\
		{																																		\
			off_pos = pos + j;																													\
			*off_pos = (type)((float)*off_pos/normal);																							\
		}																																		\
		pos[dim] = 0.0f;																														\
	}																																			\
	else																																		\
	{																																			\
		for(j = 0; j < dim; j++)																												\
		{																																		\
			off_pos = pos + j;																													\
			*off_pos = 0.0f;																													\
		}																																		\
		pos[dim] = 0.0f;																														\
	}																																			\
}



#define exp_disc_deriv_output_kernel(name, type, exp_fct)																						\
__global__ void exp_disc_deriv_output_kernel_##name																								\
(void *i_delta_o, void *i_output, void *i_target, int dim, int biased_dim, 																		\
	int offset, int length, int batch_size, size_t size, float TC_scale_factor, int halved, int revert)											\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int j, k, pos;																																\
																																				\
	type* delta_o = (type*)i_delta_o;																											\
	type* output  = (type*)i_output;																											\
	/*type* target  = (type*)i_target;*/																										\
																																				\
	float vmax = 0.0f, vmin = 1.0f;																												\
																																				\
	pos = i*biased_dim;																															\
																																				\
	if(i >= batch_size)																															\
		return;																																	\
																																				\
	if(i < length)																																\
	{																																			\
		for(k = 0; k < length; k++)																												\
		{																																		\
			if((float)output[k*biased_dim+1] > vmax)																							\
				vmax = (float)output[k*biased_dim+1];																							\
			if((float)output[k*biased_dim+1] < vmin)																							\
				vmin = (float)output[k*biased_dim+1];																							\
		}																																		\
																																				\
		if(revert)																																\
		{																																		\
			delta_o[pos+0] = (type) (0.0f);																										\
			for(j = 1; j < dim; j++)																											\
			{																																	\
				delta_o[pos+j] = (type) ((((float)output[pos+j] - vmax)/(vmax-vmin))*TC_scale_factor);											\
			}																																	\
			delta_o[pos+dim] = (type) 0.0f;																										\
		}																																		\
		else																																	\
		{																																		\
			if(halved && i < batch_size/2)																										\
			{																																	\
				delta_o[pos+0] = (type) (0.0f);																									\
				for(j = 1; j < dim; j++)																										\
					delta_o[pos+j] = (type) (((float)output[pos+j] - 0.0f)*TC_scale_factor);													\
				delta_o[pos+dim] = (type) 0.0f;																									\
			}																																	\
			else																																\
			{																																	\
				delta_o[pos+0] = (type) (0.0f);																									\
				for(j = 1; j < dim; j++)																										\
					delta_o[pos+j] = (type) (((float)output[pos+j] - 1.0f)*TC_scale_factor);													\
				delta_o[pos+dim] = (type) 0.0f;																									\
			}																																	\
		}																																		\
	}																																			\
	else																																		\
	{																																			\
		for(j = 0; j < dim; j++)																												\
			delta_o[pos+j] = (type) 0.0f;																										\
		delta_o[pos+dim] = (type) 0.0f;																											\
	}																																			\
}



#define old_exp_disc_deriv_output_kernel(name, type, exp_fct)																					\
__global__ void old_exp_disc_deriv_output_kernel_##name																							\
(void *i_delta_o, void *i_output, void *i_target, int dim, int biased_dim, 																		\
	int offset, int length, int batch_size, size_t size, float TC_scale_factor, int halved, int revert)											\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int j, pos;																																	\
																																				\
	type* delta_o = (type*)i_delta_o;																											\
	type* output  = (type*)i_output;																											\
	type* target  = (type*)i_target;																											\
																																				\
	float vmax = 0.0f;																															\
	int arg_max = 0;																															\
																																				\
	pos = i*biased_dim;																															\
																																				\
	if(i >= batch_size)																															\
		return;																																	\
																																				\
	if(i < length)																																\
	{																																			\
		vmax = (float)output[pos+1];																											\
		for(j = 1; j < dim; j++)																												\
		{																																		\
			if((float)output[pos+j] > vmax)																										\
			{																																	\
				vmax = (float)output[pos+j];																									\
				arg_max = j;																													\
			}																																	\
		}																																		\
																																				\
		if(revert)																																\
		{																																		\
			delta_o[pos+0] = (type) (((float)(((float)output[pos+0])) - 0.0f)																	\
				*((float)output[pos+0]+0.0f)*(1.0f-(float)output[pos+0])*TC_scale_factor);														\
			for(j = 1; j < dim; j++)																											\
				delta_o[pos+j] = (type) (((float)(((float)output[pos+j])) - 1.0f)																\
					*((float)output[pos+j]+0.0f)*(1.0f-(float)output[pos+j])*TC_scale_factor);													\
			delta_o[pos+dim] = (type) 0.0f;																										\
		}																																		\
		else																																	\
		{																																		\
			if(halved && i < batch_size/2)																										\
			{																																	\
				delta_o[pos+0] = (type) (((float)(((float)output[pos+0])) - 1.0f)																\
					*((float)output[pos+0]+0.0f)*(1.0f-(float)output[pos+0])*TC_scale_factor);													\
				for(j = 1; j < dim; j++)																										\
					delta_o[pos+j] = (type) (((float)(((float)output[pos+j])) - 0.0f)															\
						*((float)output[pos+j]+0.0f)*(1.0f-(float)output[pos+j])*TC_scale_factor);												\
				delta_o[pos+dim] = (type) 0.0f;																									\
			}																																	\
			else																																\
			{																																	\
				delta_o[pos+0] = (type) (((float)(((float)output[pos+0])) - 0.0f)																\
					*((float)output[pos+0]+0.0f)*(1.0f-(float)output[pos+0])*TC_scale_factor);													\
				for(j = 1; j < dim; j++)																										\
					delta_o[pos+j] = (type) (((float)(((float)output[pos+j])) - 1.0f)															\
						*((float)output[pos+j]+0.0f)*(1.0f-(float)output[pos+j])*TC_scale_factor);												\
				delta_o[pos+dim] = (type) 0.0f;																									\
			}																																	\
		}																																		\
	}																																			\
	else																																		\
	{																																			\
		for(j = 0; j < dim; j++)																												\
			delta_o[pos+j] = (type) 0.0f;																										\
		delta_o[pos+dim] = (type) 0.0f;																											\
	}																																			\
}


/* lack weight clipping or batch notrm to work properly*/
#define w_gan_exp_disc_deriv_output_kernel(name, type, exp_fct)																					\
__global__ void w_gan_exp_disc_deriv_output_kernel_##name																						\
(void *i_delta_o, void *i_output, void *i_target, int dim, int biased_dim, 																		\
	int offset, int length, int batch_size, size_t size, float TC_scale_factor, int halved, int revert)											\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int j, pos;																																	\
																																				\
	type* delta_o = (type*)i_delta_o;																											\
	type* output  = (type*)i_output;																											\
	type* target  = (type*)i_target;																											\
																																				\
	float avg_real = 0.0f, avg_fake = 0.0f;																										\
	int arg_max = 0;																															\
																																				\
	pos = i*biased_dim;																															\
																																				\
	if(i >= batch_size)																															\
		return;																																	\
																																				\
	if(i < length)																																\
	{																																			\
		if(revert)																																\
		{																																		\
			for(j = 0; j < batch_size; j++)																										\
				avg_fake += (float)output[j*biased_dim];																						\
			avg_fake /= batch_size;																												\
			delta_o[pos+0] = (type) 0.0f;																										\
			for(j = 1; j < dim; j++)																											\
				delta_o[pos+j] = (type) (avg_fake*TC_scale_factor);																				\
			delta_o[pos+dim] = (type) 0.0f;																										\
		}																																		\
		else																																	\
		{																																		\
			if(halved && i < batch_size/2)																										\
			{																																	\
				for(j = 0; j < batch_size/2; j++)																								\
					avg_fake += (float)output[j*biased_dim];																					\
				avg_fake /= batch_size/2;																										\
				delta_o[pos+0] = (type) 0.0f;																									\
				for(j = 1; j < dim; j++)																										\
					delta_o[pos+j] = (type) (-avg_fake*TC_scale_factor);																		\
				delta_o[pos+dim] = (type) 0.0f;																									\
			}																																	\
			else																																\
			{																																	\
				for(j = batch_size/2; j < batch_size; j++)																						\
					avg_real += (float)output[j*biased_dim];																					\
				avg_real /= batch_size/2;																										\
				delta_o[pos+0] = (type) 0.0f;																									\
				for(j = 1; j < dim; j++)																										\
					delta_o[pos+j] = (type) (avg_real*TC_scale_factor);																			\
				delta_o[pos+dim] = (type) 0.0f;																									\
			}																																	\
		}																																		\
	}																																			\
	else																																		\
	{																																			\
		for(j = 0; j < dim; j++)																												\
			delta_o[pos+j] = (type) 0.0f;																										\
		delta_o[pos+dim] = (type) 0.0f;																											\
	}																																			\
}


//#####################################################
//		  YOLO activation related templates
//#####################################################

#define YOLO_activation_kernel(name, type, exp_fct)																								\
__global__ void YOLO_activation_kernel_##name(void *i_tab, int flat_offset, size_t len, yolo_param y_param, size_t size, int class_softmax)		\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	if(i >= size)																																\
		return;																																	\
																																				\
	type *tab = (type*) i_tab;																													\
																																				\
	int nb_class = y_param.nb_class, nb_param = y_param.nb_param;																				\
	/*Default values are in activ_function.c (set_yolo_params)*/																				\
	float **sm_tab = y_param.slopes_and_maxes_tab;																								\
	float normal = 0.0f;																														\
	type vmax;																																	\
	int fit_dim = y_param.fit_dim;																												\
	int col, in_col, j;																															\
																																				\
	col = i / flat_offset;																														\
	in_col = col%(8+nb_class+nb_param);																											\
																																				\
	/*Position*/																																\
	if(in_col >= 0 && in_col < 3)																												\
	{																																			\
		if(fit_dim > in_col)																													\
		{																																		\
			tab[i] = -(type)sm_tab[0][0]*tab[i];																								\
			if(tab[i] > (type)sm_tab[0][1])																										\
				tab[i] = (type)sm_tab[0][1];																									\
			else if(tab[i] < (type)sm_tab[0][2])																								\
				tab[i] = (type)sm_tab[0][2];																									\
			tab[i] = 1.0f/(1.0f + exp_fct(tab[i]));																								\
		}																																		\
		else																																	\
			tab[i] = 0.5f; /*Center of the cell*/																								\
		return;																																	\
	}																																			\
																																				\
	/*Box size*/																																\
	if(in_col >= 3 && in_col < 6)																												\
	{																																			\
		if(fit_dim > in_col-3)																													\
		{																																		\
			tab[i] = (type)sm_tab[1][0]*tab[i];																									\
			if(tab[i] > (type)sm_tab[1][1])																										\
				tab[i] = (type)sm_tab[1][1];																									\
			else if(tab[i] < (type)(sm_tab[1][2]))																								\
				tab[i] = (sm_tab[1][2]);																										\
		}																																		\
		else																																	\
			tab[i] = 0.0f; /*Output = prior*/																									\
		return;																																	\
	}																																			\
																																				\
	/*Object probability*/																														\
	if(in_col == 6)																																\
	{																																			\
		tab[i] = -(type)sm_tab[2][0]*tab[i];																									\
		if(tab[i] > (type)sm_tab[2][1])																											\
			tab[i] = (type)sm_tab[2][1];																										\
		else if(tab[i] < (type)sm_tab[2][2])																									\
			tab[i] = (type)sm_tab[2][2];																										\
		tab[i] = 1.0f/(1.0f + exp_fct(tab[i]));																									\
		return;																																	\
	}																																			\
																																				\
	/*Objectness (Obj. quality => based on IoU)*/																								\
	if(in_col == 7)																																\
	{																																			\
		tab[i] = -(type)sm_tab[3][0]*tab[i];																									\
		if(tab[i] > (type)sm_tab[3][1])																											\
			tab[i] = (type)sm_tab[3][1];																										\
		else if(tab[i] < (type)sm_tab[3][2])																									\
			tab[i] = (type)sm_tab[3][2];																										\
		tab[i] = 1.0f/(1.0f + exp_fct(tab[i]));																									\
		return;																																	\
	}																																			\
																																				\
	/*Classes*/																																	\
	if(in_col >= 8 && in_col < 8+nb_class)																										\
	{																																			\
		if(class_softmax)																														\
		{																																		\
			if(in_col != 8)																														\
				return;																															\
			vmax = tab[i];																														\
			for(j = 1; j < nb_class; j++)																										\
				if(tab[i+j*flat_offset] > vmax)																									\
					vmax = tab[i+j*flat_offset];																								\
																																				\
			for(j = 0; j < nb_class; j++)																										\
			{																																	\
				tab[i+j*flat_offset] = exp_fct((tab[i+j*flat_offset]-vmax));																	\
				normal += (float)tab[i+j*flat_offset];																							\
			}																																	\
																																				\
			for(j = 0; j < nb_class; j++)																										\
				tab[i+j*flat_offset] = (type)((float)tab[i+j*flat_offset]/normal);																\
		}																																		\
		else																																	\
		{																																		\
			tab[i] = -(type)sm_tab[4][0]*tab[i];																								\
			if(tab[i] > (type)sm_tab[4][1])																										\
				tab[i] = (type)sm_tab[4][1];																									\
			else if(tab[i] < (type)sm_tab[4][2])																								\
				tab[i] = (type)sm_tab[4][2];																									\
			tab[i] = 1.0f/(1.0f + exp_fct(tab[i]));																								\
		}																																		\
		return;																																	\
	}																																			\
																																				\
	/*Additional parameters (regression)*/																										\
	if(in_col >= 8+nb_class)																													\
	{																																			\
		tab[i] = (type)sm_tab[5][0]*tab[i];																										\
		if(tab[i] > (type)sm_tab[5][1])																											\
			tab[i] = (type)sm_tab[5][1];																										\
		else if(tab[i] < (type)(sm_tab[5][2]))																									\
			tab[i] = (sm_tab[5][2]);																											\
		return;																																	\
	}																																			\
}

__device__ float gpu_IoU_fct(float *output, float *target)
{
	float inter_w, inter_h, inter_d, inter_3d, uni_3d;
	
	inter_w = max(0.0f, min(output[3], target[3]) - max(output[0], target[0]));
	inter_h = max(0.0f, min(output[4], target[4]) - max(output[1], target[1]));
	inter_d = max(0.0f, min(output[5], target[5]) - max(output[2], target[2]));
	
	inter_3d = inter_w * inter_h * inter_d;
	uni_3d =  abs(output[3]-output[0])*abs(output[4]-output[1])*abs(output[5]-output[2])
			+ abs(target[3]-target[0])*abs(target[4]-target[1])*abs(target[5]-target[2])
			- inter_3d;
	
	return ((float)inter_3d)/(float)uni_3d;
}


__device__ float gpu_GIoU_fct(float *output, float *target)
{
	float inter_w, inter_h, inter_d, inter_3d, uni_3d, enclose_3d, enclose_w, enclose_h, enclose_d;
	
	inter_w = max(0.0f, min(output[3], target[3]) - max(output[0], target[0]));
	inter_h = max(0.0f, min(output[4], target[4]) - max(output[1], target[1]));
	inter_d = max(0.0f, min(output[5], target[5]) - max(output[2], target[2]));
	
	inter_3d = inter_w * inter_h * inter_d;
	uni_3d =  abs(output[3]-output[0])*abs(output[4]-output[1])*abs(output[5]-output[2])
			+ abs(target[3]-target[0])*abs(target[4]-target[1])*abs(target[5]-target[2])
			- inter_3d;
	enclose_w = (max(output[3], target[3]) - min(output[0], target[0]));
	enclose_h = (max(output[4], target[4]) - min(output[1], target[1]));
	enclose_d = (max(output[5], target[5]) - min(output[2], target[2]));
	enclose_3d = enclose_w * enclose_h * enclose_d;
	
	return (((float)inter_3d)/(float)uni_3d - (float)(enclose_3d - uni_3d)/(float)enclose_3d);
}

//order: xmin, ymin, zmin, xmax, ymax, zmax
// Take into acount the distance in IoU, useful for crowded images
// or to put the emhasis on positionning in objectness score
__device__ float gpu_DIoU_fct(float *output, float *target)
{
	float inter_w, inter_h, inter_d, inter_3d, uni_3d, enclose_w, enclose_h, enclose_d;
	float cx_a, cx_b, cy_a, cy_b, cz_a, cz_b, dist_cent, diag_enclose;
	
	inter_w = max(0.0f, min(output[3], target[3]) - max(output[0], target[0]));
	inter_h = max(0.0f, min(output[4], target[4]) - max(output[1], target[1]));
	inter_d = max(0.0f, min(output[5], target[5]) - max(output[2], target[2]));
	
	inter_3d = inter_w * inter_h * inter_d;
	uni_3d =  abs(output[3]-output[0])*abs(output[4]-output[1])*abs(output[5]-output[2])
			+ abs(target[3]-target[0])*abs(target[4]-target[1])*abs(target[5]-target[2])
			- inter_3d;
	enclose_w = (max(output[3], target[3]) - min(output[0], target[0]));
	enclose_h = (max(output[4], target[4]) - min(output[1], target[1]));
	enclose_d = (max(output[5], target[5]) - min(output[2], target[2]));
	
	cx_a = (output[3] + output[0])*0.5; cx_b = (target[3] + target[0])*0.5; 
	cy_a = (output[4] + output[1])*0.5; cy_b = (target[4] + target[1])*0.5;
	cz_a = (output[5] + output[2])*0.5; cz_b = (target[5] + target[2])*0.5;
	dist_cent = sqrt((cx_a - cx_b)*(cx_a - cx_b) + (cy_a - cy_b)*(cy_a - cy_b) + (cz_a - cz_b)*(cz_a - cz_b));
	diag_enclose = sqrt(enclose_w*enclose_w + enclose_h*enclose_h + enclose_d*enclose_d);
	
	return ((float)inter_3d)/(float)uni_3d - (float)(dist_cent/diag_enclose);
}

// Distance penality is less in this version for a given distance between boxes
// More suited for usual VOC images, or sparse astro images
__device__ float gpu_DIoU2_fct(float *output, float *target)
{
	float inter_w, inter_h, inter_d, inter_3d, uni_3d, enclose_w, enclose_h, enclose_d;
	float cx_a, cx_b, cy_a, cy_b, cz_a, cz_b, dist_cent, diag_enclose;
	
	inter_w = max(0.0f, min(output[3], target[3]) - max(output[0], target[0]));
	inter_h = max(0.0f, min(output[4], target[4]) - max(output[1], target[1]));
	inter_d = max(0.0f, min(output[5], target[5]) - max(output[2], target[2]));
	
	inter_3d = inter_w * inter_h * inter_d;
	uni_3d =  abs(output[3]-output[0])*abs(output[4]-output[1])*abs(output[5]-output[2])
			+ abs(target[3]-target[0])*abs(target[4]-target[1])*abs(target[5]-target[2])
			- inter_3d;
	enclose_w = (max(output[3], target[3]) - min(output[0], target[0]));
	enclose_h = (max(output[4], target[4]) - min(output[1], target[1]));
	enclose_d = (max(output[5], target[5]) - min(output[2], target[2]));
	
	cx_a = (output[3] + output[0])*0.5; cx_b = (target[3] + target[0])*0.5; 
	cy_a = (output[4] + output[1])*0.5; cy_b = (target[4] + target[1])*0.5;
	cz_a = (output[5] + output[2])*0.5; cz_b = (target[5] + target[2])*0.5;
	dist_cent = ((cx_a - cx_b)*(cx_a - cx_b) + (cy_a - cy_b)*(cy_a - cy_b) + (cz_a - cz_b)*(cz_a - cz_b));
	diag_enclose = (enclose_w*enclose_w + enclose_h*enclose_h + enclose_d*enclose_d);
	
	return ((float)inter_3d)/(float)uni_3d - (float)(dist_cent/diag_enclose);
}


typedef float(*pointFunction_gpu_IoU)(float*, float*); 
__device__ pointFunction_gpu_IoU device_gpu_IoU_fct  = gpu_IoU_fct;
__device__ pointFunction_gpu_IoU device_gpu_GIoU_fct = gpu_GIoU_fct;
__device__ pointFunction_gpu_IoU device_gpu_DIoU_fct = gpu_DIoU_fct;
__device__ pointFunction_gpu_IoU device_gpu_DIoU2_fct = gpu_DIoU2_fct;


#define YOLO_deriv_error_kernel(name, type)																										\
__global__ void YOLO_deriv_error_kernel_##name																									\
	(void *i_delta_o, void *i_output, void *i_target, int flat_target_size, int flat_output_size, 												\
	int nb_area_w, int nb_area_h, int nb_area_d, yolo_param y_param, size_t size, float TC_scale_factor, int nb_im_iter)						\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	if(i >= size)																																\
		return;																																	\
																																				\
	type *delta_o = (type*) i_delta_o;																											\
	type *output  = (type*) i_output;																											\
	type *target  = (type*) i_target;																											\
																																				\
	/* Define many "shorts" for y_param content to enhance code redeability*/																	\
	int nb_box = y_param.nb_box, nb_class = y_param.nb_class, nb_param = y_param.nb_param; 														\
	int strict_box_size_association = y_param.strict_box_size_association;																		\
	int fit_dim = y_param.fit_dim, rand_startup = y_param.rand_startup;																			\
	float rand_prob_best_box_assoc = y_param.rand_prob_best_box_assoc;																			\
	float rand_prob = y_param.rand_prob;																										\
	float min_prior_forced_scaling = y_param.min_prior_forced_scaling;																			\
	int class_softmax = y_param.class_softmax, diff_flag = y_param.diff_flag;																	\
	int prior_dist_type = y_param.prior_dist_type;																								\
	void *block_state = y_param.block_state;																									\
																																				\
	float coord_scale = y_param.scale_tab[0], size_scale  = y_param.scale_tab[1];																\
	float prob_scale  = y_param.scale_tab[2], obj_scale   = y_param.scale_tab[3];																\
	float class_scale = y_param.scale_tab[4], param_scale = y_param.scale_tab[5];																\
																																				\
	float *prior_size         = y_param.prior_size;																								\
	int   *cell_size          = y_param.cell_size;																								\
	float *param_ind_scale    = y_param.param_ind_scale;																						\
	float *lambda_noobj_prior = y_param.noobj_prob_prior;																						\
	float **sm_tab            = y_param.slopes_and_maxes_tab;																					\
	int   *target_cell_mask   = y_param.target_cell_mask;																						\
	float *IoU_table          = y_param.IoU_table;																								\
	float *dist_prior         = y_param.dist_prior;																								\
	int   *box_locked         = y_param.box_locked;																								\
	float *box_in_pix         = y_param.box_in_pix;																								\
																																				\
	float size_max_sat = expf(sm_tab[1][1]), size_min_sat = expf(sm_tab[1][2]);																	\
	float good_IoU_lim      = y_param.IoU_limits[0], low_IoU_best_box_assoc = y_param.IoU_limits[1];											\
	float min_prob_IoU_lim  = y_param.IoU_limits[2], min_obj_IoU_lim        = y_param.IoU_limits[3];											\
	float min_class_IoU_lim = y_param.IoU_limits[4], min_param_IoU_lim      = y_param.IoU_limits[5];											\
	float diff_IoU_lim      = y_param.IoU_limits[6], diff_obj_lim           = y_param.IoU_limits[7];											\
	int fit_pos = y_param.fit_parts[0], fit_size  = y_param.fit_parts[1], fit_prob  = y_param.fit_parts[2]; 									\
	int fit_obj = y_param.fit_parts[3], fit_class = y_param.fit_parts[4], fit_param = y_param.fit_parts[5];										\
																																				\
	int j, k, l, l_o, l_t;																														\
	int c_batch, f_offset, best_prior_id, nb_obj_target, s_p_i = 0;																				\
	int nb_in_cell, id_in_cell, l_r_b = -1, resp_box = -1, resp_targ = -1, targ_diff_flag = 0;													\
	float best_dist, c_dist, max_IoU, current_IoU;																								\
	int cell_pos[3], c_nb_area[3], obj_c[3];																									\
	float *c_box_in_pix, *c_prior_size;																											\
	float obj_in_offset[6], out_int[6], targ_int[6], targ_size[3];																				\
	float class_only_IoU = -2.0f;																												\
																																				\
	c_nb_area[0] = nb_area_w; c_nb_area[1] = nb_area_h; c_nb_area[2] = nb_area_d;																\
	c_batch = i / flat_output_size;																												\
	target += flat_target_size * c_batch;																										\
	f_offset = size;																															\
																																				\
	i = i % flat_output_size;																													\
	cell_pos[2] = i / (c_nb_area[0]*c_nb_area[1]);																								\
	cell_pos[1] = (int)(i % (c_nb_area[0]*c_nb_area[1])) / c_nb_area[0];																		\
	cell_pos[0] = (int)(i % (c_nb_area[0]*c_nb_area[1])) % c_nb_area[0];																		\
																																				\
	delta_o += (c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) 																						\
		* c_batch + cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0];												\
	output  += (c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) 																						\
		* c_batch + cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0];												\
																																				\
	target_cell_mask +=	((c_nb_area[0]*c_nb_area[1]*c_nb_area[2])*c_batch * y_param.max_nb_obj_per_image);										\
	target_cell_mask +=	(cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * y_param.max_nb_obj_per_image;		\
																																				\
	/*Could redume memory footprint with a max_nb_obj_per_cell parameter*/																		\
	IoU_table += ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2])*c_batch * y_param.max_nb_obj_per_image * nb_box);									\
	IoU_table += (cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * y_param.max_nb_obj_per_image * nb_box;		\
																																				\
	dist_prior += ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2])*c_batch * y_param.max_nb_obj_per_image * nb_box);									\
	dist_prior += (cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * y_param.max_nb_obj_per_image * nb_box;		\
																																				\
	box_locked += ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) * c_batch * nb_box);																\
	box_locked += (cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * nb_box;									\
																																				\
	box_in_pix += ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) * c_batch * 6 * nb_box);															\
	box_in_pix += (cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * 6 * nb_box;								\
																																				\
	nb_obj_target = target[0];																													\
	target++;																																	\
																																				\
	if(nb_obj_target == -1)																														\
	{																																			\
		nb_obj_target = 1;																														\
		class_only_IoU = good_IoU_lim; 																											\
	}																																			\
																																				\
	best_dist = 100000000;																														\
	for(k = 0; k < nb_box; k++)																													\
	{																																			\
		box_locked[k] = 0;																														\
		c_box_in_pix = box_in_pix + k*6;																										\
		c_prior_size = prior_size + k*3;																										\
		l_o = k*(8+nb_class+nb_param);																											\
		for(l = 0; l < 3; l++)																													\
			c_box_in_pix[l] = ((float)output[(l_o+l)*f_offset] + cell_pos[l]) * cell_size[l];													\
		for(l = 0; l < 3; l++)																													\
			c_box_in_pix[l+3] = c_prior_size[l]*expf((float)output[(l_o+l+3)*f_offset]);														\
																																				\
		/* Min prior best association could be improved by using the "fit_dim" parameter to avoid definition issues with unused dimensions */	\
		/* This would allow to verify if each used dimension is smaller, rather that using a surface criteria (later in this function) */		\
		c_dist = sqrt(c_prior_size[0]*c_prior_size[0] 																							\
					+ c_prior_size[1]*c_prior_size[1]																							\
					+ c_prior_size[2]*c_prior_size[2]);																							\
		if(c_dist < best_dist)																													\
		{																																		\
			best_dist = c_dist;																													\
			s_p_i = k;																															\
		}																																		\
	}																																			\
																																				\
	nb_in_cell = 0;																																\
	for(j = 0; j < nb_obj_target; j++)																											\
	{																																			\
		l_t = j*(7+nb_param+diff_flag);																											\
		for(l = 0; l < 6; l++)																													\
			targ_int[l] = target[l_t+1+l];																										\
																																				\
		/* Search for targets that should be predicted by the current cell element */															\
		target_cell_mask[j] = 1;																												\
		for(l = 0; l < 3; l++)																													\
		{																																		\
			obj_c[l] = (int)( ((float)target[l_t+l+4] + (float)target[l_t+l+1])*0.5f / cell_size[l]);											\
			/* If target outside the current cell element, set target flag to 0*/																\
			if(obj_c[l] != cell_pos[l])																											\
				target_cell_mask[j] = 0;																										\
		}																																		\
																																				\
		if(target_cell_mask[j] == 1)																											\
			nb_in_cell++;																														\
																																				\
		/* Flag all the "Good but not best boxes" for all targets regardless of the grid element */												\
		for(k = 0; k < nb_box; k++)																												\
		{																																		\
			if(box_locked[k] != 0)																												\
				continue;																														\
			c_box_in_pix = box_in_pix+k*6;																										\
			for(l = 0; l < 6; l++)																												\
				out_int[l] = c_box_in_pix[l%3] + copysignf(0.5f,l-2.5f)*c_box_in_pix[3+l%3];													\
																																				\
			current_IoU = y_param.c_IoU_fct(out_int, targ_int);																					\
			if(current_IoU > good_IoU_lim)																										\
				box_locked[k] = 1;																												\
		}																																		\
	}																																			\
																																				\
	/* For all targets in cell compute the IoU with the predictions and distances to the priors */												\
	id_in_cell = 0;																																\
	for(j = 0; j < nb_obj_target; j++)																											\
	{																																			\
		if(target_cell_mask[j] == 0)																											\
			continue;																															\
																																				\
		l_t = j*(7+nb_param+diff_flag);																											\
		for(l = 0; l < 6; l++)																													\
			targ_int[l] = target[l_t+1+l];																										\
		for(l = 0; l < 3; l++)																													\
			targ_size[l] = targ_int[l+3] - targ_int[l];																							\
																																				\
		for(k = 0; k < nb_box; k++)																												\
		{																																		\
			c_box_in_pix = box_in_pix+k*6;																										\
			for(l = 0; l < 6; l++)																												\
				out_int[l] = c_box_in_pix[l%3] + copysignf(0.5f,l-2.5f)*c_box_in_pix[3+l%3];													\
																																				\
			current_IoU = y_param.c_IoU_fct(out_int, targ_int);																					\
			IoU_table[id_in_cell*nb_box + k] = current_IoU;																						\
			dist_prior[id_in_cell*nb_box + k] = -2.0f;																							\
		}																																		\
																																				\
		/* Restrict the association to the l best theoritical prior (times repetition of identical priors) */									\
		if(strict_box_size_association > 0)																										\
		{																																		\
			if(prior_dist_type == DIST_IOU)																										\
				for(l = 0; l < 6; l++)																											\
					targ_int[l] = copysignf(0.5f,l-2.5f)*targ_size[l%3];																		\
																																				\
			for(k = 0; k < nb_box; k++)																											\
			{																																	\
				c_prior_size = prior_size + k*3;																								\
				switch(prior_dist_type)																											\
				{																																\
					case DIST_IOU:																												\
						for(l = 0; l < 6; l++)																									\
							out_int[l] = copysignf(0.5f,l-2.5f)*c_prior_size[l%3];																\
						dist_prior[id_in_cell*nb_box + k] = 1.0f - y_param.c_IoU_fct(out_int, targ_int);										\
						break;																													\
																																				\
					default:																													\
					case DIST_SIZE:																												\
						dist_prior[id_in_cell*nb_box + k] = sqrt(																				\
							 (targ_size[0]-c_prior_size[0])*(targ_size[0]-c_prior_size[0])														\
							+(targ_size[1]-c_prior_size[1])*(targ_size[1]-c_prior_size[1])														\
							+(targ_size[2]-c_prior_size[2])*(targ_size[2]-c_prior_size[2]));													\
						break;																													\
																																				\
					case DIST_OFFSET:																											\
						for(l = 0; l < 3; l++)																									\
						{																														\
							obj_in_offset[l+3] = targ_size[l]/c_prior_size[l];																	\
							if(obj_in_offset[l+3] < size_min_sat)																				\
								obj_in_offset[l+3] = logf(size_min_sat);																		\
							else if(obj_in_offset[l+3] > size_max_sat)																			\
								obj_in_offset[l+3] = logf(size_max_sat);																		\
							else																												\
								obj_in_offset[l+3] = logf(obj_in_offset[l+3]);																	\
						}																														\
																																				\
						dist_prior[id_in_cell*nb_box + k] = 																					\
							 fabsf(obj_in_offset[3])																							\
							+fabsf(obj_in_offset[4])																							\
							+fabsf(obj_in_offset[5]);																							\
						break;																													\
				}																																\
			}																																	\
																																				\
			for(l = 0; l < strict_box_size_association; l++)																					\
			{																																	\
				best_dist = 1000000.0f;	best_prior_id = -1;																						\
				for(k = 0; k < nb_box; k++)																										\
					if(dist_prior[id_in_cell*nb_box+k] > 0.0 && dist_prior[id_in_cell*nb_box+k] < best_dist)									\
					{																															\
						best_dist = dist_prior[id_in_cell*nb_box+k];																			\
						best_prior_id = k;																										\
					}																															\
				for(k = 0; k < nb_box; k++) /* Flag the closest theoritical prior (and identical ones if any) */								\
				{																																\
					c_prior_size = prior_size + k*3;																							\
					if(prior_size[best_prior_id*3+0] == c_prior_size[0] 																		\
						&& prior_size[best_prior_id*3+1] == c_prior_size[1] 																	\
						&& prior_size[best_prior_id*3+2] == c_prior_size[2])																	\
						dist_prior[id_in_cell*nb_box+k] = -2.0f;																				\
				}																																\
			}																																	\
		}																																		\
																																				\
		id_in_cell++;																															\
	}																																			\
																																				\
	for(id_in_cell = 0; id_in_cell < nb_in_cell; id_in_cell++)																					\
	{																																			\
		/* Force a random box association with only criteria being that the box is not already used */											\
		/* Used as a startup phase to get all the priors closer to the objects to detect */														\
		if(nb_im_iter <= rand_startup)																											\
		{																																		\
			resp_targ = id_in_cell;	resp_box = -1;																								\
			for(k = 0; k < 2*nb_box; k++)																										\
			{																																	\
				resp_box = (int)(hiprand_uniform(&(((hiprandState_t*)block_state)[i]))*nb_box);													\
				if(box_locked[resp_box] != 2)																									\
					break;																														\
				resp_box = -1;																													\
			}																																	\
																																				\
			if(resp_box == -1)																													\
				continue;																														\
																																				\
			l = 0;																																\
			for(j = 0; j < nb_obj_target; j++)																									\
			{																																	\
				l += target_cell_mask[j];																										\
				if(l == resp_targ + 1)																											\
					break;																														\
			}																																	\
			l_t = j*(7+nb_param+diff_flag);																										\
		}																																		\
		else																																	\
		{																																		\
			max_IoU = -2.0f; resp_box = -1;	resp_targ = -1;																						\
			for(j = 0; j < nb_in_cell; j++)																										\
				for(k = 0; k < nb_box; k++)																										\
					if(IoU_table[j*nb_box+k] > max_IoU && dist_prior[j*nb_box+k] < -1.0)														\
					{																															\
						max_IoU = IoU_table[j*nb_box+k];																						\
						resp_targ = j;																											\
						resp_box = k;																											\
					}																															\
																																				\
			/* If strict_box_size > 0 and no more good prior is available, or if there is more targets than boxes */							\
			/* In that case all the remaining target are unable to be associated to */ 															\
			/* any other box and the id_in_cell loop must be stoped */																			\
			if(resp_box == -1)																													\
				continue;																														\
																																				\
			/* l is the "best" index in the "in cell" list */																					\
			/* Need to get back the original target index from the "in cell" index */															\
			l = 0;																																\
			for(j = 0; j < nb_obj_target; j++)																									\
			{																																	\
				l += target_cell_mask[j];																										\
				if(l == resp_targ + 1)																											\
					break;																														\
			}																																	\
			/* The appropriate j value is set after this early stop loop */																		\
			l_t = j*(7+nb_param+diff_flag);																										\
																																				\
			for(l = 0; l < 6; l++)																												\
				targ_int[l] = target[l_t+1+l];																									\
			for(l = 0; l < 3; l++)																												\
				targ_size[l] = targ_int[l+3] - targ_int[l];																						\
																																				\
			if(hiprand_uniform(&(((hiprandState_t*)block_state)[i])) < rand_prob)																	\
			{																																	\
				for(k = 0; k < 2*nb_box; k++)																									\
				{																																\
					l_r_b = (int)(hiprand_uniform(&(((hiprandState_t*)block_state)[i]))*nb_box);													\
					if(box_locked[l_r_b] != 2)																									\
					{																															\
						resp_box = l_r_b;																										\
						break;																													\
					}																															\
				}																																\
			}																																	\
			/* Force the association to the smallest prior (or identical) if the target is too small */											\
			else if(targ_size[0] < min_prior_forced_scaling*prior_size[s_p_i*3+0]																\
				&& targ_size[1] < min_prior_forced_scaling*prior_size[s_p_i*3+1]																\
				&& targ_size[2] < min_prior_forced_scaling*prior_size[s_p_i*3+2])																\
			{																																	\
				max_IoU = -2.0f; 																												\
				for(k = 0; k < nb_box; k++)																										\
				{																																\
					c_prior_size = prior_size + k*3;																							\
					if((prior_size[s_p_i*3+0] == c_prior_size[k+0] 																				\
						&& prior_size[s_p_i*3+1] == c_prior_size[k+1] 																			\
						&& prior_size[s_p_i*3+2] == c_prior_size[k+2]) 																			\
						&& IoU_table[resp_targ*nb_box+k] > max_IoU)																				\
					{																															\
						max_IoU = IoU_table[resp_targ*nb_box+k];																				\
						resp_box = k;																											\
					}																															\
				}																																\
			}																																	\
			/* If prediction is too bad, associate it to the best theoritical prior instead (might found the same box again) */					\
			/* Also force the best theoritical prior association at a small rate */																\
			else if(max_IoU < low_IoU_best_box_assoc || 																						\
				hiprand_uniform(&(((hiprandState_t*)block_state)[i])) < rand_prob_best_box_assoc)													\
			{																																	\
				if(prior_dist_type == DIST_IOU)																									\
					for(l = 0; l < 6; l++)																										\
						targ_int[l] = copysignf(0.5f,l-2.5f)*targ_size[l%3];																	\
																																				\
				best_dist = 100000.0f; best_prior_id = -1;																						\
				for(k = 0; k < nb_box; k++)																										\
				{																																\
					c_prior_size = prior_size + k*3;																							\
					switch(prior_dist_type)																										\
					{																															\
						case DIST_IOU:																											\
							for(l = 0; l < 6; l++)																								\
								out_int[l] = copysignf(0.5f,l-2.5f)*c_prior_size[l%3];															\
							c_dist = 1.0f - y_param.c_IoU_fct(out_int, targ_int);																\
							break;																												\
																																				\
						default:																												\
						case DIST_SIZE:																											\
							c_dist = sqrt(																										\
								 (targ_size[0]-c_prior_size[0])*(targ_size[0]-c_prior_size[0])													\
								+(targ_size[1]-c_prior_size[1])*(targ_size[1]-c_prior_size[1])													\
								+(targ_size[2]-c_prior_size[2])*(targ_size[2]-c_prior_size[2]));												\
							break;																												\
																																				\
						case DIST_OFFSET:																										\
							for(l = 0; l < 3; l++)																								\
							{																													\
								obj_in_offset[l+3] = targ_size[l]/c_prior_size[l];																\
								if(obj_in_offset[l+3] < size_min_sat)																			\
									obj_in_offset[l+3] = logf(size_min_sat);																	\
								else if(obj_in_offset[l+3] > size_max_sat)																		\
									obj_in_offset[l+3] = logf(size_max_sat);																	\
								else																											\
									obj_in_offset[l+3] = logf(obj_in_offset[l+3]);																\
							}																													\
																																				\
							c_dist = 																											\
								 fabsf(obj_in_offset[3])																						\
								+fabsf(obj_in_offset[4])																						\
								+fabsf(obj_in_offset[5]);																						\
							break;																												\
					}																															\
					if(c_dist < best_dist)																										\
					{																															\
						best_dist = c_dist;																										\
						best_prior_id = k;																										\
					}																															\
				}																																\
				max_IoU = -2.0f;																												\
				for(k = 0; k < nb_box; k++)																										\
				{																																\
					c_prior_size = prior_size + k*3;																							\
					if((c_prior_size[best_prior_id*3+0] == c_prior_size[0] 																		\
						&& c_prior_size[best_prior_id*3+1] == c_prior_size[1] 																	\
						&& c_prior_size[best_prior_id*3+2] == c_prior_size[2])															 		\
						&& IoU_table[resp_targ*nb_box+k] > max_IoU)																				\
					{																															\
						max_IoU = IoU_table[resp_targ*nb_box+k];																				\
						resp_box = k;																											\
					}																															\
				}																																\
				/* Should always get a resp_box != -1, regarding all previous conditions */														\
			}																																	\
		}																																		\
																																				\
		/* Mark the target as already associated by removing its contributions to the IoU table */												\
		/* Only usefull if the "difficult and bad condition" is fulfilled to prevent this target to be selected again */						\
		for(k = 0; k < nb_box; k++)																												\
			IoU_table[resp_targ*nb_box + k] = -2.0f;																							\
																																				\
		c_box_in_pix = box_in_pix + resp_box*6;																									\
		for(l = 0; l < 6; l++)																													\
			out_int[l] = c_box_in_pix[l%3] + copysignf(0.5f,l-2.5f)*c_box_in_pix[3+l%3];														\
																																				\
		for(l = 0; l < 6; l++)																													\
			targ_int[l] = target[l_t+1+l];																										\
		for(l = 0; l < 3; l++)																													\
			targ_size[l] = targ_int[l+3] - targ_int[l];																							\
																																				\
		max_IoU = y_param.c_IoU_fct(out_int, targ_int);																							\
		if(max_IoU > 0.98f)																														\
			max_IoU = 0.98f;																													\
		if(class_only_IoU > -2.0f)																												\
			max_IoU = class_only_IoU; /*regardless of actual IoU because class only box is not precise*/										\
																																				\
		l_o = resp_box*(8+nb_class+nb_param);																									\
		c_prior_size = prior_size + 3*resp_box;																									\
																																				\
		/* Positive reinforcement */ 																											\
		targ_diff_flag = 0;																														\
		if(diff_flag)	/* Cast from mixed precision type to float is always possible, but not necessary to int directly */						\
			targ_diff_flag = (int)((float)target[l_t+7+nb_param]);																				\
																																				\
		/* If the target is flagged as "difficult", only update the matching box if the prediction is already confident enough */				\
		/* The target is removed from the list anyway, and the corresponding box fall to "background" or "Good_but_not_best" case*/				\
		if(diff_flag && targ_diff_flag > 0 && (max_IoU < diff_IoU_lim || (float)output[(l_o+7)*f_offset] < diff_obj_lim))						\
			continue;																															\
																																				\
		/* Mark the box as already associated by removing its contributions to the IoU table */													\
		for(j = 0; j < nb_in_cell; j++)																											\
			IoU_table[j*nb_box + resp_box] = -2.0f;																								\
																																				\
		box_locked[resp_box] = 2;																												\
																																				\
		for(l = 0; l < 3; l++)																													\
			obj_in_offset[l] = ((targ_int[l+3] + targ_int[l])*0.5f - cell_pos[l]*cell_size[l])/(float)cell_size[l];								\
		for(l = 0; l < 3; l++)																													\
		{																																		\
			obj_in_offset[l+3] = targ_size[l]/c_prior_size[l];																					\
			if(obj_in_offset[l+3] < size_min_sat)																								\
				obj_in_offset[l+3] = logf(size_min_sat);																						\
			else if(obj_in_offset[l+3] > size_max_sat)																							\
				obj_in_offset[l+3] = logf(size_max_sat);																						\
			else																																\
				obj_in_offset[l+3] = logf(obj_in_offset[l+3]);																					\
		}																																		\
																																				\
		/* Note: most of the following could be replaced by function pointers to avoid so much switch statements */								\
		switch(fit_pos)																															\
		{																																		\
			case 1:																																\
				for(k = 0; k < 3; k++)																											\
				{																																\
					if(fit_dim > k && class_only_IoU < -1.9f && (diff_flag == 0 || targ_diff_flag < 3))											\
						delta_o[(l_o+k)*f_offset] = (type)(TC_scale_factor*sm_tab[0][0]															\
							*coord_scale*(float)output[(l_o+k)*f_offset]																		\
							*(1.0f-(float)output[(l_o+k)*f_offset])																				\
							*((float)output[(l_o+k)*f_offset]-obj_in_offset[k]));																\
					else																														\
						delta_o[(l_o+k)*f_offset] = (type)(0.0f);																				\
				}																																\
				break;																															\
			case 0:																																\
				for(k = 0; k < 3; k++)																											\
				{																																\
					if(fit_dim > k)																												\
						delta_o[(l_o+k)*f_offset] = (type)(TC_scale_factor*sm_tab[0][0]															\
							*coord_scale*(float)output[(l_o+k)*f_offset]																		\
							*(1.0f-(float)output[(l_o+k)*f_offset])																				\
							*((float)output[(l_o+k)*f_offset]-0.5f));																			\
					else																														\
						delta_o[(l_o+k)*f_offset] = (type)(0.0f);																				\
				}																																\
				break;																															\
			case -1:																															\
				for(k = 0; k < 3; k++)																											\
					delta_o[(l_o+k)*f_offset] = (type)(0.0f);																					\
				break;																															\
		}																																		\
																																				\
		switch(fit_size)																														\
		{																																		\
			case 1:																																\
				for(k = 0; k < 3; k++)																											\
				{																																\
					if(fit_dim > k && class_only_IoU < -1.9f && (diff_flag == 0 || targ_diff_flag < 3))											\
						delta_o[(l_o+k+3)*f_offset] = (type) (TC_scale_factor*sm_tab[1][0]														\
							*size_scale*((float)output[(l_o+k+3)*f_offset]-obj_in_offset[k+3]));												\
					else																														\
						delta_o[(l_o+k+3)*f_offset] = (type) (0.0f);																			\
				}																																\
				break;																															\
			case 0:																																\
				for(k = 0; k < 3; k++)																											\
				{																																\
					if(fit_dim > k)																												\
						delta_o[(l_o+k+3)*f_offset] = (type) (TC_scale_factor*sm_tab[1][0]														\
							*size_scale*((float)output[(l_o+k+3)*f_offset]-0.0f));																\
					else																														\
						delta_o[(l_o+k+3)*f_offset] = (type) (0.0f);																			\
				}																																\
				break;																															\
			case -1:																															\
				for(k = 0; k < 3; k++)																											\
					delta_o[(l_o+k+3)*f_offset] = (type) (0.0f);																				\
				break;																															\
		}																																		\
																																				\
		switch(fit_prob)																														\
		{																																		\
			case 1:																																\
				if(max_IoU > min_prob_IoU_lim)																									\
					delta_o[(l_o+6)*f_offset] = (type)(TC_scale_factor*sm_tab[2][0]																\
						*prob_scale*(float)output[(l_o+6)*f_offset]																				\
						*(1.0f-(float)output[(l_o+6)*f_offset])																					\
						*((float)output[(l_o+6)*f_offset]-0.98f));																				\
				else																															\
					delta_o[(l_o+6)*f_offset] = (type)(0.0f);																					\
				break;																															\
			case 0:																																\
				delta_o[(l_o+6)*f_offset] = (type)(TC_scale_factor*sm_tab[2][0]																	\
					*prob_scale*(float)output[(l_o+6)*f_offset]																					\
					*(1.0f-(float)output[(l_o+6)*f_offset])																						\
					*((float)output[(l_o+6)*f_offset]-0.5f));																					\
				break;																															\
			case -1:																															\
				delta_o[(l_o+6)*f_offset] = (type)(0.0f);																						\
				break;																															\
		}																																		\
																																				\
		switch(fit_obj)																															\
		{																																		\
			case 1:																																\
				if(max_IoU > min_obj_IoU_lim)																									\
					delta_o[(l_o+7)*f_offset] = (type)(TC_scale_factor*sm_tab[3][0]																\
						*obj_scale*(float)output[(l_o+7)*f_offset]																				\
						*(1.0f-(float)output[(l_o+7)*f_offset])																					\
						*((float)output[(l_o+7)*f_offset]-(1.0+max_IoU)*0.5));																	\
				else																															\
					delta_o[(l_o+7)*f_offset] = (type)(0.0f);																					\
				break;																															\
			case 0:																																\
				delta_o[(l_o+7)*f_offset] = (type)(TC_scale_factor*sm_tab[3][0]																	\
					*obj_scale*(float)output[(l_o+7)*f_offset]																					\
					*(1.0f-(float)output[(l_o+7)*f_offset])																						\
					*((float)output[(l_o+7)*f_offset]-0.5f));																					\
				break;																															\
			case -1:																															\
				delta_o[(l_o+7)*f_offset] = (type)(0.0f);																						\
				break;																															\
		}																																		\
																																				\
		/* Note : mean square error on classes => could be changed to soft max but difficult to balance */										\
		switch(fit_class)																														\
		{																																		\
			case 1:																																\
				if(max_IoU > min_class_IoU_lim && (diff_flag == 0 || targ_diff_flag < 2))														\
				{																																\
					if(class_softmax)																											\
					{																															\
						for(k = 0; k < nb_class; k++)																							\
						{																														\
							if(k == (int) target[l_t]-1)																						\
								delta_o[(l_o+8+k)*f_offset] = (type) (TC_scale_factor															\
									*class_scale*((float)output[(l_o+8+k)*f_offset]-1.0f));														\
							else																												\
								delta_o[(l_o+8+k)*f_offset] = (type) (TC_scale_factor															\
									*class_scale*((float)output[(l_o+8+k)*f_offset]-0.0f));														\
						}																														\
					}																															\
					else																														\
					{																															\
						for(k = 0; k < nb_class; k++)																							\
						{																														\
							if(k == (int) target[l_t]-1)																						\
								delta_o[(l_o+8+k)*f_offset] = (type) (TC_scale_factor*sm_tab[4][0]												\
									*class_scale*(float)output[(l_o+8+k)*f_offset]																\
									*(1.0f-(float)output[(l_o+8+k)*f_offset])																	\
									*((float)output[(l_o+8+k)*f_offset]-0.98f));																\
							else																												\
								delta_o[(l_o+8+k)*f_offset] = (type) (TC_scale_factor*sm_tab[4][0]												\
									*class_scale*(float)output[(l_o+8+k)*f_offset]																\
									*(1.0f-(float)output[(l_o+8+k)*f_offset])																	\
									*((float)output[(l_o+8+k)*f_offset]-0.02f));																\
						}																														\
					}																															\
				}																																\
				else																															\
					for(k = 0; k < nb_class; k++)																								\
						delta_o[(l_o+8+k)*f_offset] = (type) (0.0f);																			\
				break;																															\
			case 0:																																\
				if(class_softmax)																												\
				{																																\
					/* Could compute CE with target = 1/nb_class, but in this case perfect classification error > 0 (still minimum) */			\
					for(k = 0; k < nb_class; k++)																								\
						delta_o[(l_o+8+k)*f_offset] = (type) (0.0f);																			\
				}																																\
				else																															\
				{																																\
					for(k = 0; k < nb_class; k++)																								\
						delta_o[(l_o+8+k)*f_offset] = (type) (TC_scale_factor*sm_tab[4][0]														\
							*class_scale*(float)output[(l_o+8+k)*f_offset]																		\
							*(1.0f-(float)output[(l_o+8+k)*f_offset])																			\
							*((float)output[(l_o+8+k)*f_offset]-0.5f));																			\
				}																																\
				break;																															\
			case -1:																															\
				for(k = 0; k < nb_class; k++)																									\
					delta_o[(l_o+8+k)*f_offset] = (type) (0.0f);																				\
				break;																															\
		}																																		\
																																				\
		/* Linear activation of additional parameters */																						\
		switch(fit_param)																														\
		{																																		\
			case 1:																																\
				if(max_IoU > min_param_IoU_lim && (diff_flag == 0 || targ_diff_flag < 2))														\
					for(k = 0; k < nb_param; k++)																								\
						delta_o[(l_o+8+nb_class+k)*f_offset] = 																					\
							(type) (param_ind_scale[k]*TC_scale_factor*sm_tab[5][0]*param_scale													\
							*((float)output[(l_o+8+nb_class+k)*f_offset]-(float)target[l_t+7+k]));												\
				else																															\
					for(k = 0; k < nb_param; k++)																								\
						delta_o[(l_o+8+nb_class+k)*f_offset] = (type) (0.0f);																	\
				break;																															\
			case 0:																																\
				for(k = 0; k < nb_param; k++)																									\
					delta_o[(l_o+8+nb_class+k)*f_offset] = 																						\
						(type) (param_ind_scale[k]*TC_scale_factor*sm_tab[5][0]*param_scale														\
						*((float)output[(l_o+8+nb_class+k)*f_offset]-0.5f));																	\
				break;																															\
			case -1:																															\
				for(k = 0; k < nb_param; k++)																									\
					delta_o[(l_o+8+nb_class+k)*f_offset] = (type) (0.0f);																		\
				break;																															\
		}																																		\
	}																																			\
																																				\
	for(j = 0; j < nb_box; j++)																													\
	{																																			\
		/* If no match only update Objectness toward 0 */																						\
		/* (here it means error compute)! (no coordinate nor class update) */																	\
		l_o = j*(8+nb_class+nb_param);																											\
		if(box_locked[j] != 2)																													\
		{																																		\
			for(k = 0; k < 6; k++)																												\
				delta_o[(l_o+k)*f_offset] = (type) 0.0f;																						\
																																				\
			if(box_locked[j] == 1)																												\
			{																																	\
				delta_o[(l_o+6)*f_offset] = (type) 0.0f;																						\
				delta_o[(l_o+7)*f_offset] = (type) 0.0f;																						\
			}																																	\
			else																																\
			{																																	\
				switch(fit_prob)																												\
				{																																\
					case 1:																														\
						delta_o[(l_o+6)*f_offset] = (type)(																						\
							TC_scale_factor*sm_tab[2][0]*(lambda_noobj_prior[j])																\
							*prob_scale*(float)output[(l_o+6)*f_offset]																			\
							*(1.0f-(float)output[(l_o+6)*f_offset])																				\
							*((float)output[(l_o+6)*f_offset]-0.02f));																			\
						break;																													\
					case 0:																														\
						delta_o[(l_o+6)*f_offset] = (type)(																						\
							TC_scale_factor*sm_tab[2][0]*(lambda_noobj_prior[j])																\
							*prob_scale*(float)output[(l_o+6)*f_offset]																			\
							*(1.0f-(float)output[(l_o+6)*f_offset])																				\
							*((float)output[(l_o+6)*f_offset]-0.5f));																			\
						break;																													\
					case -1:																													\
						delta_o[(l_o+6)*f_offset] = (type)(0.0f);																				\
						break;																													\
				}																																\
				switch(fit_obj)																													\
				{																																\
					case 1:																														\
						delta_o[(l_o+7)*f_offset] = (type)(																						\
							TC_scale_factor*sm_tab[3][0]*(lambda_noobj_prior[j])																\
							*obj_scale*(float)output[(l_o+7)*f_offset]																			\
							*(1.0f-(float)output[(l_o+7)*f_offset])																				\
							*((float)output[(l_o+7)*f_offset]-0.02f));																			\
						break;																													\
					case 0:																														\
						delta_o[(l_o+7)*f_offset] = (type)(																						\
							TC_scale_factor*sm_tab[3][0]*(lambda_noobj_prior[j])																\
							*obj_scale*(float)output[(l_o+7)*f_offset]																			\
							*(1.0f-(float)output[(l_o+7)*f_offset])																				\
							*((float)output[(l_o+7)*f_offset]-0.5f));																			\
						break;																													\
					case -1:																													\
						delta_o[(l_o+7)*f_offset] = (type)(0.0f);																				\
						break;																													\
				}																																\
			}																																	\
																																				\
			for(k = 0; k < nb_class; k++)																										\
				delta_o[(l_o+8+k)*f_offset] = (type) (0.0f);																					\
																																				\
			for(k = 0; k < nb_param; k++)																										\
				delta_o[(l_o+8+nb_class+k)*f_offset] = (type) (0.0f);																			\
		}																																		\
	}																																			\
}


#define YOLO_error_kernel(name, type)																											\
__global__ void YOLO_error_kernel_##name																										\
	(float *output_error, void *i_output, void *i_target, int flat_target_size, int flat_output_size, 											\
	int nb_area_w, int nb_area_h, int nb_area_d, yolo_param y_param, size_t size)																\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	if(i >= size)																																\
		return;																																	\
																																				\
	type *output = (type*) i_output;																											\
	type *target = (type*) i_target;																											\
																																				\
	/* Define many "shorts" for y_param content to enhance code redeability*/																	\
	int nb_box = y_param.nb_box, nb_class = y_param.nb_class, nb_param = y_param.nb_param; 														\
	int strict_box_size_association = y_param.strict_box_size_association;																		\
	int fit_dim = y_param.fit_dim;																												\
	float min_prior_forced_scaling = y_param.min_prior_forced_scaling;																			\
	int class_softmax = y_param.class_softmax, diff_flag = y_param.diff_flag;																	\
	int prior_dist_type = y_param.prior_dist_type;																								\
	int error_type = y_param.error_type;																										\
																																				\
	float coord_scale = y_param.scale_tab[0], size_scale  = y_param.scale_tab[1];																\
	float prob_scale  = y_param.scale_tab[2], obj_scale   = y_param.scale_tab[3];																\
	float class_scale = y_param.scale_tab[4], param_scale = y_param.scale_tab[5];																\
																																				\
	float *prior_size         = y_param.prior_size;																								\
	int   *cell_size          = y_param.cell_size;																								\
	float *lambda_noobj_prior = y_param.noobj_prob_prior;																						\
	float **sm_tab            = y_param.slopes_and_maxes_tab;																					\
	float *param_ind_scale    = y_param.param_ind_scale;																						\
	float *IoU_monitor        = y_param.IoU_monitor;																							\
	int   *target_cell_mask   = y_param.target_cell_mask;																						\
	float *IoU_table          = y_param.IoU_table;																								\
	float *dist_prior         = y_param.dist_prior;																								\
	int   *box_locked         = y_param.box_locked;																								\
	float *box_in_pix         = y_param.box_in_pix;																								\
																																				\
	float size_max_sat = expf(sm_tab[1][1]), size_min_sat = expf(sm_tab[1][2]);																	\
	float good_IoU_lim      = y_param.IoU_limits[0], low_IoU_best_box_assoc = y_param.IoU_limits[1];											\
	float min_prob_IoU_lim  = y_param.IoU_limits[2], min_obj_IoU_lim        = y_param.IoU_limits[3];											\
	float min_class_IoU_lim = y_param.IoU_limits[4], min_param_IoU_lim      = y_param.IoU_limits[5];											\
	float diff_IoU_lim      = y_param.IoU_limits[6], diff_obj_lim           = y_param.IoU_limits[7];											\
	int fit_pos = y_param.fit_parts[0], fit_size  = y_param.fit_parts[1], fit_prob  = y_param.fit_parts[2]; 									\
	int fit_obj = y_param.fit_parts[3], fit_class = y_param.fit_parts[4], fit_param = y_param.fit_parts[5];										\
																																				\
	int j, k, l, l_o, l_t;																														\
	int c_batch, f_offset, best_prior_id, nb_obj_target, s_p_i = 0;																				\
	int nb_in_cell, id_in_cell, resp_box = -1, resp_targ = -1, targ_diff_flag = 0;																\
	float best_dist, c_dist, max_IoU, current_IoU;																								\
	int cell_pos[3], c_nb_area[3], obj_c[3];																									\
	float *c_box_in_pix, *c_prior_size;																											\
	float obj_in_offset[6], out_int[6], targ_int[6], targ_size[3];																				\
	float class_only_IoU = -2.0f;																												\
																																				\
	c_nb_area[0] = nb_area_w; c_nb_area[1] = nb_area_h; c_nb_area[2] = nb_area_d;																\
	c_batch = i / flat_output_size;																												\
	target += flat_target_size * c_batch;																										\
	f_offset = size;																															\
																																				\
	i = i % flat_output_size;																													\
	cell_pos[2] = i / (c_nb_area[0]*c_nb_area[1]);																								\
	cell_pos[1] = (int)(i % (c_nb_area[0]*c_nb_area[1])) % c_nb_area[0];																		\
	cell_pos[0] = (int)(i % (c_nb_area[0]*c_nb_area[1])) / c_nb_area[0];																		\
																																				\
	output_error += (c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) 																					\
		* c_batch + cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0];												\
	output += (c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) 																							\
		* c_batch + cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0];												\
																																				\
	IoU_monitor += 2 * nb_box * ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) 																		\
		* c_batch + cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]);											\
																																				\
	target_cell_mask +=	(c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) * c_batch * y_param.max_nb_obj_per_image;										\
	target_cell_mask +=	(cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * y_param.max_nb_obj_per_image;		\
																																				\
	/*Could redume memory footprint with a max_nb_obj_per_cell parameter*/																		\
	IoU_table += ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) * c_batch * y_param.max_nb_obj_per_image * nb_box);									\
	IoU_table += (cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * y_param.max_nb_obj_per_image * nb_box;		\
																																				\
	dist_prior += ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2])*c_batch * y_param.max_nb_obj_per_image * nb_box);									\
	dist_prior += (cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) *  y_param.max_nb_obj_per_image * nb_box;	\
																																				\
	box_locked += ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) * c_batch * nb_box);																\
	box_locked += (cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * nb_box;									\
																																				\
	box_in_pix += ((c_nb_area[0]*c_nb_area[1]*c_nb_area[2]) * c_batch * 6 * nb_box);															\
	box_in_pix += (cell_pos[2]*c_nb_area[0]*c_nb_area[1] + cell_pos[1]*c_nb_area[0] + cell_pos[0]) * 6 * nb_box;								\
																																				\
	nb_obj_target = target[0];																													\
	target++;																																	\
																																				\
	if(nb_obj_target == -1)																														\
	{																																			\
		nb_obj_target = 1;																														\
		class_only_IoU = good_IoU_lim; 																											\
	}																																			\
																																				\
	best_dist = 100000000;																														\
	for(k = 0; k < nb_box; k++)																													\
	{																																			\
		box_locked[k] = 0;																														\
		c_box_in_pix = box_in_pix + k*6;																										\
		c_prior_size = prior_size + k*3;																										\
		l_o = k*(8+nb_class+nb_param);																											\
		for(l = 0; l < 3; l++)																													\
			c_box_in_pix[l] = ((float)output[(l_o+l)*f_offset] + cell_pos[l]) * cell_size[l];													\
		for(l = 0; l < 3; l++)																													\
			c_box_in_pix[l+3] = c_prior_size[l]*expf((float)output[(l_o+l+3)*f_offset]);														\
																																				\
		/* Min prior best association could be improved by using the "fit_dim" parameter to avoid definition issues with unused dimensions */	\
		/* This would allow to verify if each used dimension is smaller, rather that using a surface criteria (later in this function) */		\
		c_dist = sqrt(c_prior_size[0]*c_prior_size[0] 																							\
					+ c_prior_size[1]*c_prior_size[1]																							\
					+ c_prior_size[2]*c_prior_size[2]);																							\
		if(c_dist < best_dist)																													\
		{																																		\
			best_dist = c_dist;																													\
			s_p_i = k;																															\
		}																																		\
																																				\
		IoU_monitor[k*2] = -1.0f;																												\
		IoU_monitor[k*2+1] = -1.0f;																												\
	}																																			\
																																				\
	nb_in_cell = 0;																																\
	for(j = 0; j < nb_obj_target; j++)																											\
	{																																			\
		l_t = j*(7+nb_param+diff_flag);																											\
		for(l = 0; l < 6; l++)																													\
			targ_int[l] = target[l_t+1+l];																										\
																																				\
		/* Search for targets that should be predicted by the current cell element */															\
		target_cell_mask[j] = 1;																												\
		for(l = 0; l < 3; l++)																													\
		{																																		\
			obj_c[l] = (int)( ((float)target[l_t+l+4] + (float)target[l_t+l+1])*0.5f / cell_size[l]);											\
			/* If target outside the current cell element, set target flag to 0*/																\
			if(obj_c[l] != cell_pos[l])																											\
				target_cell_mask[j] = 0;																										\
		}																																		\
																																				\
		if(target_cell_mask[j] == 1)																											\
			nb_in_cell++;																														\
																																				\
		/* Flag all the "Good but not best boxes" for all targets regardless of the grid element */												\
		for(k = 0; k < nb_box; k++)																												\
		{																																		\
			if(box_locked[k] != 0)																												\
				continue;																														\
			c_box_in_pix = box_in_pix+k*6;																										\
			for(l = 0; l < 6; l++)																												\
				out_int[l] = c_box_in_pix[l%3] + copysignf(0.5f,l-2.5f)*c_box_in_pix[3+l%3];													\
																																				\
			current_IoU = y_param.c_IoU_fct(out_int, targ_int);																					\
			if(current_IoU > good_IoU_lim)																										\
				box_locked[k] = 1;																												\
		}																																		\
	}																																			\
																																				\
	/* For all targets in cell compute the IoU with the predictions and distances to the priors */												\
	id_in_cell = 0;																																\
	for(j = 0; j < nb_obj_target; j++)																											\
	{																																			\
		if(target_cell_mask[j] == 0)																											\
			continue;																															\
																																				\
		l_t = j*(7+nb_param+diff_flag);																											\
		for(l = 0; l < 6; l++)																													\
			targ_int[l] = target[l_t+1+l];																										\
		for(l = 0; l < 3; l++)																													\
			targ_size[l] = targ_int[l+3] - targ_int[l];																							\
																																				\
		for(k = 0; k < nb_box; k++)																												\
		{																																		\
			c_box_in_pix = box_in_pix+k*6;																										\
			for(l = 0; l < 6; l++)																												\
				out_int[l] = c_box_in_pix[l%3] + copysignf(0.5f,l-2.5f)*c_box_in_pix[3+l%3];													\
																																				\
			current_IoU = y_param.c_IoU_fct(out_int, targ_int);																					\
			IoU_table[id_in_cell*nb_box + k] = current_IoU;																						\
			dist_prior[id_in_cell*nb_box + k] = -2.0f;																							\
		}																																		\
																																				\
		/* Restrict the association to the l best theoritical prior (times repetition of identical priors) */									\
		if(error_type == ERR_COMPLETE && strict_box_size_association > 0)																		\
		{																																		\
			if(prior_dist_type == DIST_IOU)																										\
				for(l = 0; l < 6; l++)																											\
					targ_int[l] = copysignf(0.5f,l-2.5f)*targ_size[l%3];																		\
																																				\
			for(k = 0; k < nb_box; k++)																											\
			{																																	\
				c_prior_size = prior_size + k*3;																								\
				switch(prior_dist_type)																											\
				{																																\
					case DIST_IOU:																												\
						for(l = 0; l < 6; l++)																									\
							out_int[l] = copysignf(0.5f,l-2.5f)*c_prior_size[l%3];																\
						dist_prior[id_in_cell*nb_box + k] = 1.0f - y_param.c_IoU_fct(out_int, targ_int);										\
						break;																													\
																																				\
					default:																													\
					case DIST_SIZE:																												\
						dist_prior[id_in_cell*nb_box + k] = sqrt(																				\
							 (targ_size[0]-c_prior_size[0])*(targ_size[0]-c_prior_size[0])														\
							+(targ_size[1]-c_prior_size[1])*(targ_size[1]-c_prior_size[1])														\
							+(targ_size[2]-c_prior_size[2])*(targ_size[2]-c_prior_size[2]));													\
						break;																													\
																																				\
					case DIST_OFFSET:																											\
						for(l = 0; l < 3; l++)																									\
						{																														\
							obj_in_offset[l+3] = targ_size[l]/c_prior_size[l];																	\
							if(obj_in_offset[l+3] < size_min_sat)																				\
								obj_in_offset[l+3] = logf(size_min_sat);																		\
							else if(obj_in_offset[l+3] > size_max_sat)																			\
								obj_in_offset[l+3] = logf(size_max_sat);																		\
							else																												\
								obj_in_offset[l+3] = logf(obj_in_offset[l+3]);																	\
						}																														\
																																				\
						dist_prior[id_in_cell*nb_box + k] = 																					\
							 fabsf(obj_in_offset[3])																							\
							+fabsf(obj_in_offset[4])																							\
							+fabsf(obj_in_offset[5]);																							\
						break;																													\
				}																																\
			}																																	\
																																				\
			for(l = 0; l < strict_box_size_association; l++)																					\
			{																																	\
				best_dist = 1000000.0f;	best_prior_id = -1;																						\
				for(k = 0; k < nb_box; k++)																										\
					if(dist_prior[id_in_cell*nb_box+k] > 0.0 && dist_prior[id_in_cell*nb_box+k] < best_dist)									\
					{																															\
						best_dist = dist_prior[id_in_cell*nb_box+k];																			\
						best_prior_id = k;																										\
					}																															\
				for(k = 0; k < nb_box; k++) /* Flag the closest theoritical prior (and identical ones if any) */								\
				{																																\
					c_prior_size = prior_size + k*3;																							\
					if(prior_size[best_prior_id*3+0] == c_prior_size[0] 																		\
						&& prior_size[best_prior_id*3+1] == c_prior_size[1] 																	\
						&& prior_size[best_prior_id*3+2] == c_prior_size[2])																	\
						dist_prior[id_in_cell*nb_box+k] = -2.0f;																				\
				}																																\
			}																																	\
		}																																		\
																																				\
		id_in_cell++;																															\
	}																																			\
																																				\
	for(id_in_cell = 0; id_in_cell < nb_in_cell; id_in_cell++)																					\
	{																																			\
		/* No random association in error display*/																								\
		max_IoU = -2.0f; resp_box = -1;	resp_targ = -1;																							\
		for(j = 0; j < nb_in_cell; j++)																											\
			for(k = 0; k < nb_box; k++)																											\
				if(IoU_table[j*nb_box+k] > max_IoU && dist_prior[j*nb_box+k] < -1.0f)															\
				{																																\
					max_IoU = IoU_table[j*nb_box+k];																							\
					resp_targ = j;																												\
					resp_box = k;																												\
				}																																\
																																				\
		/* If strict_box_size > 0 and no more good prior is available, or if there is more targets than boxes */								\
		/* In that case all the remaining target are unable to be associated to */ 																\
		/* any other box and the id_in_cell loop must be stoped */																				\
		if(resp_box == -1)																														\
			continue;																															\
																																				\
		/* l is the "best" index in the "in cell" list */																						\
		/*Need to get back the original target index from the "in cell" index*/																	\
		l = 0;																																	\
		for(j = 0; j < nb_obj_target; j++)																										\
		{																																		\
			l += target_cell_mask[j];																											\
			if(l == resp_targ + 1)																												\
				break;																															\
		}																																		\
		/* The appropriate j is defined after this early stop loop*/																			\
		l_t = j*(7+nb_param+diff_flag);																											\
																																				\
		if(error_type == ERR_COMPLETE)																											\
		{																																		\
			for(l = 0; l < 6; l++)																												\
				targ_int[l] = target[l_t+1+l];																									\
			for(l = 0; l < 3; l++)																												\
				targ_size[l] = targ_int[l+3] - targ_int[l];																						\
																																				\
			/* Force the association to the smallest prior (or identical) if the target is too small */											\
			if(targ_size[0] < min_prior_forced_scaling*prior_size[s_p_i*3+0]																	\
				&& targ_size[1] < min_prior_forced_scaling*prior_size[s_p_i*3+1]																\
				&& targ_size[2] < min_prior_forced_scaling*prior_size[s_p_i*3+2])																\
			{																																	\
				max_IoU = -2.0f; best_dist = prior_size[s_p_i*3+0]*prior_size[s_p_i*3+1]*prior_size[s_p_i*3+2];									\
				for(k = 0; k < nb_box; k++)																										\
				{																																\
					c_prior_size = prior_size + k*3;																							\
					if((prior_size[s_p_i*3+0] == c_prior_size[k+0] 																				\
						&& prior_size[s_p_i*3+1] == c_prior_size[k+1] 																			\
						&& prior_size[s_p_i*3+2] == c_prior_size[k+2]) 																			\
						&& IoU_table[resp_targ*nb_box+k] > max_IoU)																				\
					{																															\
						max_IoU = IoU_table[resp_targ*nb_box+k];																				\
						resp_box = k;																											\
					}																															\
				}																																\
			}																																	\
			/* If prediction is too bad, associate it with the best theoritical prior instead (might found the same box again) */				\
			/* Also force the best theoritical prior association at a small rate */																\
			else if(max_IoU < low_IoU_best_box_assoc)																							\
			{																																	\
				if(prior_dist_type == DIST_IOU)																									\
					for(l = 0; l < 6; l++)																										\
						targ_int[l] = copysignf(0.5f,l-2.5f)*targ_size[l%3];																	\
																																				\
				best_dist = 100000.0f; best_prior_id = -1;																						\
				for(k = 0; k < nb_box; k++)																										\
				{																																\
					c_prior_size = prior_size + k*3;																							\
					switch(prior_dist_type)																										\
					{																															\
						case DIST_IOU:																											\
							for(l = 0; l < 6; l++)																								\
								out_int[l] = copysignf(0.5f,l-2.5f)*c_prior_size[l%3];															\
							c_dist = 1.0f - y_param.c_IoU_fct(out_int, targ_int);																\
							break;																												\
																																				\
						default:																												\
						case DIST_SIZE:																											\
							c_dist = sqrt(																										\
								 (targ_size[0]-c_prior_size[0])*(targ_size[0]-c_prior_size[0])													\
								+(targ_size[1]-c_prior_size[1])*(targ_size[1]-c_prior_size[1])													\
								+(targ_size[2]-c_prior_size[2])*(targ_size[2]-c_prior_size[2]));												\
							break;																												\
																																				\
						case DIST_OFFSET:																										\
							for(l = 0; l < 3; l++)																								\
							{																													\
								obj_in_offset[l+3] = targ_size[l]/c_prior_size[l];																\
								if(obj_in_offset[l+3] < size_min_sat)																			\
									obj_in_offset[l+3] = logf(size_min_sat);																	\
								else if(obj_in_offset[l+3] > size_max_sat)																		\
									obj_in_offset[l+3] = logf(size_max_sat);																	\
								else																											\
									obj_in_offset[l+3] = logf(obj_in_offset[l+3]);																\
							}																													\
																																				\
							c_dist = 																											\
								 fabsf(obj_in_offset[3])																						\
								+fabsf(obj_in_offset[4])																						\
								+fabsf(obj_in_offset[5]);																						\
							break;																												\
					}																															\
					if(c_dist < best_dist)																										\
					{																															\
						best_dist = c_dist;																										\
						best_prior_id = k;																										\
					}																															\
				}																																\
				max_IoU = -2.0f;																												\
				for(k = 0; k < nb_box; k++)																										\
				{																																\
					c_prior_size = prior_size + k*3;																							\
					if((c_prior_size[best_prior_id*3+0] == c_prior_size[0] 																		\
						&& c_prior_size[best_prior_id*3+1] == c_prior_size[1] 																	\
						&& c_prior_size[best_prior_id*3+2] == c_prior_size[2])															 		\
						&& IoU_table[resp_targ*nb_box+k] > max_IoU)																				\
					{																															\
						max_IoU = IoU_table[resp_targ*nb_box+k];																				\
						resp_box = k;																											\
					}																															\
				}																																\
				/* Should always get a resp_box != -1, regarding all previous conditions */														\
			}																																	\
		}																																		\
																																				\
		/* Mark the target as already associated by removing its contributions to the IoU table */												\
		for(k = 0; k < nb_box; k++)																												\
			IoU_table[resp_targ*nb_box + k] = -2.0f;																							\
																																				\
		c_box_in_pix = box_in_pix + resp_box*6;																									\
		for(l = 0; l < 6; l++)																													\
			out_int[l] = c_box_in_pix[l%3] + copysignf(0.5f,l-2.5f)*c_box_in_pix[3+l%3];														\
																																				\
		for(l = 0; l < 6; l++)																													\
			targ_int[l] = target[l_t+1+l];																										\
		for(l = 0; l < 3; l++)																													\
			targ_size[l] = targ_int[l+3] - targ_int[l];																							\
																																				\
		max_IoU = y_param.c_IoU_fct(out_int, targ_int);																							\
		if(max_IoU > 0.98f)																														\
			max_IoU = 0.98f;																													\
		if(class_only_IoU > -2.0f)																												\
			max_IoU = class_only_IoU; /*regardless of actual IoU because class only box is not precise*/										\
																																				\
		l_o = resp_box*(8+nb_class+nb_param);																									\
		c_prior_size = prior_size + 3*resp_box;																									\
																																				\
		/* Positive reinforcement */ 																											\
		targ_diff_flag = 0;																														\
		if(diff_flag)	/* Cast from mixed precision type to float is always possible, but not necessary to int directly */						\
			targ_diff_flag = (int)((float)target[l_t+7+nb_param]);																				\
																																				\
		/* If the target is flagged as "difficult", only update the matching box if the prediction is already confident enough */				\
		/* The target is removed from the list anyway, and the corresponding box fall to "background" or "Good_but_not_best" case*/				\
		if(diff_flag && targ_diff_flag > 0																										\
			&& (error_type == ERR_NATURAL || max_IoU < diff_IoU_lim || (float)output[(l_o+7)*f_offset] < diff_obj_lim))							\
			continue;																															\
																																				\
		/* Mark the box as already associated by removing its contributions to the IoU table */													\
		for(j = 0; j < nb_in_cell; j++)																											\
			IoU_table[j*nb_box + resp_box] = -2.0f;																								\
																																				\
		box_locked[resp_box] = 2;																												\
																																				\
		IoU_monitor[resp_box*2] = (float)output[(l_o+7)*f_offset];																				\
		IoU_monitor[resp_box*2+1] = max_IoU;																									\
																																				\
		for(l = 0; l < 3; l++)																													\
			obj_in_offset[l] = ((targ_int[l+3] + targ_int[l])*0.5f - cell_pos[l]*cell_size[l])/(float)cell_size[l];								\
		for(l = 0; l < 3; l++)																													\
		{																																		\
			obj_in_offset[l+3] = targ_size[l]/c_prior_size[l];																					\
			if(obj_in_offset[l+3] < size_min_sat)																								\
				obj_in_offset[l+3] = logf(size_min_sat);																						\
			else if(obj_in_offset[l+3] > size_max_sat)																							\
				obj_in_offset[l+3] = logf(size_max_sat);																						\
			else																																\
				obj_in_offset[l+3] = logf(obj_in_offset[l+3]);																					\
		}																																		\
																																				\
		switch(fit_pos)																															\
		{																																		\
			case 1:																																\
				for(k = 0; k < 3; k++)																											\
				{																																\
					if(fit_dim > k && class_only_IoU < -1.9f && (diff_flag == 0 || targ_diff_flag < 3))											\
						output_error[(l_o+k)*f_offset] = 0.5f*coord_scale																		\
							*((float)output[(l_o+k)*f_offset]-obj_in_offset[k])																	\
							*((float)output[(l_o+k)*f_offset]-obj_in_offset[k]);																\
					else																														\
						output_error[(l_o+k)*f_offset] = 0.0f;																					\
				}																																\
				break;																															\
			case 0:																																\
				for(k = 0; k < 3; k++)																											\
				{																																\
					if(fit_dim > k)																												\
						output_error[(l_o+k)*f_offset] = 0.5f*coord_scale																		\
							*((float)output[(l_o+k)*f_offset]-0.0f)																				\
							*((float)output[(l_o+k)*f_offset]-0.0f);																			\
					else																														\
						output_error[(l_o+k)*f_offset] = 0.0f;																					\
				}																																\
				break;																															\
			case -1:																															\
				for(k = 0; k < 3; k++)																											\
					output_error[(l_o+k)*f_offset] = 0.0f;																						\
				break;																															\
		}																																		\
																																				\
		switch(fit_size)																														\
		{																																		\
			case 1:																																\
				for(k = 0; k < 3; k++)																											\
				{																																\
					if(fit_dim > k && class_only_IoU < -1.9f && (diff_flag == 0 || targ_diff_flag < 3))											\
						output_error[(l_o+k+3)*f_offset] = 0.5f*size_scale																		\
						*((float)output[(l_o+k+3)*f_offset]-obj_in_offset[k+3])																	\
						*((float)output[(l_o+k+3)*f_offset]-obj_in_offset[k+3]);																\
					else																														\
						output_error[(l_o+k+3)*f_offset] = 0.0f;																				\
				}																																\
				break;																															\
			case 0:																																\
				for(k = 0; k < 3; k++)																											\
				{																																\
					if(fit_dim > k)																												\
						output_error[(l_o+k+3)*f_offset] = 0.5f*size_scale																		\
						*((float)output[(l_o+k+3)*f_offset]-0.0f)																				\
						*((float)output[(l_o+k+3)*f_offset]-0.0f);																				\
					else																														\
						output_error[(l_o+k+3)*f_offset] = 0.0f;																				\
				}																																\
				break;																															\
			case -1:																															\
				for(k = 0; k < 3; k++)																											\
					output_error[(l_o+k+3)*f_offset] = 0.0f;																					\
				break;																															\
		}																																		\
																																				\
		switch(fit_prob)																														\
		{																																		\
			case 1:																																\
				if(max_IoU > min_prob_IoU_lim || error_type == ERR_NATURAL)																		\
					output_error[(l_o+6)*f_offset] = 0.5f*prob_scale																			\
						*((float)output[(l_o+6)*f_offset]-0.98f)																				\
						*((float)output[(l_o+6)*f_offset]-0.98f);																				\
				else																															\
					output_error[(l_o+6)*f_offset] = 0.0f;																						\
				break;																															\
			case 0:																																\
				output_error[(l_o+6)*f_offset] = 0.5f*prob_scale																				\
					*((float)output[(l_o+6)*f_offset]-0.5f)																						\
					*((float)output[(l_o+6)*f_offset]-0.5f);																					\
				break;																															\
			case -1:																															\
				output_error[(l_o+6)*f_offset] = 0.0f;																							\
				break;																															\
		}																																		\
																																				\
		switch(fit_obj)																															\
		{																																		\
			case 1:																																\
				if(max_IoU > min_obj_IoU_lim || error_type == ERR_NATURAL)																		\
					output_error[(l_o+7)*f_offset] = 0.5f*obj_scale																				\
						*((float)output[(l_o+7)*f_offset]-(1.0+max_IoU)*0.5)																	\
						*((float)output[(l_o+7)*f_offset]-(1.0+max_IoU)*0.5);																	\
				else																															\
					output_error[(l_o+7)*f_offset] = 0.0f;																						\
				break;																															\
			case 0:																																\
				output_error[(l_o+7)*f_offset] = 0.5f*obj_scale																					\
					*((float)output[(l_o+7)*f_offset]-0.5)																						\
					*((float)output[(l_o+7)*f_offset]-0.5);																						\
				break;																															\
			case -1:																															\
				output_error[(l_o+7)*f_offset] = 0.0f;																							\
				break;																															\
		}																																		\
																																				\
		/*Note : mean square error on classes => could be changed to soft max but difficult to balance*/										\
		switch(fit_class)																														\
		{																																		\
			case 1:																																\
				if((max_IoU > min_class_IoU_lim && (diff_flag == 0 || targ_diff_flag < 2)) || error_type == ERR_NATURAL)						\
				{																																\
					if(class_softmax)																											\
					{																															\
						for(k = 0; k < nb_class; k++)																							\
						{																														\
							if(k == (int)target[l_t]-1)																							\
							{																													\
								if((float)output[(l_o+8+k)*f_offset] > 0.0000001f)																\
									output_error[(l_o+8+k)*f_offset] = class_scale																\
										*(-logf((float)output[(l_o+8+k)*f_offset]));															\
								else																											\
									output_error[(l_o+8+k)*f_offset] = class_scale*(-logf(0.0000001f));											\
							}																													\
							else																												\
								output_error[(l_o+8+k)*f_offset] = 0.0f;																		\
						}																														\
					}																															\
					else																														\
					{																															\
						for(k = 0; k < nb_class; k++)																							\
						{																														\
							if(k == (int)target[l_t]-1)																							\
								output_error[(l_o+8+k)*f_offset] = 0.5f*class_scale																\
									*((float)output[(l_o+8+k)*f_offset]-0.98f)																	\
									*((float)output[(l_o+8+k)*f_offset]-0.98f);																	\
							else																												\
								output_error[(l_o+8+k)*f_offset] = 0.5f*class_scale																\
									*((float)output[(l_o+8+k)*f_offset]-0.02f)																	\
									*((float)output[(l_o+8+k)*f_offset]-0.02f);																	\
						}																														\
					}																															\
				}																																\
				else																															\
					for(k = 0; k < nb_class; k++)																								\
						output_error[(l_o+8+k)*f_offset] = 0.0f;																				\
				break;																															\
			case 0:																																\
				if(class_softmax)																												\
				{																																\
					/* Could compute CE with target = 1/nb_class, but in this case perfect classification error > 0 (still minimum) */			\
					for(k = 0; k < nb_class; k++)																								\
						output_error[(l_o+8+k)*f_offset] = 0.0f;																				\
				}																																\
				else																															\
				{																																\
					for(k = 0; k < nb_class; k++)																								\
						output_error[(l_o+8+k)*f_offset] = 0.5f*class_scale																		\
							*((float)output[(l_o+8+k)*f_offset]-0.5f)																			\
							*((float)output[(l_o+8+k)*f_offset]-0.5f);																			\
				}																																\
				break;																															\
			case -1:																															\
				for(k = 0; k < nb_class; k++)																									\
					output_error[(l_o+8+k)*f_offset] = 0.0f;																					\
				break;																															\
		}																																		\
																																				\
		/*Linear error of additional parameters*/																								\
		switch(fit_param)																														\
		{																																		\
			case 1:																																\
				if((max_IoU > min_param_IoU_lim && (diff_flag == 0 || targ_diff_flag < 2)) || error_type == ERR_NATURAL)						\
					for(k = 0; k < nb_param; k++)																								\
						output_error[(l_o+8+nb_class+k)*f_offset] = (param_ind_scale[k]*0.5f*param_scale										\
							*((float)output[(l_o+8+nb_class+k)*f_offset]-(float)target[l_t+7+k])												\
							*((float)output[(l_o+8+nb_class+k)*f_offset]-(float)target[l_t+7+k]));												\
				else																															\
					for(k = 0; k < nb_param; k++)																								\
						output_error[(l_o+8+nb_class+k)*f_offset] = 0.0f;																		\
				break;																															\
			case 0:																																\
				for(k = 0; k < nb_param; k++)																									\
					output_error[(l_o+8+nb_class+k)*f_offset] = (param_ind_scale[k]*0.5f*param_scale											\
						*((float)output[(l_o+8+nb_class+k)*f_offset]-0.5f)																		\
						*((float)output[(l_o+8+nb_class+k)*f_offset]-0.5f));																	\
				break;																															\
			default:																															\
			case -1:																															\
				for(k = 0; k < nb_param; k++)																									\
					output_error[(l_o+8+nb_class+k)*f_offset] = 0.0f;																			\
				break;																															\
		}																																		\
	}																																			\
																																				\
	for(j = 0; j < nb_box; j++)																													\
	{																																			\
		/*If no match only update Objectness toward 0 */																						\
		/*(here it means error compute)! (no coordinate nor class update)*/																		\
		l_o = j*(8+nb_class+nb_param);																											\
		if(box_locked[j] != 2)																													\
		{																																		\
			for(k = 0; k < 6; k++)																												\
				output_error[(l_o+k)*f_offset] = 0.0f;																							\
																																				\
			if(box_locked[j] == 1)																												\
			{																																	\
				output_error[(l_o+6)*f_offset] = 0.0f;																							\
				output_error[(l_o+7)*f_offset] = 0.0f;																							\
			}																																	\
			else																																\
			{																																	\
				switch(fit_prob)																												\
				{																																\
					case 1:																														\
						output_error[(l_o+6)*f_offset] = 0.5f*(lambda_noobj_prior[j])*prob_scale												\
							*((float)output[(l_o+6)*f_offset]-0.02f)																			\
							*((float)output[(l_o+6)*f_offset]-0.02f);																			\
						break;																													\
					case 0:																														\
						output_error[(j*(8+nb_class+nb_param)+6)*f_offset] = 0.5f*(lambda_noobj_prior[j])*prob_scale							\
							*((float)output[(l_o+6)*f_offset]-0.5f)																				\
							*((float)output[(l_o+6)*f_offset]-0.5f);																			\
						break;																													\
					case -1:																													\
						output_error[(l_o+6)*f_offset] = 0.0f;																					\
						break;																													\
				}																																\
																																				\
				switch(fit_obj)																													\
				{																																\
					case 1:																														\
						output_error[(l_o+7)*f_offset] = 0.5f*(lambda_noobj_prior[j])*obj_scale													\
							*((float)output[(l_o+7)*f_offset]-0.02f)																			\
							*((float)output[(l_o+7)*f_offset]-0.02f);																			\
						break;																													\
					case 0:																														\
						output_error[(l_o+7)*f_offset] = 0.5f*(lambda_noobj_prior[j])*obj_scale													\
							*((float)output[(l_o+7)*f_offset]-0.5f)																				\
							*((float)output[(l_o+7)*f_offset]-0.5f);																			\
						break;																													\
					case -1:																													\
						output_error[(l_o+7)*f_offset] = 0.0f;																					\
						break;																													\
				}																																\
			}																																	\
																																				\
			for(k = 0; k < nb_class; k++)																										\
				output_error[(l_o+8+k)*f_offset] = 0.0f;																						\
																																				\
			for(k = 0; k < nb_param; k++)																										\
				output_error[(l_o+8+nb_class+k)*f_offset] = 0.0f;																				\
																																				\
		}																																		\
	}																																			\
}


#define typed_cuda_activ_fct_association(name)																									\
void typed_cuda_activ_fct_association_##name(network *net)																						\
{																																				\
	net->cu_inst.cu_linear_activ_fcts.deriv_output_error_fct = quadratic_deriv_output_error_kernel_##name;										\
	net->cu_inst.cu_linear_activ_fcts.output_error_fct = quadratic_output_error_kernel_##name;													\
																																				\
	net->cu_inst.cu_ReLU_activ_fcts.activ_fct = ReLU_activation_kernel_##name;																	\
	net->cu_inst.cu_ReLU_activ_fcts.deriv_fct = ReLU_deriv_kernel_##name;																		\
	net->cu_inst.cu_ReLU_activ_fcts.deriv_output_error_fct = quadratic_deriv_output_error_kernel_##name;										\
	net->cu_inst.cu_ReLU_activ_fcts.output_error_fct = quadratic_output_error_kernel_##name; 													\
																																				\
	net->cu_inst.cu_logistic_activ_fcts.activ_fct = logistic_activation_kernel_##name;															\
	net->cu_inst.cu_logistic_activ_fcts.deriv_fct = logistic_deriv_kernel_##name;																\
	net->cu_inst.cu_logistic_activ_fcts.deriv_output_error_fct = quadratic_deriv_output_error_kernel_##name;									\
	net->cu_inst.cu_logistic_activ_fcts.output_error_fct = quadratic_output_error_kernel_##name;												\
																																				\
	net->cu_inst.cu_softmax_activ_fcts.activ_fct = softmax_activation_kernel_##name;															\
	net->cu_inst.cu_softmax_activ_fcts.deriv_output_error_fct = cross_entropy_deriv_output_error_kernel_##name;									\
	net->cu_inst.cu_softmax_activ_fcts.output_error_fct = cross_entropy_output_error_kernel_##name;												\
																																				\
	net->cu_inst.cu_YOLO_activ_fcts.activ_fct = YOLO_activation_kernel_##name;																	\
	net->cu_inst.cu_YOLO_activ_fcts.deriv_output_error_fct = YOLO_deriv_error_kernel_##name;													\
	net->cu_inst.cu_YOLO_activ_fcts.output_error_fct = YOLO_error_kernel_##name;																\
																																				\
	net->cu_inst.cu_auxil_fcts.cu_exp_disc_activation_kernel = exp_disc_activation_kernel_##name;												\
	net->cu_inst.cu_auxil_fcts.cu_exp_disc_deriv_output_kernel = exp_disc_deriv_output_kernel_##name;											\
}


ReLU_activation_kernel(FP32, float);
ReLU_deriv_kernel(FP32, float);
quadratic_deriv_output_error_kernel(FP32, float);
quadratic_output_error_kernel(FP32, float);
logistic_activation_kernel(FP32, float, expf);
logistic_deriv_kernel(FP32, float);
softmax_activation_kernel(FP32, float, expf);
cross_entropy_deriv_output_error_kernel(FP32, float);
cross_entropy_output_error_kernel(FP32, float);
exp_disc_activation_kernel(FP32, float, expf);
exp_disc_deriv_output_kernel(FP32, float, expf);
YOLO_activation_kernel(FP32, float, expf);
YOLO_deriv_error_kernel(FP32, float);
YOLO_error_kernel(FP32, float);
typed_cuda_activ_fct_association(FP32);


#if defined(GEN_VOLTA) || defined(GEN_AMPERE) 
ReLU_activation_kernel(FP16, half);
ReLU_deriv_kernel(FP16, half);
quadratic_deriv_output_error_kernel(FP16, half);
quadratic_output_error_kernel(FP16, half);
logistic_activation_kernel(FP16, half, expf);
logistic_deriv_kernel(FP16, half);
softmax_activation_kernel(FP16, half, expf);
cross_entropy_deriv_output_error_kernel(FP16, half);
cross_entropy_output_error_kernel(FP16, half);
exp_disc_activation_kernel(FP16, half, expf);
exp_disc_deriv_output_kernel(FP16, half, expf)
YOLO_activation_kernel(FP16, half, expf);
YOLO_deriv_error_kernel(FP16, half);
YOLO_error_kernel(FP16, half);
typed_cuda_activ_fct_association(FP16);
#endif


#if defined(GEN_AMPERE) 
ReLU_activation_kernel(BF16, hip_bfloat16);
ReLU_deriv_kernel(BF16, hip_bfloat16);
quadratic_deriv_output_error_kernel(BF16, hip_bfloat16);
quadratic_output_error_kernel(BF16, hip_bfloat16);
logistic_activation_kernel(BF16, hip_bfloat16, expf);
logistic_deriv_kernel(BF16, hip_bfloat16);
softmax_activation_kernel(BF16, hip_bfloat16, expf);
cross_entropy_deriv_output_error_kernel(BF16, hip_bfloat16);
cross_entropy_output_error_kernel(BF16, hip_bfloat16);
exp_disc_activation_kernel(BF16, hip_bfloat16, expf);
exp_disc_deriv_output_kernel(BF16, hip_bfloat16, expf);
YOLO_activation_kernel(BF16, hip_bfloat16, expf);
YOLO_deriv_error_kernel(BF16, hip_bfloat16);
YOLO_error_kernel(BF16, hip_bfloat16);
typed_cuda_activ_fct_association(BF16);
#endif


//#####################################################
//		 Linear activation related functions
//#####################################################

void cuda_linear_activation(layer *current)
{
	//empty on purpose
}


void cuda_linear_deriv(layer *previous)
{
	//empty on purpose
}

void cuda_linear_deriv_output_error(layer *current)
{	
	linear_param *param = (linear_param*)current->activ_param;
	
	cu_blocks = ( param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_linear_activ_fcts.deriv_output_error_fct<<< cu_blocks, cu_threads >>>
		(current->delta_o, current->output, current->c_network->target,
		param->dim, param->biased_dim, param->offset, current->c_network->length, 
		param->size, current->c_network->TC_scale_factor);
}

void cuda_linear_output_error(layer *current)
{	
	linear_param *param = (linear_param*)current->activ_param;
	cu_blocks = (param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_linear_activ_fcts.output_error_fct<<< cu_blocks, cu_threads >>>
		((float*)current->c_network->output_error, current->output, current->c_network->target, 
		param->dim, param->biased_dim, param->offset, current->c_network->length, param->size);
}


//#####################################################
//		 ReLU activation related functions
//#####################################################

void cuda_ReLU_activation(layer *current)
{
	ReLU_param *param = (ReLU_param*)current->activ_param;
	cu_blocks = ( param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_ReLU_activ_fcts.activ_fct<<< cu_blocks, cu_threads >>>
		(current->output, param->dim, param->biased_dim, param->offset, param->saturation, 
		param->leaking_factor, current->c_network->length, param->size);
}


void cuda_ReLU_deriv(layer *previous)
{
	ReLU_param *param = (ReLU_param*)previous->activ_param;
	cu_blocks = ( param->size + cu_threads - 1) / cu_threads;
	
	previous->c_network->cu_inst.cu_ReLU_activ_fcts.deriv_fct<<< cu_blocks, cu_threads >>>
		(previous->delta_o, previous->output, param->dim, param->biased_dim, param->offset, 
		param->saturation, param->leaking_factor, previous->c_network->length, param->size);
}


// Should re write an output function to take into account ReLU for Conv output format
void cuda_ReLU_deriv_output_error(layer* current)
{
	ReLU_param *param = (ReLU_param*)current->activ_param;
	cu_blocks = ( param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_ReLU_activ_fcts.deriv_output_error_fct<<< cu_blocks, cu_threads >>>
		(current->delta_o, current->output, current->c_network->target, param->dim, param->biased_dim,
		param->offset, current->c_network->length, param->size, current->c_network->TC_scale_factor);
	
	current->c_network->cu_inst.cu_ReLU_activ_fcts.deriv_fct<<< cu_blocks, cu_threads >>>
		(current->delta_o, current->output, param->dim, param->biased_dim,
		param->offset, param->saturation, param->leaking_factor, current->c_network->length, param->size);
}

void cuda_ReLU_output_error(layer* current)
{
	ReLU_param *param = (ReLU_param*)current->activ_param;	
	cu_blocks = (param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_ReLU_activ_fcts.output_error_fct<<< cu_blocks, cu_threads >>>
		((float*)current->c_network->output_error, current->output, current->c_network->target, 
		param->dim, param->biased_dim, param->offset, current->c_network->length, param->size);
}


//#####################################################
//		 Logistic activation related functions
//#####################################################

void cuda_logistic_activation(layer *current)
{
	logistic_param *param = (logistic_param*)current->activ_param;
	cu_blocks = (param->size + cu_threads - 1) / cu_threads;

	current->c_network->cu_inst.cu_logistic_activ_fcts.activ_fct<<< cu_blocks, cu_threads >>>
		(current->output, param->beta, param->saturation, param->dim, 
		param->biased_dim, param->offset, current->c_network->length, param->size);
}


void cuda_logistic_deriv(layer *previous)
{
	logistic_param *param = (logistic_param*)previous->activ_param;
	cu_blocks = (param->size + cu_threads - 1) / cu_threads;
	
	previous->c_network->cu_inst.cu_logistic_activ_fcts.deriv_fct<<< cu_blocks, cu_threads >>>
		(previous->delta_o, previous->output, param->beta, param->dim, 
		param->biased_dim, param->offset, previous->c_network->length, param->size);
}


void cuda_logistic_deriv_output_error(layer* current)
{
	logistic_param *param = (logistic_param*)current->activ_param;
	cu_blocks = (param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_logistic_activ_fcts.deriv_output_error_fct<<< cu_blocks, cu_threads >>>
		(current->delta_o, current->output, current->c_network->target, param->dim, param->biased_dim, 
		param->offset, current->c_network->length, param->size, current->c_network->TC_scale_factor);
	
	current->c_network->cu_inst.cu_logistic_activ_fcts.deriv_fct<<< cu_blocks, cu_threads >>>
		(current->delta_o, current->output, param->beta, param->dim, 
		param->biased_dim, param->offset, current->c_network->length, param->size);
}


void cuda_logistic_output_error(layer* current)
{
	logistic_param *param = (logistic_param*)current->activ_param;
	cu_blocks = (param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_logistic_activ_fcts.output_error_fct<<< cu_blocks, cu_threads >>>
		((float*)current->c_network->output_error, current->output, current->c_network->target,
		param->dim, param->biased_dim, param->offset, current->c_network->length, param->size);
}

//#####################################################
//		 Softmax activation related functions
//#####################################################

void cuda_softmax_activation(layer *current)
{
	softmax_param *param = (softmax_param*)current->activ_param;
	cu_blocks = (current->c_network->batch_size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_softmax_activ_fcts.activ_fct<<< cu_blocks, cu_threads >>>
		(current->output, param->dim, param->biased_dim, param->offset, 
		current->c_network->length, current->c_network->batch_size, param->size);
}


void cuda_softmax_deriv(layer *previous)
{
	printf("ERROR: Softmax activation can not be used in the middle of the network !\n");
	exit(EXIT_FAILURE);
}


void cuda_softmax_deriv_output_error(layer *current)
{
	//use by default a cross entropy error
	softmax_param *param = (softmax_param*)current->activ_param;
	cu_blocks = (param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_softmax_activ_fcts.deriv_output_error_fct<<< cu_blocks, cu_threads >>>
		(current->delta_o, current->output, current->c_network->target,
		param->dim, param->biased_dim, param->offset, current->c_network->length,
		param->size, current->c_network->TC_scale_factor);
}


void cuda_softmax_output_error(layer *current)
{
	//use by default a cross entropy error
	softmax_param *param = (softmax_param*)current->activ_param;
	cu_blocks = (param->size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_softmax_activ_fcts.output_error_fct<<< cu_blocks, cu_threads >>>
		((float*)current->c_network->output_error, current->output, 
		current->c_network->target, param->dim, param->biased_dim, param->offset, 
		current->c_network->length, param->size);
}


void cuda_semi_supervised_gan_deriv_output_error(layer *current, int halved, int reversed)
{
	//First half unsuperfvised fake	
	//Second half supervised true (for now)
	linear_param *param = (linear_param*)current->activ_param;
	/*cu_blocks = (current->c_network->batch_size + cu_threads - 1) / cu_threads;
	current->c_network->cu_inst.cu_auxil_fcts.cu_exp_disc_activation_kernel<<< cu_blocks, cu_threads >>>
		(current->output, param->dim, param->biased_dim, param->offset, 
		current->c_network->length, current->c_network->batch_size, param->size, halved, reversed);
	*/
	cu_blocks = (current->c_network->batch_size + cu_threads - 1) / cu_threads;
	current->c_network->cu_inst.cu_auxil_fcts.cu_exp_disc_deriv_output_kernel<<< cu_blocks, cu_threads >>>
		(current->delta_o, current->output, current->c_network->target,
		param->dim, param->biased_dim, param->offset, current->c_network->length, 
		current->c_network->batch_size, param->size, current->c_network->TC_scale_factor, halved, reversed);
}

//#####################################################
//		 YOLO activation related functions
//#####################################################

void cuda_YOLO_activation(layer *current)
{
	yolo_param *a_param = (yolo_param*)current->activ_param;
	conv_param *c_param = (conv_param*)current->param;
	cu_blocks = (current->c_network->out_size *
			current->c_network->batch_size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_YOLO_activ_fcts.activ_fct<<< cu_blocks, cu_threads >>>
		(current->output, c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] * current->c_network->batch_size,
		a_param->biased_dim*current->c_network->batch_size, *a_param, a_param->size, a_param->class_softmax);
}


void cuda_YOLO_deriv(layer *previous)
{
	printf("ERROR : YOLO activation can not be used in the middle of the network !\n");
	exit(EXIT_FAILURE);
}


void cuda_YOLO_deriv_output_error(layer *current)
{
	yolo_param *a_param = (yolo_param*)current->activ_param;
	conv_param *c_param = (conv_param*)current->param;
	cu_blocks = (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] *
			current->c_network->batch_size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_YOLO_activ_fcts.deriv_output_error_fct<<< cu_blocks, cu_threads >>>
		(current->delta_o, current->output, current->c_network->target, current->c_network->output_dim, 
		c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2], c_param->nb_area[0], c_param->nb_area[1], c_param->nb_area[2], 
		*a_param, c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] * current->c_network->batch_size, 
		current->c_network->TC_scale_factor, current->c_network->iter * current->c_network->train.size);
}


void cuda_YOLO_output_error(layer *current)
{
	yolo_param *a_param = (yolo_param*)current->activ_param;
	conv_param *c_param = (conv_param*)current->param;
	cu_blocks = (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] *
			current->c_network->batch_size + cu_threads - 1) / cu_threads;
	
	current->c_network->cu_inst.cu_YOLO_activ_fcts.output_error_fct<<< cu_blocks, cu_threads >>>
		((float*)current->c_network->output_error, current->output, current->c_network->target, current->c_network->output_dim, 
		c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2], c_param->nb_area[0], c_param->nb_area[1], c_param->nb_area[2], 
		*a_param, c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] * current->c_network->batch_size);
}


void cuda_YOLO_activ_init(layer *current)
{
	float *temp_tab, *temp_tab2, **temp_tab3;
	
	size_t nb_area_flat;

	yolo_param* a_param = (yolo_param*)current->activ_param;
	
	nb_area_flat = ((conv_param*)current->param)->nb_area[0]
		* ((conv_param*)current->param)->nb_area[1]
		* ((conv_param*)current->param)->nb_area[2];
	
	switch(((yolo_param*)a_param)->IoU_type)
	{
		case IOU:
			hipMemcpyFromSymbol(&((yolo_param*)a_param)->c_IoU_fct, device_gpu_IoU_fct, sizeof(pointFunction_gpu_IoU));
			break;
			
		default:
		case GIOU:
			hipMemcpyFromSymbol(&((yolo_param*)a_param)->c_IoU_fct, device_gpu_GIoU_fct, sizeof(pointFunction_gpu_IoU));
			break;
			
		case DIOU:
			hipMemcpyFromSymbol(&((yolo_param*)a_param)->c_IoU_fct, device_gpu_DIoU_fct, sizeof(pointFunction_gpu_IoU));
			break;
		
		case DIOU2:
			hipMemcpyFromSymbol(&((yolo_param*)a_param)->c_IoU_fct, device_gpu_DIoU2_fct, sizeof(pointFunction_gpu_IoU));
			break;
	}
	
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->prior_size,
		((yolo_param*)a_param)->nb_box * 3, 1);
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->noobj_prob_prior,
		((yolo_param*)a_param)->nb_box, 1);
	cuda_convert_table_int(&((yolo_param*)a_param)->cell_size, 3, 1);
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->scale_tab, 6, 1);
	
	temp_tab = ((yolo_param*)a_param)->slopes_and_maxes_tab[0];
	hipMalloc(&temp_tab2, 6 * 3 * sizeof(float));
	hipMemcpy(temp_tab2, temp_tab, 6 * 3 * sizeof(float), hipMemcpyHostToDevice);
	for(int i = 0; i < 6; i++)
		((yolo_param*)a_param)->slopes_and_maxes_tab[i] = &temp_tab2[i*3];
	temp_tab3 = ((yolo_param*)a_param)->slopes_and_maxes_tab;
	hipMalloc(&((yolo_param*)a_param)->slopes_and_maxes_tab, 6 * sizeof(float*));
	hipMemcpy(((yolo_param*)a_param)->slopes_and_maxes_tab, temp_tab3,
			6 * sizeof(float*), hipMemcpyHostToDevice);
	
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->param_ind_scale,
		((yolo_param*)a_param)->nb_param,1);
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->IoU_limits, 8, 1);
	cuda_convert_table_int(&((yolo_param*)a_param)->fit_parts, 6, 1);
	
	hipMalloc((void**)(&((yolo_param*)a_param)->block_state), ((conv_param*)current->param)->nb_filters 
			* nb_area_flat * current->c_network->batch_size * sizeof(hiprandState_t));
	cu_blocks = ((conv_param*)current->param)->nb_filters * current->c_network->batch_size 
		* (size_t)(nb_area_flat  + cu_threads - 1) / cu_threads;
	init_block_state<<< cu_blocks, cu_threads>>>(time(NULL),(hiprandState_t*)((yolo_param*)a_param)->block_state, 
		((conv_param*)current->param)->nb_filters * nb_area_flat * current->c_network->batch_size);
	
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->IoU_monitor,
		2 *((yolo_param*)a_param)->nb_box * current->c_network->batch_size * nb_area_flat, 0);
	cuda_convert_table_int(&((yolo_param*)a_param)->target_cell_mask,
		((yolo_param*)a_param)->max_nb_obj_per_image * current->c_network->batch_size * nb_area_flat, 0);
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->IoU_table,
		((yolo_param*)a_param)->max_nb_obj_per_image * ((yolo_param*)a_param)->nb_box 
		* current->c_network->batch_size * nb_area_flat, 0);
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->dist_prior,
		((yolo_param*)a_param)->max_nb_obj_per_image * ((yolo_param*)a_param)->nb_box 
		* current->c_network->batch_size * nb_area_flat, 0);
	cuda_convert_table_int(&((yolo_param*)a_param)->box_locked,
		((yolo_param*)a_param)->nb_box * current->c_network->batch_size * nb_area_flat, 0);
	cuda_convert_table_FP32((void**)&((yolo_param*)a_param)->box_in_pix,
		6 * ((yolo_param*)a_param)->nb_box * current->c_network->batch_size * nb_area_flat, 0);
}



//#####################################################
//		 GENERAL FUNCTION ASSOCIATIONS
//#####################################################


void init_typed_cuda_activ(network* net)
{
	switch(net->cu_inst.use_cuda_TC)
	{
		default:
		case FP32C_FP32A:
		case TF32C_FP32A:
			typed_cuda_activ_fct_association_FP32(net);
			break;
			
		case FP16C_FP32A:
		case FP16C_FP16A:
			#if defined(GEN_VOLTA) || defined(GEN_AMPERE)
			typed_cuda_activ_fct_association_FP16(net);
			#else
			printf("ERROR: CIANNA not compiled with FP16 compute capability (GEN_VOLTA minimum)\n");
			exit(EXIT_FAILURE);
			#endif
			break;
		
		case BF16C_FP32A:
			#if defined(GEN_AMPERE)
			typed_cuda_activ_fct_association_BF16(net);
			#else
			printf("ERROR: CIANNA not compiled with BF16 compute capability (GEN_AMPERE minimum)\n");
			exit(EXIT_FAILURE);
			#endif
			break;
	}
}


void cuda_define_activation(layer *current)
{	
	switch(current->activation_type)
	{
		case RELU:
			current->activation = cuda_ReLU_activation;
			current->deriv_activation = cuda_ReLU_deriv;
			break;
		
		case LOGISTIC:
			current->activation = cuda_logistic_activation;
			current->deriv_activation = cuda_logistic_deriv;
			break;
			
		case SOFTMAX:
			current->activation = cuda_softmax_activation;
			current->deriv_activation = cuda_softmax_deriv;
			break;
			
		case YOLO:
			current->activation = cuda_YOLO_activation;
			current->deriv_activation = cuda_YOLO_deriv;
			cuda_YOLO_activ_init(current);
			break;
			
		case LINEAR:
		default:
			current->activation = cuda_linear_activation;
			current->deriv_activation = cuda_linear_deriv;
			break;
	}
}


void cuda_deriv_output_error(layer *current)
{
	switch(current->activation_type)
	{
		case RELU:
			cuda_ReLU_deriv_output_error(current);
			break;
		
		case LOGISTIC:
			cuda_logistic_deriv_output_error(current);
			break;
			
		case SOFTMAX:
			cuda_softmax_deriv_output_error(current);
			break;
			
		case YOLO:
			cuda_YOLO_deriv_output_error(current);
			break;
			
		case LINEAR:
		default:
			cuda_linear_deriv_output_error(current);
			break;
	
	}
}

void cuda_output_error_fct(layer* current)
{
	switch(current->activation_type)
	{
		case RELU:
			cuda_ReLU_output_error(current);
			break;
		
		case LOGISTIC:
			cuda_logistic_output_error(current);
			break;
			
		case SOFTMAX:
			cuda_softmax_output_error(current);
			break;
			
		case YOLO:
			cuda_YOLO_output_error(current);
			break;
			
		case LINEAR:
		default:
			cuda_linear_output_error(current);
			break;
	
	}
}




//#####################################################








