#include "hip/hip_runtime.h"


/*
	Copyright (C) 2020 David Cornu
	for the Convolutional Interactive Artificial 
	Neural Networks by/for Astrophysicists (CIANNA) Code
	(https://github.com/Deyht/CIANNA)

	Licensed under the Apache License, Version 2.0 (the "License");
	you may not use this file except in compliance with the License.
	You may obtain a copy of the License at

		http://www.apache.org/licenses/LICENSE-2.0

	Unless required by applicable law or agreed to in writing, software
	distributed under the License is distributed on an "AS IS" BASIS,
	WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
	See the License for the specific language governing permissions and
	limitations under the License.
*/



#include "../prototypes.h"

static int cu_blocks;
static dense_param *d_param;

//public are in prototypes.h

//used to reshape output of Conv layer that has the result of filter 1 continuous for the batch
//convert into all filters continuous for image 1, then image 2, ...
#define cuda_flat_dense(name, type) 																											\
__global__ void cuda_flat_dense_##name																											\
	(void* i_in, void* i_out, float bias, int map_size, int flatten_size, int nb_map, int batch_size, int size)									\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int map_id, image_id, pos;																													\
																																				\
	type* in = (type*) i_in;																													\
	type* out = (type*) i_out;																													\
																																				\
	if(i < size)																																\
	{																																			\
		image_id = i / flatten_size;																											\
		map_id = (i % flatten_size)/map_size;																									\
		pos = (i % flatten_size)%map_size;																										\
																																				\
		if(map_id >= nb_map)																													\
			out[i] = (type) bias;																												\
		else																																	\
			out[i] = in[map_id*(map_size*batch_size) + image_id*map_size + pos];																\
	}																																			\
}


#define cuda_reroll_batch(name, type) 																											\
__global__ void cuda_reroll_batch_##name																										\
	(void* i_in, void* i_out, int map_size, int flatten_size, int nb_map, int batch_size, int size)												\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
	int map_id, image_id, pos;																													\
																																				\
	type* in = (type*) i_in;																													\
	type* out = (type*) i_out;																													\
																																				\
	if(i < size)																																\
	{																																			\
		map_id = i / (map_size*batch_size);																										\
		image_id = (i % (map_size*batch_size))/map_size;																						\
		pos = (i % (map_size*batch_size))%map_size;																								\
																																				\
		out[i] = in[image_id*(flatten_size) + map_id*map_size + pos];																			\
	}																																			\
}


__global__ void cuda_dropout_select_dense(int* mask, int size, int biased_dim, float drop_rate, void* states)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	hiprandState_t* c_states = (hiprandState_t*) states;
	
	float rand;
	if(i < size)
	{
		rand = hiprand_uniform(&c_states[i]);
		if(rand < drop_rate && (i+1) % (biased_dim) != 0)
			mask[i] = 0;
		else
			mask[i] = 1;
	}
}

#define cuda_dropout_apply_dense(name, type) 																									\
__global__ void cuda_dropout_apply_dense_##name(void* i_table, int* mask, int size)																\
{																																				\
	int i = blockIdx.x*blockDim.x + threadIdx.x;																								\
																																				\
	type* table = (type*) i_table;																												\
																																				\
	if(i < size)																																\
		table[i] *= mask[i];																													\
}



cuda_flat_dense(FP32, float);
cuda_reroll_batch(FP32, float);
cuda_dropout_apply_dense(FP32, float);

#if defined(GEN_VOLTA) || defined(GEN_AMPERE) 
cuda_flat_dense(FP16, half);
cuda_reroll_batch(FP16, half);
cuda_dropout_apply_dense(FP16, half);
#endif

#if defined (GEN_AMPERE)
cuda_flat_dense(BF16, hip_bfloat16);
cuda_reroll_batch(BF16, hip_bfloat16);
cuda_dropout_apply_dense(BF16, hip_bfloat16);
#endif



void cuda_dense_init(network *net)
{
	switch(net->cu_inst.use_cuda_TC)
	{
		default:
		case FP32C_FP32A:
		case TF32C_FP32A:
			net->cu_inst.cu_dense_fcts.flat_dense_fct = cuda_flat_dense_FP32;
			net->cu_inst.cu_dense_fcts.reroll_fct = cuda_reroll_batch_FP32;
			net->cu_inst.cu_dense_fcts.drop_apply_fct = cuda_dropout_apply_dense_FP32;
			break;
		
		case FP16C_FP32A:
		case FP16C_FP16A:
			#if defined(GEN_VOLTA) || defined(GEN_AMPERE) 
			net->cu_inst.cu_dense_fcts.flat_dense_fct = cuda_flat_dense_FP16;
			net->cu_inst.cu_dense_fcts.reroll_fct = cuda_reroll_batch_FP16;
			net->cu_inst.cu_dense_fcts.drop_apply_fct = cuda_dropout_apply_dense_FP16;
			#else
			printf("ERROR: CIANNA not compiled with FP16 compute capability (GEN_VOLTA minimum)\n");
			exit(EXIT_FAILURE);
			#endif
			break;

		case BF16C_FP32A:
			#if defined (GEN_AMPERE)
			net->cu_inst.cu_dense_fcts.flat_dense_fct = cuda_flat_dense_BF16;
			net->cu_inst.cu_dense_fcts.reroll_fct = cuda_reroll_batch_BF16;
			net->cu_inst.cu_dense_fcts.drop_apply_fct = cuda_dropout_apply_dense_BF16;
			#else
			printf("ERROR: CIANNA not compiled with BF16 compute capability (GEN_AMPERE minimum)\n");
			exit(EXIT_FAILURE);
			#endif
			break;
	}
}

size_t cuda_convert_dense_layer(layer *current)
{
	d_param = (dense_param*)current->param;
	size_t vram_approx = 0;
	#if defined(GEN_VOLTA) || defined(GEN_AMPERE)
	float* temp_tab;
	#endif
	
	network* net = current->c_network;
	
	if(current->previous != NULL)
	{
		switch(current->previous->type)
		{	
			case CONV:
				vram_approx += cuda_convert_table(net, &(d_param->flat_input), d_param->in_size*net->batch_size,0);
				if(!net->inference_only)
					vram_approx += cuda_convert_table(net, &(d_param->flat_delta_o),
						(((conv_param*)current->previous->param)->nb_area[0] 
							* ((conv_param*)current->previous->param)->nb_area[1] 
							* ((conv_param*)current->previous->param)->nb_area[2] 
							* ((conv_param*)current->previous->param)->nb_filters + 1) 
							* net->batch_size,0);
				break;
				
			case POOL:
				vram_approx += cuda_convert_table(net, &(d_param->flat_input), d_param->in_size * net->batch_size,0);
				if(!net->inference_only)
					vram_approx += cuda_convert_table(net, &(d_param->flat_delta_o),
						(((pool_param*)current->previous->param)->nb_area[0]
							* ((pool_param*)current->previous->param)->nb_area[1] 
							* ((pool_param*)current->previous->param)->nb_area[2] 
							* ((pool_param*)current->previous->param)->nb_maps + 1) 
							* net->batch_size,0);
				break;
				
			case DENSE:
			default:
				d_param->flat_delta_o = current->previous->delta_o;
				break;
		}
	}
	
	switch(net->cu_inst.use_cuda_TC)
	{
		default:
		case FP32C_FP32A:
		case TF32C_FP32A:
			vram_approx += cuda_convert_table(net, &(d_param->weights), d_param->in_size*(d_param->nb_neurons+1),0);
			d_param->FP32_weights = d_param->weights;
			break;
		
		case FP16C_FP32A:
		case FP16C_FP16A:
			#if defined(GEN_VOLTA) || defined(GEN_AMPERE)
			temp_tab = (float*)d_param->weights;
			hipMalloc(&(d_param->FP32_weights), d_param->in_size*(d_param->nb_neurons+1)*sizeof(float));
			vram_approx += d_param->in_size*(d_param->nb_neurons+1)*sizeof(float);
			hipMemcpy(d_param->FP32_weights, temp_tab, d_param->in_size 
				* (d_param->nb_neurons+1) * sizeof(float),hipMemcpyHostToDevice);
			free(temp_tab);
			hipMalloc(&(d_param->weights), d_param->in_size*(d_param->nb_neurons+1)*sizeof(half));
			vram_approx += d_param->in_size*(d_param->nb_neurons+1)*sizeof(half);
			#endif
			break;
			
		case BF16C_FP32A:
			#if defined(GEN_AMPERE) 
			temp_tab = (float*)d_param->weights;
			hipMalloc(&(d_param->FP32_weights), d_param->in_size*(d_param->nb_neurons+1)*sizeof(float));
			vram_approx += d_param->in_size*(d_param->nb_neurons+1)*sizeof(float);
			hipMemcpy(d_param->FP32_weights, temp_tab, d_param->in_size 
				* (d_param->nb_neurons+1) * sizeof(float),hipMemcpyHostToDevice);
			free(temp_tab);
			hipMalloc(&(d_param->weights),d_param->in_size*(d_param->nb_neurons+1)*sizeof(hip_bfloat16));
			vram_approx += d_param->in_size*(d_param->nb_neurons+1)*sizeof(hip_bfloat16);
			#endif
			break;
	}
	
	vram_approx += cuda_convert_table(net, &(current->output), (d_param->nb_neurons+1) 
		* net->batch_size,0);
		
	if(current->dropout_rate > 0.01f)
	{
		vram_approx += cuda_convert_table_int(&(d_param->dropout_mask), (d_param->nb_neurons+1) * net->batch_size, 0);
		hipMalloc((void**) &d_param->block_state, ((d_param->nb_neurons+1) * net->batch_size) * sizeof(hiprandState_t));
		vram_approx += ((d_param->nb_neurons+1) * net->batch_size) * sizeof(hiprandState_t);
		cu_blocks = ((d_param->nb_neurons+1) * net->batch_size + cu_threads - 1) / cu_threads;
		init_block_state<<< cu_blocks, cu_threads>>>(time(NULL),(hiprandState_t*)d_param->block_state, (d_param->nb_neurons+1)*net->batch_size);
	}
	
	if(!net->inference_only)
	{
		vram_approx += cuda_convert_table(net, &(d_param->update), d_param->in_size*(d_param->nb_neurons+1),0);
		vram_approx += cuda_convert_table(net, &(current->delta_o), (d_param->nb_neurons+1) * net->batch_size,0);
	}

	return vram_approx;
}


void cuda_forward_dense_layer(layer *current)
{
	int nb_area_w, nb_area_h, nb_area_d, depth;
	
	void *ref_input;
	void *w_alpha;
	float w_f_alpha;
	float c_dr = 0.0f;
	
	network* net = current->c_network;
	
	#if defined(GEN_VOLTA) || defined(GEN_AMPERE)
	half w_h_alpha;
	if(net->cu_inst.use_cuda_TC == FP16C_FP16A)
		w_alpha = &w_h_alpha;	
	else
	#endif
		w_alpha = &w_f_alpha;
	
	if(net->length == 0)
		return;
	
	d_param = (dense_param*) current->param;
	
	cuda_master_weight_copy(net, (float*)d_param->FP32_weights, d_param->weights, 
		d_param->in_size*(d_param->nb_neurons+1));
	
	if(current->previous == NULL)
		current->input = net->input;
	
	ref_input = current->input;
		
	if(current->previous != NULL && current->previous->type != DENSE)
	{
		//Use a converted (flatten) input if needed
		switch(current->previous->type)
		{
			case POOL:
				nb_area_w = ((pool_param*)current->previous->param)->nb_area[0];
				nb_area_h = ((pool_param*)current->previous->param)->nb_area[1];
				nb_area_d = ((pool_param*)current->previous->param)->nb_area[2];
				depth = ((pool_param*)current->previous->param)->nb_maps;
				break;
				
			case CONV:
			default:
				nb_area_w = ((conv_param*)current->previous->param)->nb_area[0];
				nb_area_h = ((conv_param*)current->previous->param)->nb_area[1];
				nb_area_d = ((conv_param*)current->previous->param)->nb_area[2];
				depth = ((conv_param*)current->previous->param)->nb_filters;
				break;
		}
		
		cu_blocks = ((nb_area_w * nb_area_h * nb_area_d * depth + 1) 
			* net->batch_size + cu_threads - 1) / cu_threads;
		
		net->cu_inst.cu_dense_fcts.flat_dense_fct<<< cu_blocks, cu_threads >>>(current->input, 
			d_param->flat_input, current->bias_value, nb_area_w * nb_area_h * nb_area_d ,
			nb_area_w * nb_area_h * nb_area_d * depth + 1, depth, net->batch_size, 
			(nb_area_w * nb_area_h * nb_area_d * depth + 1) * net->batch_size);
		
		ref_input = d_param->flat_input;
	}
	
	if(net->is_inference && net->inference_drop_mode == AVG_MODEL && current->previous != NULL)
	{
		c_dr = current->previous->dropout_rate;
		
		if(c_dr <= 0.01f)
		{
			if(net->cu_inst.use_cuda_TC == FP16C_FP16A)
				*((half*)w_alpha) = 1.0f;
			else
				*((float*)w_alpha) = 1.0f;
		}
		else
		{
			c_dr = ((d_param->in_size-1)*(1.0f-c_dr)+1)/d_param->in_size;
			if(net->cu_inst.use_cuda_TC == FP16C_FP16A)
				*((half*)w_alpha) = c_dr;	
			else
				*((float*)w_alpha) = c_dr;
		}
	}
	else
	{
		if(net->cu_inst.use_cuda_TC == FP16C_FP16A)
			*((half*)w_alpha) = 1.0f;	
		else
			*((float*)w_alpha) = 1.0f;
	}
	
	hipblasGemmEx(cu_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, d_param->nb_neurons+1, 
		net->batch_size, d_param->in_size, w_alpha, d_param->weights, cuda_data_type, 
		d_param->nb_neurons+1, ref_input, cuda_data_type, d_param->in_size, cu_beta, 
		current->output, cuda_data_type, d_param->nb_neurons+1, cuda_compute_type, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	
	current->activation(current);

	if(current->dropout_rate > 0.01f && (!net->is_inference || net->inference_drop_mode == MC_MODEL))
	{
		// Must check performance impact -> the present approach is due to the hiprand behavior
		cu_blocks = ((d_param->nb_neurons+1) * net->batch_size + cu_threads - 1) / cu_threads;
		cuda_dropout_select_dense<<<cu_blocks, cu_threads>>>(d_param->dropout_mask, (d_param->nb_neurons+1) * net->batch_size, 
			d_param->nb_neurons+1, current->dropout_rate, (hiprandState_t*) d_param->block_state);	
		
		net->cu_inst.cu_dense_fcts.drop_apply_fct<<<cu_blocks, cu_threads>>>(current->output, 
			d_param->dropout_mask, (d_param->nb_neurons+1) * net->batch_size);
	}
}


void cuda_backward_dense_layer(layer* current)
{
	int nb_area_w, nb_area_h, nb_area_d, depth;
	void* ref_input;

	network* net = current->c_network;

	d_param = (dense_param*) current->param;	
	
	if(current->dropout_rate > 0.01f)
	{
		cu_blocks = ((d_param->nb_neurons+1) * net->batch_size + cu_threads - 1) / cu_threads;
		
		net->cu_inst.cu_dense_fcts.drop_apply_fct<<<cu_blocks, cu_threads>>>(current->delta_o, 
			d_param->dropout_mask, (d_param->nb_neurons+1) * net->batch_size);
	}
	
	//######################## ERROR PROPAGATION ########################

	ref_input = current->input;
	
	//skip error prop if previous is the input layer
	if(current->previous != NULL)
	{
		hipblasGemmEx(cu_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, d_param->in_size, 
			net->batch_size, d_param->nb_neurons+1, cu_alpha, d_param->weights, cuda_data_type, 
			d_param->nb_neurons+1, current->delta_o, cuda_data_type, d_param->nb_neurons+1, cu_beta, 
			d_param->flat_delta_o, cuda_data_type, d_param->in_size, cuda_compute_type,
			CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		//if previous layer is dense then flat_delta_o = previous->delta_o
		
		if(current->previous->type == POOL || current->previous->type == CONV)
		{
			switch(current->previous->type)
			{
				case POOL:
					nb_area_w = ((pool_param*)current->previous->param)->nb_area[0];
					nb_area_h = ((pool_param*)current->previous->param)->nb_area[1];
					nb_area_d = ((pool_param*)current->previous->param)->nb_area[2];
					depth = ((pool_param*)current->previous->param)->nb_maps;
					break;
			
				case CONV:
				default:
					nb_area_w = ((conv_param*)current->previous->param)->nb_area[0];
					nb_area_h = ((conv_param*)current->previous->param)->nb_area[1];
					nb_area_d = ((conv_param*)current->previous->param)->nb_area[2];
					depth = ((conv_param*)current->previous->param)->nb_filters;
					break;
			}
			
			//Need to unroll delta_o to already be in the proper format for deriv calculation
			cu_blocks = (nb_area_w * nb_area_h * nb_area_d * depth 
				* net->batch_size + cu_threads - 1) / cu_threads;
			
			net->cu_inst.cu_dense_fcts.reroll_fct<<< cu_blocks, cu_threads >>>(d_param->flat_delta_o, 
				current->previous->delta_o, nb_area_w * nb_area_h * nb_area_d, 
				nb_area_w * nb_area_h * nb_area_d * depth + 1, depth, net->batch_size,
				nb_area_w * nb_area_h * nb_area_d * depth * net->batch_size);
		}
		
		current->previous->deriv_activation(current->previous);
	}
		
	//########################  WEIGHTS UPDATE   ########################
	if(current->previous != NULL && current->previous->type != DENSE)
		ref_input = d_param->flat_input;
	
	if(!current->frozen)
	{
		set_cu_learning_rate_and_momentum(net);
		
		hipblasGemmEx(cu_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, d_param->nb_neurons+1, d_param->in_size,
			net->batch_size, cu_learning_rate, current->delta_o, cuda_data_type, 
			d_param->nb_neurons+1, ref_input, cuda_data_type, d_param->in_size, cu_momentum,
			d_param->update, cuda_data_type, d_param->nb_neurons+1, cuda_compute_type, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
		
		cuda_update_weights(net, d_param->FP32_weights, d_param->update, net->learning_rate*net->weight_decay, 
			d_param->in_size * (d_param->nb_neurons+1));
	}
}


void cuda_dense_define(layer *current)
{
	current->forward = cuda_forward_dense_layer;
	current->backprop = cuda_backward_dense_layer;
}







