#include "hip/hip_runtime.h"


/*
	Copyright (C) 2020 David Cornu
	for the Convolutional Interactive Artificial 
	Neural Networks by/for Astrophysicists (CIANNA) Code
	(https://github.com/Deyht/CIANNA)

	Licensed under the Apache License, Version 2.0 (the "License");
	you may not use this file except in compliance with the License.
	You may obtain a copy of the License at

		http://www.apache.org/licenses/LICENSE-2.0

	Unless required by applicable law or agreed to in writing, software
	distributed under the License is distributed on an "AS IS" BASIS,
	WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
	See the License for the specific language governing permissions and
	limitations under the License.
*/



#include "../prototypes.h"

static int cu_blocks;
static conv_param *c_param;

//public are in prototypes.h

//private
void cuda_forward_conv_layer(layer *current);
void cuda_backward_conv_layer(layer *current);

__global__ void cuda_add_bias_im2col_FP32(float* output, float bias_value, int flat_f_size, int size);
__global__ void cuda_add_bias_im2col_FP16(half* output, float bias_value, int flat_f_size, int size);
__global__ void cuda_add_bias_im2col_BF16(hip_bfloat16* output, float bias_value, int flat_f_size, int size);
__global__ void cuda_rotate_filter_matrix_FP32(float* in, float* out, int nb_rows, int TC_padding, int depth_size, int nb_filters_in, int len);
__global__ void cuda_rotate_filter_matrix_FP16(half* in, half* out, int nb_rows, int TC_padding, int depth_size, int nb_filters_in, int len);
__global__ void cuda_rotate_filter_matrix_BF16(hip_bfloat16* in, hip_bfloat16* out, int nb_rows, int TC_padding, int depth_size, int nb_filters_in, int len);
/*
__global__ void cuda_unroll_conv_FP32(float* in, float* out, int map_size, int flatten_size, int nb_map, int batch_size, int size);
__global__ void cuda_unroll_conv_FP16(half* in, half* out, int map_size, int flatten_size, int nb_map, int batch_size, int size);
__global__ void cuda_reroll_delta_o_FP32(float* in, float* out, int map_size, int flatten_size, int nb_map, int batch_size, int size);
__global__ void cuda_reroll_delta_o_FP16(half* in, half* out, int map_size, int flatten_size, int nb_map, int batch_size, int size);
*/
__global__ void im2col_kernel_v4_FP32(float* output, float* input, int image_size, int flat_image_size, int stride_w, int stride_h ,int stride_d, int padding_w, int padding_h, int padding_d, int internal_padding_w, int internal_padding_h, int internal_padding_d, int channel, int channel_padding, int image_padding, int TC_padding, int batch_size, int f_size_w, int f_size_h, int f_size_d, int flat_f_size, int w_size, int h_size, int d_size, int nb_area_w, int nb_area_h, int bias_in, int bias_out);
__global__ void im2col_kernel_v4_FP16(half* output, half* input, int image_size, int flat_image_size, int stride_w, int stride_h ,int stride_d, int padding_w, int padding_h, int padding_d, int internal_padding_w, int internal_padding_h, int internal_padding_d, int channel, int channel_padding, int image_padding, int TC_padding, int batch_size, int f_size_w, int f_size_h, int f_size_d, int flat_f_size, int w_size, int h_size, int d_size, int nb_area_w, int nb_area_h, int bias_in, int bias_out);
__global__ void im2col_kernel_v4_BF16(hip_bfloat16* output, hip_bfloat16* input, int image_size, int flat_image_size, int stride_w, int stride_h ,int stride_d, int padding_w, int padding_h, int padding_d, int internal_padding_w, int internal_padding_h, int internal_padding_d, int channel, int channel_padding, int image_padding, int TC_padding, int batch_size, int f_size_w, int f_size_h, int f_size_d, int flat_f_size, int w_size, int h_size, int d_size, int nb_area_w, int nb_area_h, int bias_in, int bias_out);
/*__global__ void im2col_kernel_v5_FP16(half* output, half* input, int image_size, int flat_image_size, int stride, int padding, int internal_padding, int depth, int depth_padding, int image_padding, int batch_size, int f_size, int flat_f_size, int w_size, int nb_area_w, int bias);*/

__global__ void init_block_state_conv(unsigned int seed, hiprandState_t* states);
__global__ void cuda_dropout_select_conv(int* mask, int size, float drop_rate, hiprandState_t* states);
__global__ void cuda_dropout_apply_conv_FP32(float* table, int batch_size, int dim, int* mask, int size);
__global__ void cuda_dropout_apply_conv_FP16(half* table, int batch_size, int dim, int* mask, int size);
__global__ void cuda_dropout_apply_conv_BF16(hip_bfloat16* table, int batch_size, int dim, int* mask, int size);


void cuda_conv_define(layer *current)
{
	current->forward = cuda_forward_conv_layer;
	current->backprop = cuda_backward_conv_layer;
}

long long int cuda_convert_conv_layer(layer *current)
{
	c_param = (conv_param*)current->param;
	long long int vram_approx = 0;
	float* temp_tab;

	switch(current->c_network->use_cuda_TC)
	{
		default:
		case FP32C_FP32A:
		case TF32C_FP32A:
			vram_approx += cuda_convert_table(current->c_network, &(c_param->filters), c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding));
			break;
		
		case FP16C_FP32A:
		case FP16C_FP16A:
			temp_tab = (float*)c_param->filters;
			hipMalloc(&(c_param->FP32_filters), c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding)*sizeof(float));
			vram_approx += c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding)*sizeof(float);
			hipMemcpy(c_param->FP32_filters, temp_tab, c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding) * sizeof(float),hipMemcpyHostToDevice);
			free(temp_tab);
			hipMalloc(&(c_param->filters), c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding)*sizeof(half));
			vram_approx += c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding)*sizeof(half);
			break;
		
		case BF16C_FP32A:
			temp_tab = (float*)c_param->filters;
			hipMalloc(&(c_param->FP32_filters), c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding)*sizeof(float));
			vram_approx += c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding)*sizeof(float);
			hipMemcpy(c_param->FP32_filters, temp_tab, c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding) * sizeof(float),hipMemcpyHostToDevice);
			free(temp_tab);
			hipMalloc(&(c_param->filters), c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding)*sizeof(hip_bfloat16));
			vram_approx += c_param->nb_filters 
				* (c_param->flat_f_size + c_param->TC_padding)*sizeof(hip_bfloat16);
			break;
	}

	vram_approx += cuda_convert_table(current->c_network, &(c_param->update), c_param->nb_filters 
		* (c_param->flat_f_size + c_param->TC_padding));
	
	vram_approx += cuda_convert_table(current->c_network, &(c_param->rotated_filters), c_param->nb_filters 
		* (c_param->flat_f_size-1));
	
	vram_approx += cuda_convert_table(current->c_network, &(current->output), c_param->nb_filters 
		* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]) * current->c_network->batch_size);
	vram_approx += cuda_convert_table(current->c_network, &(current->delta_o), c_param->nb_filters 
		* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]) * current->c_network->batch_size);
	
	vram_approx += cuda_convert_table(current->c_network, &(c_param->im2col_input), 
		((c_param->flat_f_size + c_param->TC_padding) * c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]) 
		* current->c_network->batch_size);
	vram_approx += cuda_convert_table(current->c_network, &(c_param->im2col_delta_o), 
		(long long int) current->c_network->batch_size * (c_param->prev_size[0]*c_param->prev_size[1]*c_param->prev_size[2]) 
		* (c_param->f_size[0] * c_param->f_size[1] * c_param->f_size[2] * c_param->nb_filters));
	
	if(c_param->dropout_rate > 0.01)
	{
		vram_approx += cuda_convert_table_int(current->c_network, &(c_param->dropout_mask), c_param->nb_filters * (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
		hipMalloc((void**) &c_param->block_state, (c_param->nb_filters * (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2])) * sizeof(hiprandState_t));
		vram_approx += (c_param->nb_filters * (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2])) * sizeof(hiprandState_t);
		cu_blocks = (c_param->nb_filters * (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
		init_block_state_conv<<< cu_blocks, 1>>>(time(NULL),(hiprandState_t*)c_param->block_state);
	}
	
	return vram_approx;
}


void cuda_forward_conv_layer(layer *current)
{
	int depth_padding;
	int image_padding;
	int im2col_prev_bias;
	int dim_a, dim_b, dim_c;
	
	void *w_alpha;
	float w_f_alpha;
	half w_h_alpha;
	
	if(current->c_network->use_cuda_TC == FP16C_FP16A)
		w_alpha = &w_h_alpha;
	else
		w_alpha = &w_f_alpha;
	float c_dr;

	if(current->c_network->length == 0)
		return;
	c_param = (conv_param*) current->param;
	
	if(current->previous == NULL)
	{
		//if previous layer is input layer then remove the added bias on the image
		//and interpret it as continuous RGB images
		//size in line format
		depth_padding = c_param->prev_size[0] * c_param->prev_size[1] * c_param->prev_size[2];
		image_padding = c_param->prev_size[0] * c_param->prev_size[1] * c_param->prev_size[2] * c_param->prev_depth;
		current->input = current->c_network->input;
		im2col_prev_bias = 1;
	}
	else
	{
		//if previous layer is a CONV (or pool) then the format is all image in R, then all image in B, ...
		//it also not contain a bias directly in the image
		depth_padding = c_param->prev_size[0] * c_param->prev_size[1] * c_param->prev_size[2] * current->c_network->batch_size;
		image_padding = c_param->prev_size[0] * c_param->prev_size[1] * c_param->prev_size[2];
		im2col_prev_bias = 0;
		current->input = current->previous->output;
	}
	
	switch(current->c_network->use_cuda_TC)
	{
		default:
		case FP32C_FP32A:
		case TF32C_FP32A:
			//nothing to do
			break;
		
		case FP16C_FP32A:
		case FP16C_FP16A:
			cuda_master_weight_FP32_to_FP16((float*)c_param->FP32_filters, (half*)c_param->filters, 
				c_param->nb_filters * (c_param->flat_f_size + c_param->TC_padding));
			break;
				
		case BF16C_FP32A:
			cuda_master_weight_FP32_to_BF16((float*)c_param->FP32_filters, (hip_bfloat16*)c_param->filters, 
				c_param->nb_filters * (c_param->flat_f_size + c_param->TC_padding));
			break;
	}
	
	if(current->c_network->batch_size <= 2)
		dim_c = 1;
	//else if(current->c_network->batch_size <= 6)
	//	dim_c = 2;
	else
		dim_c = 2;

	if(c_param->nb_filters <= 8)
		dim_b = 4;
	else
		dim_b = 8;
		
	if(c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] <= 8)
		dim_a = 4;
	else
		dim_a = 8;
	
	//dim_c = 1; dim_b = 1; dim_a = 32;
	
	dim3 threadsPerBlock2(dim_a, dim_b, dim_c);
	//create numBlocks regarding the layer dimensions
	dim3 numBlocks2(((c_param->prev_size[0] * c_param->prev_size[1] * c_param->prev_size[2]) + threadsPerBlock2.x - 1) / threadsPerBlock2.x,
    	(c_param->prev_depth + threadsPerBlock2.y - 1) / threadsPerBlock2.y,
    	(current->c_network->batch_size + threadsPerBlock2.z - 1) / threadsPerBlock2.z);
	
	switch(current->c_network->use_cuda_TC)
	{
		default:
		case FP32C_FP32A:
		case TF32C_FP32A:
			im2col_kernel_v4_FP32<<< numBlocks2, threadsPerBlock2 >>>((float*)c_param->im2col_input,
				(float*)current->input, c_param->prev_size[0]*c_param->prev_size[1]*c_param->prev_size[2], 
				c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] * 
				(c_param->flat_f_size + c_param->TC_padding), c_param->stride[0], c_param->stride[1], c_param->stride[2],
				c_param->padding[0], c_param->padding[1], c_param->padding[2], 0, 0 ,0, 
				c_param->prev_depth, depth_padding, image_padding, c_param->TC_padding, current->c_network->batch_size, 
				c_param->f_size[0], c_param->f_size[1], c_param->f_size[2], c_param->flat_f_size, 
				c_param->prev_size[0], c_param->prev_size[1], c_param->prev_size[2], 
				c_param->nb_area[0], c_param->nb_area[1], im2col_prev_bias, 1);
			break;
			
		case FP16C_FP32A:
		case FP16C_FP16A:
			im2col_kernel_v4_FP16<<< numBlocks2, threadsPerBlock2 >>>((half*)c_param->im2col_input,
				(half*)current->input, c_param->prev_size[0]*c_param->prev_size[1]*c_param->prev_size[2], 
				c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] * 
				(c_param->flat_f_size + c_param->TC_padding), c_param->stride[0], c_param->stride[1], c_param->stride[2],
				c_param->padding[0], c_param->padding[1], c_param->padding[2], 0, 0 ,0, 
				c_param->prev_depth, depth_padding, image_padding, c_param->TC_padding, current->c_network->batch_size, 
				c_param->f_size[0], c_param->f_size[1], c_param->f_size[2], c_param->flat_f_size, 
				c_param->prev_size[0], c_param->prev_size[1], c_param->prev_size[2], 
				c_param->nb_area[0], c_param->nb_area[1], im2col_prev_bias, 1);
			break;
			
		case BF16C_FP32A:
			im2col_kernel_v4_BF16<<< numBlocks2, threadsPerBlock2 >>>((hip_bfloat16*)c_param->im2col_input,
				(hip_bfloat16*)current->input, c_param->prev_size[0]*c_param->prev_size[1]*c_param->prev_size[2], 
				c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] *
				(c_param->flat_f_size + c_param->TC_padding), c_param->stride[0], c_param->stride[1], c_param->stride[2],
				c_param->padding[0], c_param->padding[1], c_param->padding[2], 0, 0 ,0, 
				c_param->prev_depth, depth_padding, image_padding, c_param->TC_padding, current->c_network->batch_size, 
				c_param->f_size[0], c_param->f_size[1], c_param->f_size[2], c_param->flat_f_size, 
				c_param->prev_size[0], c_param->prev_size[1], c_param->prev_size[2],
				c_param->nb_area[0], c_param->nb_area[1], im2col_prev_bias, 1);
			break;
	}

	if(current->c_network->is_inference && current->c_network->inference_drop_mode == AVG_MODEL && current->previous != NULL)
	{
		if(current->previous->type == CONV)
			c_dr = ((conv_param*)current->previous->param)->dropout_rate;
		else if(current->previous->type == POOL)
			c_dr = ((pool_param*)current->previous->param)->dropout_rate;
		else
			c_dr = 0.0f;
		c_dr = 1.0f - (((c_param->flat_f_size-1)*(1.0f-c_dr) + 1)/c_param->flat_f_size);
		//w_alpha = (1.0f - c_dr);       //account for the bias node that is never dropped
		//bias_fact = (float)(c_param->flat_f_size)/(float)(c_param->flat_f_size-1);
		if(current->c_network->use_cuda_TC == FP16C_FP16A)
			*((half*)w_alpha) = (1.0f/(1.0 + c_dr));	
		else
			*((float*)w_alpha) = (1.0f/(1.0 + c_dr));
	}
	else
	{
		if(current->c_network->use_cuda_TC == FP16C_FP16A)
			*((half*)w_alpha) = 1.0f;	
		else
			*((float*)w_alpha) = 1.0f;
	}

	//Input X filters matrix multiplication for the all batch
	hipblasGemmEx(cu_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, current->c_network->batch_size 
		* (c_param->nb_area[0]*c_param->nb_area[1]*c_param->nb_area[2]), c_param->nb_filters,
		(c_param->flat_f_size + c_param->TC_padding), w_alpha, c_param->im2col_input, cuda_data_type,
		(c_param->flat_f_size + c_param->TC_padding), c_param->filters, cuda_data_type,  
		(c_param->flat_f_size + c_param->TC_padding), cu_beta, current->output, cuda_data_type,
		current->c_network->batch_size * (c_param->nb_area[0]*c_param->nb_area[1]*c_param->nb_area[2]),
		cuda_compute_type, HIPBLAS_GEMM_DEFAULT);
	
	//Proceed to activation of the given maps regarding the activation parameter
	current->activation(current);
	
	if(c_param->dropout_rate > 0.01 && (!current->c_network->is_inference || current->c_network->inference_drop_mode == MC_MODEL))
	{
		cu_blocks = (c_param->nb_filters * (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
		cuda_dropout_select_conv<<<cu_blocks, 1>>>(c_param->dropout_mask, c_param->nb_filters 
			* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]), 
			c_param->dropout_rate, (hiprandState_t*) c_param->block_state);	
		
		dim3 threadsPerBlock(32, 8);
		dim3 numBlocks((c_param->nb_filters * (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2])
			+ threadsPerBlock.x - 1) / threadsPerBlock.x,
			(current->c_network->batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);
		
		switch(current->c_network->use_cuda_TC)
		{
			default:
			case FP32C_FP32A:
			case TF32C_FP32A:
				cuda_dropout_apply_conv_FP32<<<numBlocks, threadsPerBlock>>>((float*)current->output, 
					current->c_network->batch_size, (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]),
					c_param->dropout_mask, c_param->nb_filters 
					* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
				break;
				
			case FP16C_FP32A:
			case FP16C_FP16A:
				cuda_dropout_apply_conv_FP16<<<numBlocks, threadsPerBlock>>>((half*)current->output, 
					current->c_network->batch_size, (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]),
					c_param->dropout_mask, c_param->nb_filters 
					* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
				break;
			
			case BF16C_FP32A:
				cuda_dropout_apply_conv_BF16<<<numBlocks, threadsPerBlock>>>((hip_bfloat16*)current->output, 
					current->c_network->batch_size, (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]),
					c_param->dropout_mask, c_param->nb_filters 
					* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
				break;
		}
	}
}


void cuda_backward_conv_layer(layer *current)
{
	int k;
	int depth_padding;
	int *back_padding;
	int image_padding;
	int flat_f_size;
	int dim_a, dim_b, dim_c;

	back_padding = (int*) calloc(3, sizeof(int));

	c_param = (conv_param*) current->param;
	
	if(c_param->dropout_rate > 0.01)
	{
		dim3 threadsPerBlock(32, 8);
		dim3 numBlocks((c_param->nb_filters * (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]) + threadsPerBlock.x - 1) / threadsPerBlock.x,
			(current->c_network->batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);
		
		switch(current->c_network->use_cuda_TC)
		{
			default:
			case FP32C_FP32A:
			case TF32C_FP32A:
				cuda_dropout_apply_conv_FP32<<<numBlocks, threadsPerBlock>>>((float*)current->delta_o, 
					current->c_network->batch_size, (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]), 
					c_param->dropout_mask, c_param->nb_filters 
					* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
				break;
				
			case FP16C_FP32A:
			case FP16C_FP16A:
				cuda_dropout_apply_conv_FP16<<<numBlocks, threadsPerBlock>>>((half*)current->delta_o, 
					current->c_network->batch_size, (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]), 
					c_param->dropout_mask, c_param->nb_filters 
					* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
				break;
			
			case BF16C_FP32A:
				cuda_dropout_apply_conv_BF16<<<numBlocks, threadsPerBlock>>>((hip_bfloat16*)current->delta_o, 
					current->c_network->batch_size, (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]), 
					c_param->dropout_mask, c_param->nb_filters 
					* (c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2]));
				break;
		}
	}
	
	//######################## ERROR PROPAGATION ########################
	
	//skip error prop if previous is the input layer
	if(current->previous != NULL)
	{
		//rotate the filters
		//so the new matrix can be considered as flat_filter_size * current->c_network->batch_size rows against input_depth
		cu_blocks = (c_param->nb_filters * (c_param->flat_f_size+c_param->TC_padding) + cu_threads - 1) / cu_threads;
		
		switch(current->c_network->use_cuda_TC)
		{
			default:
			case FP32C_FP32A:
			case TF32C_FP32A:
				cuda_rotate_filter_matrix_FP32<<< cu_blocks, cu_threads >>>((float*)c_param->filters, 
					(float*)c_param->rotated_filters, (c_param->flat_f_size+c_param->TC_padding), 
					c_param->TC_padding, c_param->f_size[0] * c_param->f_size[1] * c_param->f_size[2],
					c_param->nb_filters, c_param->nb_filters*(c_param->flat_f_size+c_param->TC_padding));
				break;
				
			case FP16C_FP32A:
			case FP16C_FP16A:
				cuda_rotate_filter_matrix_FP16<<< cu_blocks, cu_threads >>>((half*)c_param->filters,
					(half*)c_param->rotated_filters, (c_param->flat_f_size+c_param->TC_padding),
					c_param->TC_padding, c_param->f_size[0] * c_param->f_size[1] * c_param->f_size[2],
					 c_param->nb_filters, c_param->nb_filters*(c_param->flat_f_size+c_param->TC_padding));
				break;
				
			case BF16C_FP32A:
				cuda_rotate_filter_matrix_BF16<<< cu_blocks, cu_threads >>>((hip_bfloat16*)c_param->filters,
					(hip_bfloat16*)c_param->rotated_filters, (c_param->flat_f_size+c_param->TC_padding),
					c_param->TC_padding, c_param->f_size[0] * c_param->f_size[1] * c_param->f_size[2],
					 c_param->nb_filters, c_param->nb_filters*(c_param->flat_f_size+c_param->TC_padding));
				break;
		}
		
		//In the backward formalism we asume continuous images (the activation maps)
		//the backprop process generate bias nodes so they must be taken into account
		
		//Warning : the convolution processed is reversed using full convolution with padding
		//this means that the meaning of nb_area_w/h and prev_size_w/h are reversed in the following operations
		
		depth_padding = c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] * current->c_network->batch_size;
		image_padding = c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2];
		flat_f_size = c_param->f_size[0] * c_param->f_size[1] * c_param->f_size[2] * c_param->nb_filters;
		
		for(k = 0; k < 3; k++)
		{
			back_padding[k] =  c_param->f_size[k] -  c_param->padding[k] - 1;
			if(back_padding[k] < 0)
				back_padding[k] = 0;
		}
		
		//Note : having higher dimensions on the left dim3 dim(a,b,c) grants better results
		if(current->c_network->batch_size <= 2)
			dim_c = 1;
		else
			dim_c = 2;
		
		if(c_param->nb_filters <= 8)
			dim_b = 4;
		else
			dim_b = 16;
			
		if(c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] <= 8)
			dim_a = 4;
		else
			dim_a = 16;
			
		//dim_c = 1; dim_b = 1; dim_a = 32;
		
		dim3 threadsPerBlock2(dim_a, dim_b, dim_c);
		//create numBlocks regarding the layer dimensions
		dim3 numBlocks2((c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] + threadsPerBlock2.x - 1) / threadsPerBlock2.x,
			(c_param->nb_filters + threadsPerBlock2.y - 1) / threadsPerBlock2.y,
			(current->c_network->batch_size + threadsPerBlock2.z - 1) / threadsPerBlock2.z);
		
		switch(current->c_network->use_cuda_TC)
		{
			default:
			case FP32C_FP32A:
			case TF32C_FP32A:
				im2col_kernel_v4_FP32<<< numBlocks2, threadsPerBlock2>>>((float*)c_param->im2col_delta_o,
					(float*)current->delta_o, c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2], 
					(c_param->prev_size[0] * c_param->prev_size[1] * c_param->prev_size[2]) * flat_f_size, 
					1, 1, 1, back_padding[0], back_padding[1], back_padding[2], 
					c_param->stride[0] - 1 , c_param->stride[1] - 1 , c_param->stride[2] - 1 ,
					c_param->nb_filters, depth_padding, image_padding, 0, current->c_network->batch_size,
					c_param->f_size[0], c_param->f_size[1], c_param->f_size[2], flat_f_size, 
					c_param->nb_area[0], c_param->nb_area[1], c_param->nb_area[2], 
					c_param->prev_size[0], c_param->prev_size[1], 0, 0);
				break;
				
			case FP16C_FP32A:
			case FP16C_FP16A:
				im2col_kernel_v4_FP16<<< numBlocks2, threadsPerBlock2>>>((half*)c_param->im2col_delta_o,
					(half*)current->delta_o, c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2], 
					(c_param->prev_size[0] * c_param->prev_size[1] * c_param->prev_size[2]) * flat_f_size, 
					1, 1, 1, back_padding[0], back_padding[1], back_padding[2], 
					c_param->stride[0] - 1 , c_param->stride[1] - 1 , c_param->stride[2] - 1 ,
					c_param->nb_filters, depth_padding, image_padding, 0, current->c_network->batch_size,
					c_param->f_size[0], c_param->f_size[1], c_param->f_size[2], flat_f_size, 
					c_param->nb_area[0], c_param->nb_area[1], c_param->nb_area[2], 
					c_param->prev_size[0], c_param->prev_size[1], 0, 0);
				break;
				
			case BF16C_FP32A:
				im2col_kernel_v4_BF16<<< numBlocks2, threadsPerBlock2>>>((hip_bfloat16*)c_param->im2col_delta_o,
					(hip_bfloat16*)current->delta_o, c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2], 
					(c_param->prev_size[0] * c_param->prev_size[1] * c_param->prev_size[2]) * flat_f_size, 
					1, 1, 1, back_padding[0], back_padding[1], back_padding[2], 
					c_param->stride[0] - 1 , c_param->stride[1] - 1 , c_param->stride[2] - 1 ,
					c_param->nb_filters, depth_padding, image_padding, 0, current->c_network->batch_size,
					c_param->f_size[0], c_param->f_size[1], c_param->f_size[2], flat_f_size, 
					c_param->nb_area[0], c_param->nb_area[1], c_param->nb_area[2], 
					c_param->prev_size[0], c_param->prev_size[1], 0, 0);
				break;
		}

		hipblasGemmEx(cu_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, c_param->prev_size[0]*c_param->prev_size[1]*c_param->prev_size[2] 
			*current->c_network->batch_size, c_param->prev_depth, c_param->f_size[0]*c_param->f_size[1]*c_param->f_size[2] 
			*c_param->nb_filters, cu_alpha, c_param->im2col_delta_o, cuda_data_type, c_param->f_size[0] 
			*c_param->f_size[1]*c_param->f_size[2]*c_param->nb_filters, c_param->rotated_filters, cuda_data_type, 
			c_param->f_size[0]*c_param->f_size[1]*c_param->f_size[2]*c_param->nb_filters, cu_beta, 
			current->previous->delta_o, cuda_data_type, c_param->prev_size[0]*c_param->prev_size[1]*c_param->prev_size[2]
			*current->c_network->batch_size, cuda_compute_type, HIPBLAS_GEMM_DEFAULT);

		//update gradiant regarding the previous layer activation function
		//WARNING : ONLY WORK IF PREVIOUS LAYER IS A CONV AS OUTPUT AND DELTA_O SHARE THE SAME DATA ORDER
		current->previous->deriv_activation(current->previous);
	}
	
	//########################  WEIGHTS UPDATE   ########################
	
	if(!current->frozen)
	{
		//based on the recovered delta_o provided by the next layer propagation
		//HIPBLAS_OP_N ,in this case, is a transpose of regular input (see forward function)
		
		set_cu_learning_rate_and_momentum(current->c_network);
		
		hipblasGemmEx(cu_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, (c_param->flat_f_size+c_param->TC_padding), c_param->nb_filters, 
			c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] * current->c_network->batch_size, 
			cu_learning_rate, c_param->im2col_input, cuda_data_type, 
			(c_param->flat_f_size + c_param->TC_padding), current->delta_o, cuda_data_type, 
			c_param->nb_area[0] * c_param->nb_area[1] * c_param->nb_area[2] * current->c_network->batch_size,
			cu_momentum, c_param->update, cuda_data_type, 
			(c_param->flat_f_size + c_param->TC_padding), cuda_compute_type, HIPBLAS_GEMM_DEFAULT);
		
		switch(current->c_network->use_cuda_TC)
		{
			case FP32C_FP32A:
			case TF32C_FP32A:
				cuda_update_weights(current->c_network, c_param->filters, c_param->update, 
					(c_param->flat_f_size + c_param->TC_padding) * c_param->nb_filters);
				break;
			
			case FP16C_FP32A:
			case FP16C_FP16A:
				cuda_update_weights(current->c_network, c_param->FP32_filters, c_param->update, 
					(c_param->flat_f_size + c_param->TC_padding) * c_param->nb_filters);
				break;
			
			case BF16C_FP32A:
				cuda_update_weights(current->c_network, c_param->FP32_filters, c_param->update, 
					(c_param->flat_f_size + c_param->TC_padding) * c_param->nb_filters);
				break;
		}
	}
	
	free(back_padding);
}


//One of the most important function, aims to convert an image into a table that contains all the
//areas that will be used for convolution. Highly redundant but still allows a significant speed up
//due to subsequent matrix operations. Currently memory bound despite only 1 load per element of the original image.
//VERSION 4.2
__global__ void im2col_kernel_v4_FP32(float* output, float* input, int image_size, int flat_image_size, int stride_w, int stride_h ,int stride_d, int padding_w, int padding_h, int padding_d, int internal_padding_w, int internal_padding_h, int internal_padding_d, int channel, int channel_padding, int image_padding, int TC_padding, int batch_size, int f_size_w, int f_size_h, int f_size_d, int flat_f_size, int w_size, int h_size, int d_size, int nb_area_w, int nb_area_h, int bias_in, int bias_out)
{
	int p = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	int i = blockIdx.z*blockDim.z + threadIdx.z;
	
	half local_pix;
	
	int w, h, d, x, y, z;
	int pos_w_filter, pos_h_filter, pos_d_filter;
	int loc;
	
	if( i < batch_size)
	{
		input += i*(image_padding + bias_in);
		output += i*(flat_image_size);
		
		if(c < channel)
		{
			input += c * channel_padding;
			output += c * f_size_w*f_size_h*f_size_d;
			if(p < image_size)
			{
				local_pix = input[p];
			
				d = (p / (w_size*h_size))*(1 + internal_padding_d) + padding_d;
				h = (p % (w_size*h_size) / w_size)*(1 + internal_padding_h) + padding_h;
				w = (p % (w_size*h_size) % w_size)*(1 + internal_padding_w) + padding_w;
				
				for(z = d/stride_d; (d-z*stride_d < f_size_d)  /*&& (z >= 0)*/ ; z -=1)
				{
					pos_d_filter = d-z*stride_d;
					if((pos_d_filter + padding_d < 0) || (pos_d_filter > d_size*(1 + internal_padding_d) + 2*padding_d - f_size_d))
						continue;
					for(x = w/stride_w; (w-x*stride_w < f_size_w) /*&& (x >= 0)*/; x -= 1)
					{
						pos_w_filter = w-x*stride_w;
						if((pos_w_filter + padding_w < 0) || (pos_w_filter > w_size*(1 + internal_padding_w) + 2*padding_w - f_size_w))
							continue;
						for(y = h/stride_h; (h-y*stride_h < f_size_h) /*&& (y >= 0)*/; y -= 1)
						{
							pos_h_filter = h-y*stride_h;
							if((pos_h_filter + padding_h < 0) || (pos_h_filter > h_size*(1 + internal_padding_h) + 2*padding_h - f_size_h))
								continue;
							loc = z*nb_area_w*nb_area_h*(flat_f_size+TC_padding) + y*nb_area_w*(flat_f_size+TC_padding) 
								+ x*(flat_f_size+TC_padding) + pos_w_filter + pos_h_filter*f_size_w + pos_d_filter*f_size_w*f_size_h;
							if((bias_out && (loc)%(flat_f_size+TC_padding) >= flat_f_size - 1))
								continue;
							if(loc >= 0 && loc < flat_image_size)
								output[loc] = local_pix;
						}
					}
				}
			}
		}
	}
}

__global__ void im2col_kernel_v4_FP16(half* output, half* input, int image_size, int flat_image_size, int stride_w, int stride_h ,int stride_d, int padding_w, int padding_h, int padding_d, int internal_padding_w, int internal_padding_h, int internal_padding_d, int channel, int channel_padding, int image_padding, int TC_padding, int batch_size, int f_size_w, int f_size_h, int f_size_d, int flat_f_size, int w_size, int h_size, int d_size, int nb_area_w, int nb_area_h, int bias_in, int bias_out)
{
	int p = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	int i = blockIdx.z*blockDim.z + threadIdx.z;
	
	half local_pix;
	
	int w, h, d, x, y, z;
	int pos_w_filter, pos_h_filter, pos_d_filter;
	int loc;
	
	if( i < batch_size)
	{
		input += i*(image_padding + bias_in);
		output += i*(flat_image_size);
		
		if(c < channel)
		{
			input += c * channel_padding;
			output += c * f_size_w*f_size_h*f_size_d;
			if(p < image_size)
			{
				local_pix = input[p];
			
				d = (p / (w_size*h_size))*(1 + internal_padding_d) + padding_d;
				h = (p % (w_size*h_size) / w_size)*(1 + internal_padding_h) + padding_h;
				w = (p % (w_size*h_size) % w_size)*(1 + internal_padding_w) + padding_w;
				
				for(z = d/stride_d; (d-z*stride_d < f_size_d)  /*&& (z >= 0)*/ ; z -=1)
				{
					pos_d_filter = d-z*stride_d;
					if((pos_d_filter + padding_d < 0) || (pos_d_filter > d_size*(1 + internal_padding_d) + 2*padding_d - f_size_d))
						continue;
					for(x = w/stride_w; (w-x*stride_w < f_size_w) /*&& (x >= 0)*/; x -= 1)
					{
						pos_w_filter = w-x*stride_w;
						if((pos_w_filter + padding_w < 0) || (pos_w_filter > w_size*(1 + internal_padding_w) + 2*padding_w - f_size_w))
							continue;
						for(y = h/stride_h; (h-y*stride_h < f_size_h) /*&& (y >= 0)*/; y -= 1)
						{
							pos_h_filter = h-y*stride_h;
							if((pos_h_filter + padding_h < 0) || (pos_h_filter > h_size*(1 + internal_padding_h) + 2*padding_h - f_size_h))
								continue;
							loc = z*nb_area_w*nb_area_h*(flat_f_size+TC_padding) + y*nb_area_w*(flat_f_size+TC_padding) 
								+ x*(flat_f_size+TC_padding) + pos_w_filter + pos_h_filter*f_size_w + pos_d_filter*f_size_w*f_size_h;
							if((bias_out && (loc)%(flat_f_size+TC_padding) >= flat_f_size - 1))
								continue;
							if(loc >= 0 && loc < flat_image_size)
								output[loc] = local_pix;
						}
					}
				}
			}
		}
	}
}

__global__ void im2col_kernel_v4_BF16(hip_bfloat16* output, hip_bfloat16* input, int image_size, int flat_image_size, int stride_w, int stride_h ,int stride_d, int padding_w, int padding_h, int padding_d, int internal_padding_w, int internal_padding_h, int internal_padding_d, int channel, int channel_padding, int image_padding, int TC_padding, int batch_size, int f_size_w, int f_size_h, int f_size_d, int flat_f_size, int w_size, int h_size, int d_size, int nb_area_w, int nb_area_h, int bias_in, int bias_out)
{
	int p = blockIdx.x*blockDim.x + threadIdx.x;
	int c = blockIdx.y*blockDim.y + threadIdx.y;
	int i = blockIdx.z*blockDim.z + threadIdx.z;
	
	hip_bfloat16 local_pix;
	
	int w, h, d, x, y, z;
	int pos_w_filter, pos_h_filter, pos_d_filter;
	int loc;
	
	if( i < batch_size)
	{
		input += i*(image_padding + bias_in);
		output += i*(flat_image_size);
		
		if(c < channel)
		{
			input += c * channel_padding;
			output += c * f_size_w*f_size_h*f_size_d;
			if(p < image_size)
			{
				local_pix = input[p];
			
				d = (p / (w_size*h_size))*(1 + internal_padding_d) + padding_d;
				h = (p % (w_size*h_size) / w_size)*(1 + internal_padding_h) + padding_h;
				w = (p % (w_size*h_size) % w_size)*(1 + internal_padding_w) + padding_w;
				
				for(z = d/stride_d; (d-z*stride_d < f_size_d)  /*&& (z >= 0)*/ ; z -=1)
				{
					pos_d_filter = d-z*stride_d;
					if((pos_d_filter + padding_d < 0) || (pos_d_filter > d_size*(1 + internal_padding_d) + 2*padding_d - f_size_d))
						continue;
					for(x = w/stride_w; (w-x*stride_w < f_size_w) /*&& (x >= 0)*/; x -= 1)
					{
						pos_w_filter = w-x*stride_w;
						if((pos_w_filter + padding_w < 0) || (pos_w_filter > w_size*(1 + internal_padding_w) + 2*padding_w - f_size_w))
							continue;
						for(y = h/stride_h; (h-y*stride_h < f_size_h) /*&& (y >= 0)*/; y -= 1)
						{
							pos_h_filter = h-y*stride_h;
							if((pos_h_filter + padding_h < 0) || (pos_h_filter > h_size*(1 + internal_padding_h) + 2*padding_h - f_size_h))
								continue;
							loc = z*nb_area_w*nb_area_h*(flat_f_size+TC_padding) + y*nb_area_w*(flat_f_size+TC_padding) + x*(flat_f_size+TC_padding) + pos_w_filter + pos_h_filter*f_size_w + pos_d_filter*f_size_w*f_size_h;
							if((bias_out && (loc)%(flat_f_size+TC_padding) >= flat_f_size - 1))
								continue;
							if(loc >= 0 && loc < flat_image_size)
								output[loc] = local_pix;
						}
					}
				}
			}
		}
	}
}


__global__ void cuda_rotate_filter_matrix_FP32(float* in, float* out, int nb_rows, int TC_padding, int depth_size, int nb_filters_in, int len)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int x, y, depth_id;
	
	if(i < len)
	{
		//#####################################
		//Rotate and move the filters
		x = i / nb_rows;
		y = i % nb_rows;
		
		if(y < nb_rows-1-TC_padding) //remove the weights of the bias nodes
		{
			depth_id = y / depth_size;
			
			out[depth_id * depth_size*nb_filters_in + x * depth_size + (depth_size - 1 - y%depth_size)] = in[x*nb_rows+y];
		}	
	}	
}


__global__ void cuda_rotate_filter_matrix_FP16(half* in, half* out, int nb_rows, int TC_padding, int depth_size, int nb_filters_in, int len)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int x, y, depth_id;
	
	if(i < len)
	{
		//#####################################
		//Rotate and move the filters
		x = i / nb_rows;
		y = i % nb_rows;
		
		if(y < nb_rows-1-TC_padding) //remove the weights of the bias nodes
		{
			depth_id = y / depth_size;
			
			out[depth_id * depth_size*nb_filters_in + x * depth_size + (depth_size - 1 - y%depth_size)] = in[x*nb_rows+y];
		}
		
	}	
}


__global__ void cuda_rotate_filter_matrix_BF16(hip_bfloat16 *in, hip_bfloat16 *out, int nb_rows, int TC_padding, int depth_size, int nb_filters_in, int len)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int x, y, depth_id;
	
	if(i < len)
	{
		//#####################################
		//Rotate and move the filters
		x = i / nb_rows;
		y = i % nb_rows;
		
		if(y < nb_rows-1-TC_padding) //remove the weights of the bias nodes
		{
			depth_id = y / depth_size;
			
			out[depth_id * depth_size*nb_filters_in + x * depth_size + (depth_size - 1 - y%depth_size)] = in[x*nb_rows+y];
		}
		
	}
}


__global__ void init_block_state_conv(unsigned int seed,  hiprandState_t* states)
{
	hiprand_init((seed << 20) + blockIdx.x, /* the seed can be the same for each core, here we pass the time in from the CPU */
              0, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! 
			     Currently use an alternative definition with Id adjunct to seed*/
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}


__global__ void cuda_dropout_select_conv(int* mask, int size, float drop_rate, hiprandState_t* states)
{
	int i = blockIdx.x;
	
	float rand;
	if(i < size)
	{
		rand = hiprand_uniform(&states[i]);
		if(rand < drop_rate)
			mask[i] = 0;
		else
			mask[i] = 1;
	}
}

__global__ void cuda_dropout_apply_conv_FP32(float* table, int batch_size, int dim, int* mask, int size)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	
	int c_depth = j / dim;
	int current_id = j % dim;
	int offset = dim*batch_size;

	if(i < batch_size && j < size)
	{
		table[i*dim + c_depth*offset + current_id] *= mask[j];
	}
}

__global__ void cuda_dropout_apply_conv_FP16(half* table, int batch_size, int dim, int* mask, int size)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;

	int c_depth = j / dim;
    int current_id = j % dim;
    int offset = dim*batch_size;

    if(i < batch_size && j < size)
    {
            table[i*dim + c_depth*offset + current_id] *= mask[j];
    }
}


__global__ void cuda_dropout_apply_conv_BF16(hip_bfloat16* table, int batch_size, int dim, int* mask, int size)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int i = blockIdx.y*blockDim.y + threadIdx.y;

	int c_depth = j / dim;
    int current_id = j % dim;
    int offset = dim*batch_size;

    if(i < batch_size && j < size)
    {
            table[i*dim + c_depth*offset + current_id] *= mask[j];
    }
}






